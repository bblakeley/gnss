#include <stdlib.h>
#include <complex.h>
#include <hipfft/hipfft.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#include "dnsparams.h"
#include "struct_def.h"

void allocate_memory(){
	// // Allocate memory for statistics structs on both device and host
	int n, nGPUs;

	// // Declare extern variables (to pull def's from declare.h)
	extern gpuinfo gpu;	
	extern fftinfo fft;
	extern statistics h_stats;
	extern statistics stats;	

  extern double **k;

  extern fielddata h_vel;
  extern fielddata vel;
  extern fielddata rhs;
  extern fielddata rhs_old;

	extern hipfftDoubleComplex **temp_advective;

	// Make local copy of number of GPUs (for readability)
	nGPUs = gpu.nGPUs;
	printf("Allocating data on %d GPUs!\n",nGPUs);

	// Allocate pinned memory on the host side that stores array of pointers for FFT operations
	hipHostAlloc((void**)&fft,         		 sizeof(fftinfo),   					       hipHostMallocMapped);		
	hipHostAlloc((void**)&fft.p1d,    			 nGPUs*sizeof(hipfftHandle *), 			 hipHostMallocMapped);		// Allocate memory for array of cufftHandles to store nGPUs worth 1d plans
	hipHostAlloc((void**)&fft.p2d,    			 nGPUs*sizeof(hipfftHandle *), 			 hipHostMallocMapped);		// Allocate memory array of 2dplans
	hipHostAlloc((void**)&fft.invp2d, 			 nGPUs*sizeof(hipfftHandle *), 			 hipHostMallocMapped);		// Array of inverse 2d plans
	hipHostAlloc((void**)&fft.wsize_f, 		 nGPUs*sizeof(size_t *), 						 hipHostMallocMapped);		// Size of workspace required for forward transform
	hipHostAlloc((void**)&fft.wsize_i, 		 nGPUs*sizeof(size_t *), 						 hipHostMallocMapped);		// Size of workspace required for inverse transform
	hipHostAlloc((void**)&fft.wspace, 			 nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);		// Array of pointers to FFT workspace on each device
	hipHostAlloc((void**)&fft.temp, 				 nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);		// Array of pointers to scratch (temporary) memory on each device
	hipHostAlloc((void**)&fft.temp_reorder, nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);		// Same as above, different temp variable

	// Allocate memory on host
	h_vel.u = (double **)malloc(sizeof(double *)*nGPUs);
	h_vel.v = (double **)malloc(sizeof(double *)*nGPUs);
	h_vel.w = (double **)malloc(sizeof(double *)*nGPUs);
	h_vel.s = (double **)malloc(sizeof(double *)*nGPUs);

	// Declare struct for holding statistics on the host
	// &h_stats = malloc(sizeof(statistics));
	// Allocate memory on host for statistics
	h_stats.Vrms    			= (double **)malloc(sizeof(double *));
	h_stats.KE      			= (double **)malloc(sizeof(double *));
	h_stats.epsilon 			= (double **)malloc(sizeof(double *));
	h_stats.eta     			= (double **)malloc(sizeof(double *));
	h_stats.l      		 		= (double **)malloc(sizeof(double *));
	h_stats.lambda 				= (double **)malloc(sizeof(double *));
	h_stats.chi   			  = (double **)malloc(sizeof(double *));
	h_stats.area_scalar  	= (double **)malloc(sizeof(double *));
	h_stats.area_tnti	  	= (double **)malloc(sizeof(double *));
	h_stats.energy_spect	= (double **)malloc(sizeof(double *));

	h_stats.Vrms[0]    			= (double *)malloc(sizeof(double)*(nt/n_stats+1));
	h_stats.KE[0]      			= (double *)malloc(sizeof(double)*(nt/n_stats+1));
	h_stats.epsilon[0] 			= (double *)malloc(sizeof(double)*(nt/n_stats+1));
	h_stats.eta[0]     			= (double *)malloc(sizeof(double)*(nt/n_stats+1));
	h_stats.l[0]      		 		= (double *)malloc(sizeof(double)*(nt/n_stats+1));
	h_stats.lambda[0] 				= (double *)malloc(sizeof(double)*(nt/n_stats+1));
	h_stats.chi[0]   			  = (double *)malloc(sizeof(double)*(nt/n_stats+1));
	h_stats.area_scalar[0]  	= (double *)malloc(sizeof(double)*(nt/n_stats+1));
	h_stats.area_tnti[0]	  	= (double *)malloc(sizeof(double)*(nt/n_stats+1));
	h_stats.energy_spect[0]		= (double *)malloc(sizeof(double)*(nt/n_stats+1)*NX/2);

	// Allocate pinned memory on the host side that stores array of pointers
	hipHostAlloc((void**)&k, nGPUs*sizeof(double *), hipHostMallocMapped);

	hipHostAlloc((void**)&vel.uh, 		 nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);
	hipHostAlloc((void**)&vel.vh, 		 nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);
	hipHostAlloc((void**)&vel.wh, 		 nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);
	hipHostAlloc((void**)&vel.sh, 		 nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);
	hipHostAlloc((void**)&vel.left, 	 nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);
	hipHostAlloc((void**)&vel.right,  nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);

	// hipHostAlloc((void**)&vel.u, 		 nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);
	// hipHostAlloc((void**)&vel.v, 		 nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);
	// hipHostAlloc((void**)&vel.w, 		 nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);
	// hipHostAlloc((void**)&vel.s, 		 nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);

	hipHostAlloc((void**)&rhs.uh, 		 nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);
	hipHostAlloc((void**)&rhs.vh, 		 nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);
	hipHostAlloc((void**)&rhs.wh, 		 nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);
	hipHostAlloc((void**)&rhs.sh, 		 nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);

	// hipHostAlloc((void**)&rhs.u, 		 nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);
	// hipHostAlloc((void**)&rhs.v, 		 nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);
	// hipHostAlloc((void**)&rhs.w, 		 nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);
	// hipHostAlloc((void**)&rhs.s, 		 nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);

	hipHostAlloc((void**)&rhs_old.uh, nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);
	hipHostAlloc((void**)&rhs_old.vh, nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);
	hipHostAlloc((void**)&rhs_old.wh, nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);
	hipHostAlloc((void**)&rhs_old.sh, nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);

	// hipHostAlloc((void**)&rhs_old.u, nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);
	// hipHostAlloc((void**)&rhs_old.v, nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);
	// hipHostAlloc((void**)&rhs_old.w, nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);
	// hipHostAlloc((void**)&rhs_old.s, nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);

	hipHostAlloc((void**)&temp_advective, nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);
	
	// For statistics
	// hipHostAlloc((void**)&stats, sizeof(stats), hipHostMallocMapped);
	hipHostAlloc((void**)&stats.Vrms,    nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);
	hipHostAlloc((void**)&stats.KE,      nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);
	hipHostAlloc((void**)&stats.epsilon, nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);
	hipHostAlloc((void**)&stats.eta,     nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);
	hipHostAlloc((void**)&stats.l,       nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);
	hipHostAlloc((void**)&stats.lambda,  nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);
	hipHostAlloc((void**)&stats.chi,     nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);
	hipHostAlloc((void**)&stats.area_scalar,     nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);
	hipHostAlloc((void**)&stats.area_tnti,     nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);
	hipHostAlloc((void**)&stats.energy_spect,     nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);

	// Allocate memory for arrays
	for (n = 0; n<nGPUs; ++n){
		hipSetDevice(n);

		h_vel.u[n] = (double *)malloc(sizeof(complex double)*gpu.nx[n]*NY*NZ2);
		h_vel.v[n] = (double *)malloc(sizeof(complex double)*gpu.nx[n]*NY*NZ2);
		h_vel.w[n] = (double *)malloc(sizeof(complex double)*gpu.nx[n]*NY*NZ2);
		h_vel.s[n] = (double *)malloc(sizeof(complex double)*gpu.nx[n]*NY*NZ2);

		checkCudaErrors( hipMalloc((void **)&k[n], sizeof(double)*NX ) );

		// Allocate memory for velocity fields
		checkCudaErrors( hipMalloc((void **)&vel.uh[n],    sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) ); 
		checkCudaErrors( hipMalloc((void **)&vel.vh[n],    sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) );
		checkCudaErrors( hipMalloc((void **)&vel.wh[n],    sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) );
		checkCudaErrors( hipMalloc((void **)&vel.sh[n],    sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) );
		checkCudaErrors( hipMalloc((void **)&vel.left[n],  sizeof(hipfftDoubleComplex)*RAD*NY*NZ2) );
		checkCudaErrors( hipMalloc((void **)&vel.right[n], sizeof(hipfftDoubleComplex)*RAD*NY*NZ2) );

		checkCudaErrors( hipMalloc((void **)&rhs.uh[n], sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) ); 
		checkCudaErrors( hipMalloc((void **)&rhs.vh[n], sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) );
		checkCudaErrors( hipMalloc((void **)&rhs.wh[n], sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) );
		checkCudaErrors( hipMalloc((void **)&rhs.sh[n], sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) );

		checkCudaErrors( hipMallocManaged((void **)&rhs_old.uh[n], sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) ); 
		checkCudaErrors( hipMallocManaged((void **)&rhs_old.vh[n], sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) );
		checkCudaErrors( hipMallocManaged((void **)&rhs_old.wh[n], sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) );
		checkCudaErrors( hipMallocManaged((void **)&rhs_old.sh[n], sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) );

		checkCudaErrors( hipMalloc((void **)&temp_advective[n],   sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) );
		checkCudaErrors( hipMalloc((void **)&fft.temp[n], 			 	 sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) );
		checkCudaErrors( hipMalloc((void **)&fft.temp_reorder[n], sizeof(hipfftDoubleComplex)*gpu.nx[n]*NZ2) );
		
		// Statistics
		checkCudaErrors( hipMallocManaged((void **)&stats.Vrms[n],    			sizeof(double)*(nt/n_stats+1)) );
		checkCudaErrors( hipMallocManaged((void **)&stats.KE[n],      			sizeof(double)*(nt/n_stats+1)) );
		checkCudaErrors( hipMallocManaged((void **)&stats.epsilon[n], 			sizeof(double)*(nt/n_stats+1)) );
		checkCudaErrors( hipMallocManaged((void **)&stats.eta[n],     			sizeof(double)*(nt/n_stats+1)) );
		checkCudaErrors( hipMallocManaged((void **)&stats.l[n],       			sizeof(double)*(nt/n_stats+1)) );
		checkCudaErrors( hipMallocManaged((void **)&stats.lambda[n], 		  sizeof(double)*(nt/n_stats+1)) );
		checkCudaErrors( hipMallocManaged((void **)&stats.chi[n],     			sizeof(double)*(nt/n_stats+1)) );
		checkCudaErrors( hipMallocManaged((void **)&stats.area_scalar[n],  sizeof(double)*(nt/n_stats+1)) );
		checkCudaErrors( hipMallocManaged((void **)&stats.area_tnti[n],  	sizeof(double)*(nt/n_stats+1)) );
		checkCudaErrors( hipMallocManaged((void **)&stats.energy_spect[n], sizeof(double)*(nt/n_stats+1)*NX/2) );

		printf("Data allocated on Device %d\n", n);
	}

		// Cast pointers to complex arrays to real array and store in the proper struct field
		vel.u = (hipfftDoubleReal **)vel.uh;
		vel.v = (hipfftDoubleReal **)vel.vh;
		vel.w = (hipfftDoubleReal **)vel.wh;
		vel.s = (hipfftDoubleReal **)vel.sh;
	
		rhs.u = (hipfftDoubleReal **)rhs.uh;
		rhs.v = (hipfftDoubleReal **)rhs.vh;
		rhs.w = (hipfftDoubleReal **)rhs.wh;
		rhs.s = (hipfftDoubleReal **)rhs.sh;

		rhs_old.u = (hipfftDoubleReal **)rhs_old.uh;
		rhs_old.v = (hipfftDoubleReal **)rhs_old.vh;
		rhs_old.w = (hipfftDoubleReal **)rhs_old.wh;
		rhs_old.s = (hipfftDoubleReal **)rhs_old.sh;			

	// Initialize everything to 0 before entering the rest of the routine
	for (n = 0; n<nGPUs; ++n){
		hipSetDevice(n);

		checkCudaErrors( hipMemset(k[n], 0, sizeof(double)*NX) );

		checkCudaErrors( hipMemset(vel.u[n], 0, sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) );
		checkCudaErrors( hipMemset(vel.v[n], 0, sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) );
		checkCudaErrors( hipMemset(vel.w[n], 0, sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) );
		checkCudaErrors( hipMemset(vel.s[n], 0, sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) );

		checkCudaErrors( hipMemset(rhs.u[n], 0, sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) );
		checkCudaErrors( hipMemset(rhs.v[n], 0, sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) );
		checkCudaErrors( hipMemset(rhs.w[n], 0, sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) );
		checkCudaErrors( hipMemset(rhs.s[n], 0, sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) );

		checkCudaErrors( hipMemset(rhs_old.u[n], 0, sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) );
		checkCudaErrors( hipMemset(rhs_old.v[n], 0, sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) );
		checkCudaErrors( hipMemset(rhs_old.w[n], 0, sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) );
		checkCudaErrors( hipMemset(rhs_old.s[n], 0, sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) );

		checkCudaErrors( hipMemset(temp_advective[n], 0, sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) );

		checkCudaErrors( hipMemset(stats.Vrms[n], 				0, sizeof(double)*(nt/n_stats+1)) );
		checkCudaErrors( hipMemset(stats.KE[n], 					0, sizeof(double)*(nt/n_stats+1)) );
		checkCudaErrors( hipMemset(stats.epsilon[n], 		0, sizeof(double)*(nt/n_stats+1)) );
		checkCudaErrors( hipMemset(stats.eta[n], 				0, sizeof(double)*(nt/n_stats+1)) );
		checkCudaErrors( hipMemset(stats.l[n], 					0, sizeof(double)*(nt/n_stats+1)) );
		checkCudaErrors( hipMemset(stats.lambda[n], 			0, sizeof(double)*(nt/n_stats+1)) );
		checkCudaErrors( hipMemset(stats.chi[n], 				0, sizeof(double)*(nt/n_stats+1)) );
		checkCudaErrors( hipMemset(stats.area_scalar[n], 0, sizeof(double)*(nt/n_stats+1)) );
		checkCudaErrors( hipMemset(stats.area_tnti[n],	  0, sizeof(double)*(nt/n_stats+1)) );
	}

	return;
}
