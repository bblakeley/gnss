#include <stdlib.h>
#include <complex.h>
#include <hipfft/hipfft.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#include "dnsparams.h"
#include "struct_def.h"

void allocate_memory(){
	// Allocate memory for statistics structs on both device and host
	int n, nGPUs;

	// // Declare extern variables (to pull def's from declare.h)
	extern gpudata gpu;	
	extern fftdata fft;
	extern statistics *stats;	
	extern profile Yprof;

  extern griddata grid;

  extern fielddata h_vel;
  extern fielddata vel;
  extern fielddata rhs;
  extern fielddata rhs_old;
	extern fielddata temp;

	// Make local copy of number of GPUs (for readability)
	nGPUs = gpu.nGPUs;
	printf("Allocating data on %d GPUs!\n",nGPUs);
	
	// Allocate pinned memory on the host side that stores array of pointers for FFT operations
	hipHostAlloc((void**)&fft,         		 sizeof(fftdata),   					       hipHostMallocMapped);		
	hipHostAlloc((void**)&fft.p1d,    			 nGPUs*sizeof(hipfftHandle *), 			 hipHostMallocMapped);		// Allocate memory for array of cufftHandles to store nGPUs worth 1d plans
	hipHostAlloc((void**)&fft.p2d,    			 nGPUs*sizeof(hipfftHandle *), 			 hipHostMallocMapped);		// Allocate memory array of 2dplans
	hipHostAlloc((void**)&fft.invp2d, 			 nGPUs*sizeof(hipfftHandle *), 			 hipHostMallocMapped);		// Array of inverse 2d plans
	hipHostAlloc((void**)&fft.p3d,          1*sizeof(hipfftHandle *),                hipHostMallocMapped); // Only used when nGPUs=1
	hipHostAlloc((void**)&fft.invp3d,       1*sizeof(hipfftHandle *),                hipHostMallocMapped); // Only used when nGPUs=1
	hipHostAlloc((void**)&fft.wsize_f, 		 nGPUs*sizeof(size_t *), 						 hipHostMallocMapped);		// Size of workspace required for forward transform
	hipHostAlloc((void**)&fft.wsize_i, 		 nGPUs*sizeof(size_t *), 						 hipHostMallocMapped);		// Size of workspace required for inverse transform
	hipHostAlloc((void**)&fft.wspace, 			 nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);		// Array of pointers to FFT workspace on each device
	hipHostAlloc((void**)&fft.temp, 				 nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);		// Array of pointers to scratch (temporary) memory on each device
	hipHostAlloc((void**)&fft.temp_reorder, nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);		// Same as above, different temp variable
	
	// Allocate memory on host to store averaged profile data
	hipHostAlloc((void**)&Yprof,         sizeof(profile),   					       hipHostMallocMapped);
	hipHostAlloc((void**)&Yprof.u,       nGPUs*sizeof(double *),    	       hipHostMallocMapped);
	hipHostAlloc((void**)&Yprof.v,       nGPUs*sizeof(double *),    	       hipHostMallocMapped);
	hipHostAlloc((void**)&Yprof.w,       nGPUs*sizeof(double *),    	       hipHostMallocMapped);
	hipHostAlloc((void**)&Yprof.s,       nGPUs*sizeof(double *),    	       hipHostMallocMapped);

	// Allocate pinned memory on the host side that stores array of pointers
	hipHostAlloc((void**)&grid, sizeof(fielddata), hipHostMallocMapped);
	hipHostAlloc((void**)&grid.kx, nGPUs*sizeof(double *), hipHostMallocMapped);

	// Allocate memory on host
	hipHostAlloc((void**)&h_vel, sizeof(fielddata), hipHostMallocMapped);
	h_vel.u = (double **)malloc(sizeof(double *)*nGPUs);
	h_vel.v = (double **)malloc(sizeof(double *)*nGPUs);
	h_vel.w = (double **)malloc(sizeof(double *)*nGPUs);
	h_vel.s = (double **)malloc(sizeof(double *)*nGPUs);

  hipHostAlloc((void**)&vel, sizeof(fielddata), hipHostMallocMapped);
	hipHostAlloc((void**)&vel.uh, 		 nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);
	hipHostAlloc((void**)&vel.vh, 		 nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);
	hipHostAlloc((void**)&vel.wh, 		 nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);
	hipHostAlloc((void**)&vel.sh, 		 nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);
	hipHostAlloc((void**)&vel.left, 	 nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);
	hipHostAlloc((void**)&vel.right,  nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);

  hipHostAlloc((void**)&rhs, sizeof(fielddata), hipHostMallocMapped);
	hipHostAlloc((void**)&rhs.uh, 		 nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);
	hipHostAlloc((void**)&rhs.vh, 		 nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);
	hipHostAlloc((void**)&rhs.wh, 		 nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);
	hipHostAlloc((void**)&rhs.sh, 		 nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);
	hipHostAlloc((void**)&rhs.left, 		 nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);
	hipHostAlloc((void**)&rhs.right, 		 nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);

  hipHostAlloc((void**)&rhs_old, sizeof(fielddata), hipHostMallocMapped);
	hipHostAlloc((void**)&rhs_old.uh, nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);
	hipHostAlloc((void**)&rhs_old.vh, nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);
	hipHostAlloc((void**)&rhs_old.wh, nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);
	hipHostAlloc((void**)&rhs_old.sh, nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);

  hipHostAlloc((void**)&temp, sizeof(fielddata), hipHostMallocMapped);
	hipHostAlloc((void**)&temp.uh, nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);
	
	// For statistics
	hipHostAlloc(&stats, nGPUs*sizeof(statistics *), hipHostMallocMapped);

	// Allocate memory for arrays on each GPU
	for (n = 0; n<nGPUs; ++n){
		hipSetDevice(n);

		h_vel.u[n] = (double *)malloc(sizeof(complex double)*gpu.nx[n]*NY*NZ2);
		h_vel.v[n] = (double *)malloc(sizeof(complex double)*gpu.nx[n]*NY*NZ2);
		h_vel.w[n] = (double *)malloc(sizeof(complex double)*gpu.nx[n]*NY*NZ2);
		h_vel.s[n] = (double *)malloc(sizeof(complex double)*gpu.nx[n]*NY*NZ2);

		checkCudaErrors( hipMalloc((void **)&grid.kx[n], sizeof(double)*NX ) );

		// Allocate memory for velocity fields
		checkCudaErrors( hipMalloc((void **)&vel.uh[n],    sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) ); 
		checkCudaErrors( hipMalloc((void **)&vel.vh[n],    sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) );
		checkCudaErrors( hipMalloc((void **)&vel.wh[n],    sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) );
		checkCudaErrors( hipMalloc((void **)&vel.sh[n],    sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) );
		checkCudaErrors( hipMalloc((void **)&vel.left[n],  sizeof(hipfftDoubleComplex)*RAD*NY*NZ2) );
		checkCudaErrors( hipMalloc((void **)&vel.right[n], sizeof(hipfftDoubleComplex)*RAD*NY*NZ2) );

		checkCudaErrors( hipMalloc((void **)&rhs.uh[n], sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) ); 
		checkCudaErrors( hipMalloc((void **)&rhs.vh[n], sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) );
		checkCudaErrors( hipMalloc((void **)&rhs.wh[n], sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) );
		checkCudaErrors( hipMalloc((void **)&rhs.sh[n], sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) );
		checkCudaErrors( hipMalloc((void **)&rhs.left[n],  sizeof(hipfftDoubleComplex)*RAD*NY*NZ2) );
		checkCudaErrors( hipMalloc((void **)&rhs.right[n],  sizeof(hipfftDoubleComplex)*RAD*NY*NZ2) );

		checkCudaErrors( hipMallocManaged((void **)&rhs_old.uh[n], sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) ); 
		checkCudaErrors( hipMallocManaged((void **)&rhs_old.vh[n], sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) );
		checkCudaErrors( hipMallocManaged((void **)&rhs_old.wh[n], sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) );
		checkCudaErrors( hipMallocManaged((void **)&rhs_old.sh[n], sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) );

		checkCudaErrors( hipMalloc((void **)&temp.uh[n],   sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) );
		checkCudaErrors( hipMalloc((void **)&fft.temp[n], 			 	 sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) );
		checkCudaErrors( hipMalloc((void **)&fft.temp_reorder[n], sizeof(hipfftDoubleComplex)*gpu.nx[n]*NZ2) );
		
		// Statistics
		checkCudaErrors( hipMallocManaged( (void **)&stats[n], sizeof(statistics) ));
		
		// Averaged Profiles
		checkCudaErrors( hipMallocManaged( (void **)&Yprof.u[n], sizeof(double)*NY) );
		checkCudaErrors( hipMallocManaged( (void **)&Yprof.v[n], sizeof(double)*NY) );
		checkCudaErrors( hipMallocManaged( (void **)&Yprof.w[n], sizeof(double)*NY) );
		checkCudaErrors( hipMallocManaged( (void **)&Yprof.s[n], sizeof(double)*NY) );

		printf("Data allocated on Device %d\n", n);
	}

		// Cast pointers to complex arrays to real array and store in the proper struct field
		vel.u = (hipfftDoubleReal **)vel.uh;
		vel.v = (hipfftDoubleReal **)vel.vh;
		vel.w = (hipfftDoubleReal **)vel.wh;
		vel.s = (hipfftDoubleReal **)vel.sh;
	
		rhs.u = (hipfftDoubleReal **)rhs.uh;
		rhs.v = (hipfftDoubleReal **)rhs.vh;
		rhs.w = (hipfftDoubleReal **)rhs.wh;
		rhs.s = (hipfftDoubleReal **)rhs.sh;

		rhs_old.u = (hipfftDoubleReal **)rhs_old.uh;
		rhs_old.v = (hipfftDoubleReal **)rhs_old.vh;
		rhs_old.w = (hipfftDoubleReal **)rhs_old.wh;
		rhs_old.s = (hipfftDoubleReal **)rhs_old.sh;		
		
		temp.u = (hipfftDoubleReal **)temp.uh;

	// Initialize everything to 0 before entering the rest of the routine
	for (n = 0; n<nGPUs; ++n){
		hipSetDevice(n);

		checkCudaErrors( hipMemset(grid.kx[n], 0.0, sizeof(double)*NX) );

		checkCudaErrors( hipMemset(vel.u[n], 0.0, sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) );
		checkCudaErrors( hipMemset(vel.v[n], 0.0, sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) );
		checkCudaErrors( hipMemset(vel.w[n], 0.0, sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) );
		checkCudaErrors( hipMemset(vel.s[n], 0.0, sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) );

		checkCudaErrors( hipMemset(rhs.u[n], 0.0, sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) );
		checkCudaErrors( hipMemset(rhs.v[n], 0.0, sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) );
		checkCudaErrors( hipMemset(rhs.w[n], 0.0, sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) );
		checkCudaErrors( hipMemset(rhs.s[n], 0.0, sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) );

		checkCudaErrors( hipMemset(rhs_old.u[n], 0.0, sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) );
		checkCudaErrors( hipMemset(rhs_old.v[n], 0.0, sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) );
		checkCudaErrors( hipMemset(rhs_old.w[n], 0.0, sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) );
		checkCudaErrors( hipMemset(rhs_old.s[n], 0.0, sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) );

		checkCudaErrors( hipMemset(temp.uh[n], 0.0, sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) );
		
		checkCudaErrors( hipMemset(Yprof.u[n], 0.0, sizeof(double)*NY) );
		checkCudaErrors( hipMemset(Yprof.v[n], 0.0, sizeof(double)*NY) );
		checkCudaErrors( hipMemset(Yprof.w[n], 0.0, sizeof(double)*NY) );
		checkCudaErrors( hipMemset(Yprof.s[n], 0.0, sizeof(double)*NY) );
	}

	return;
}


void deallocate_memory(){
	int n, nGPUs;
	// // Declare extern variables (to pull def's from declare.h)
	extern gpudata gpu;	
	extern fftdata fft;
	extern statistics h_stats;
	extern statistics *stats;	
	extern profile Yprof;

  extern griddata grid;

  extern fielddata h_vel;
  extern fielddata vel;
  extern fielddata rhs;
  extern fielddata rhs_old;

	extern fielddata temp;

	// Make local copy of number of GPUs (for readability)
	nGPUs = gpu.nGPUs;

	// Deallocate GPU memory
	for(n = 0; n<nGPUs; ++n){
		hipSetDevice(n);

		hipFree(fft.temp[n]);
		hipFree(fft.temp_reorder[n]);
   	hipFree(fft.wspace[n]);

		hipFree(grid.kx[n]);

		free(h_vel.u[n]);
		free(h_vel.v[n]);
		free(h_vel.w[n]);
		free(h_vel.s[n]);

		hipFree(vel.u[n]);
		hipFree(vel.v[n]);
		hipFree(vel.w[n]);
		hipFree(vel.s[n]);

		hipFree(rhs.u[n]);
		hipFree(rhs.v[n]);
		hipFree(rhs.w[n]);
		hipFree(rhs.s[n]);

		hipFree(rhs_old.u[n]);
		hipFree(rhs_old.v[n]);
		hipFree(rhs_old.w[n]);
		hipFree(rhs_old.s[n]);

		hipFree(temp.u[n]);

		hipFree(&stats[n]);
		// Averaged Profiles
		hipFree(Yprof.u[n]);
		hipFree(Yprof.v[n]);
		hipFree(Yprof.w[n]);
		hipFree(Yprof.s[n]);

		// Destroy cufft plans
		hipfftDestroy(fft.p1d[n]);
		hipfftDestroy(fft.p2d[n]);
		hipfftDestroy(fft.invp2d[n]);
		hipfftDestroy(fft.p3d[n]);
		hipfftDestroy(fft.invp3d[n]);
	}
	
	// Deallocate pointer arrays on host memory
	hipHostFree(gpu.gpunum);
	hipHostFree(gpu.ny);
	hipHostFree(gpu.nx);
	hipHostFree(gpu.start_x);
	hipHostFree(gpu.start_y);

	hipHostFree(grid.kx);
	hipHostFree(&grid);

	hipHostFree(temp.uh);
	hipHostFree(&temp);

	hipHostFree(fft.wsize_f);
	hipHostFree(fft.wsize_i);
	hipHostFree(fft.wspace);
	hipHostFree(fft.temp);
	hipHostFree(fft.temp_reorder);
	hipHostFree(&fft);

	hipHostFree(vel.uh);
	hipHostFree(vel.vh);
	hipHostFree(vel.wh);
	hipHostFree(vel.sh);
	hipHostFree(&vel);

	hipHostFree(rhs.uh);
	hipHostFree(rhs.vh);
	hipHostFree(rhs.wh);
	hipHostFree(rhs.sh);
	hipHostFree(&rhs);

	hipHostFree(rhs_old.uh);
	hipHostFree(rhs_old.vh);
	hipHostFree(rhs_old.wh);
	hipHostFree(rhs_old.sh);
	hipHostFree(&rhs_old);

	hipHostFree(stats);
	
	// Averaged Profiles
	hipHostFree(Yprof.u);
	hipHostFree(Yprof.v);
	hipHostFree(Yprof.w);
	hipHostFree(Yprof.s);
	hipHostFree(&Yprof);

	// Deallocate memory on CPU
	free(h_vel.u);
	free(h_vel.v);
	free(h_vel.w);
	free(h_vel.s);
	hipHostFree(&h_vel);

	return;
}
