#include <stdlib.h>
#include <complex.h>
#include <hipfft/hipfft.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#include "dnsparams.h"
#include "struct_def.h"

void allocate_memory(){
	// // Allocate memory for statistics structs on both device and host
	int n, nGPUs;

	// // Declare extern variables (to pull def's from declare.h)
	extern gpuinfo gpu;	
	extern fftinfo fft;
	extern statistics *stats;	
	extern profile Yprofile;

  extern double **k;

  extern fielddata h_vel;
  extern fielddata vel;
  extern fielddata rhs;
  extern fielddata rhs_old;

	extern hipfftDoubleComplex **temp_advective;

	// Make local copy of number of GPUs (for readability)
	nGPUs = gpu.nGPUs;
	printf("Allocating data on %d GPUs!\n",nGPUs);
	
	// Allocate pinned memory on the host side that stores array of pointers for FFT operations
	hipHostAlloc((void**)&fft,         		 sizeof(fftinfo),   					       hipHostMallocMapped);		
	hipHostAlloc((void**)&fft.p1d,    			 nGPUs*sizeof(hipfftHandle *), 			 hipHostMallocMapped);		// Allocate memory for array of cufftHandles to store nGPUs worth 1d plans
	hipHostAlloc((void**)&fft.p2d,    			 nGPUs*sizeof(hipfftHandle *), 			 hipHostMallocMapped);		// Allocate memory array of 2dplans
	hipHostAlloc((void**)&fft.invp2d, 			 nGPUs*sizeof(hipfftHandle *), 			 hipHostMallocMapped);		// Array of inverse 2d plans
	hipHostAlloc((void**)&fft.wsize_f, 		 nGPUs*sizeof(size_t *), 						 hipHostMallocMapped);		// Size of workspace required for forward transform
	hipHostAlloc((void**)&fft.wsize_i, 		 nGPUs*sizeof(size_t *), 						 hipHostMallocMapped);		// Size of workspace required for inverse transform
	hipHostAlloc((void**)&fft.wspace, 			 nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);		// Array of pointers to FFT workspace on each device
	hipHostAlloc((void**)&fft.temp, 				 nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);		// Array of pointers to scratch (temporary) memory on each device
	hipHostAlloc((void**)&fft.temp_reorder, nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);		// Same as above, different temp variable
	
	// Allocate memory on host to store averaged profile data
	hipHostAlloc((void**)&Yprofile,         sizeof(profile),   					       hipHostMallocMapped);
	hipHostAlloc((void**)&Yprofile.u,       nGPUs*sizeof(double *),    	       hipHostMallocMapped);
	hipHostAlloc((void**)&Yprofile.v,       nGPUs*sizeof(double *),    	       hipHostMallocMapped);
	hipHostAlloc((void**)&Yprofile.w,       nGPUs*sizeof(double *),    	       hipHostMallocMapped);
	hipHostAlloc((void**)&Yprofile.s,       nGPUs*sizeof(double *),    	       hipHostMallocMapped);
	hipHostAlloc((void**)&Yprofile.uu,       nGPUs*sizeof(double *),    	       hipHostMallocMapped);
	hipHostAlloc((void**)&Yprofile.vv,       nGPUs*sizeof(double *),    	       hipHostMallocMapped);
	hipHostAlloc((void**)&Yprofile.ww,       nGPUs*sizeof(double *),    	       hipHostMallocMapped);
	hipHostAlloc((void**)&Yprofile.ss,       nGPUs*sizeof(double *),    	       hipHostMallocMapped);

	// Allocate memory on host
	h_vel.u = (double **)malloc(sizeof(double *)*nGPUs);
	h_vel.v = (double **)malloc(sizeof(double *)*nGPUs);
	h_vel.w = (double **)malloc(sizeof(double *)*nGPUs);
	h_vel.s = (double **)malloc(sizeof(double *)*nGPUs);

	// Allocate pinned memory on the host side that stores array of pointers
	hipHostAlloc((void**)&k, nGPUs*sizeof(double *), hipHostMallocMapped);

	hipHostAlloc((void**)&vel.uh, 		 nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);
	hipHostAlloc((void**)&vel.vh, 		 nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);
	hipHostAlloc((void**)&vel.wh, 		 nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);
	hipHostAlloc((void**)&vel.sh, 		 nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);
	hipHostAlloc((void**)&vel.left, 	 nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);
	hipHostAlloc((void**)&vel.right,  nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);

	hipHostAlloc((void**)&rhs.uh, 		 nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);
	hipHostAlloc((void**)&rhs.vh, 		 nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);
	hipHostAlloc((void**)&rhs.wh, 		 nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);
	hipHostAlloc((void**)&rhs.sh, 		 nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);
	hipHostAlloc((void**)&rhs.left, 		 nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);
	hipHostAlloc((void**)&rhs.right, 		 nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);

	hipHostAlloc((void**)&rhs_old.uh, nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);
	hipHostAlloc((void**)&rhs_old.vh, nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);
	hipHostAlloc((void**)&rhs_old.wh, nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);
	hipHostAlloc((void**)&rhs_old.sh, nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);

	hipHostAlloc((void**)&temp_advective, nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);
	
	// For statistics
	hipHostAlloc(&stats, nGPUs*sizeof(statistics *), hipHostMallocMapped);

	// Allocate memory for arrays
	for (n = 0; n<nGPUs; ++n){
		hipSetDevice(n);

		h_vel.u[n] = (double *)malloc(sizeof(complex double)*gpu.nx[n]*NY*NZ2);
		h_vel.v[n] = (double *)malloc(sizeof(complex double)*gpu.nx[n]*NY*NZ2);
		h_vel.w[n] = (double *)malloc(sizeof(complex double)*gpu.nx[n]*NY*NZ2);
		h_vel.s[n] = (double *)malloc(sizeof(complex double)*gpu.nx[n]*NY*NZ2);

		checkCudaErrors( hipMalloc((void **)&k[n], sizeof(double)*NX ) );

		// Allocate memory for velocity fields
		checkCudaErrors( hipMalloc((void **)&vel.uh[n],    sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) ); 
		checkCudaErrors( hipMalloc((void **)&vel.vh[n],    sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) );
		checkCudaErrors( hipMalloc((void **)&vel.wh[n],    sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) );
		checkCudaErrors( hipMalloc((void **)&vel.sh[n],    sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) );
		checkCudaErrors( hipMalloc((void **)&vel.left[n],  sizeof(hipfftDoubleComplex)*RAD*NY*NZ2) );
		checkCudaErrors( hipMalloc((void **)&vel.right[n], sizeof(hipfftDoubleComplex)*RAD*NY*NZ2) );

		checkCudaErrors( hipMalloc((void **)&rhs.uh[n], sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) ); 
		checkCudaErrors( hipMalloc((void **)&rhs.vh[n], sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) );
		checkCudaErrors( hipMalloc((void **)&rhs.wh[n], sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) );
		checkCudaErrors( hipMalloc((void **)&rhs.sh[n], sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) );
		checkCudaErrors( hipMalloc((void **)&rhs.left[n],  sizeof(hipfftDoubleComplex)*RAD*NY*NZ2) );
		checkCudaErrors( hipMalloc((void **)&rhs.right[n],  sizeof(hipfftDoubleComplex)*RAD*NY*NZ2) );

		checkCudaErrors( hipMallocManaged((void **)&rhs_old.uh[n], sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) ); 
		checkCudaErrors( hipMallocManaged((void **)&rhs_old.vh[n], sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) );
		checkCudaErrors( hipMallocManaged((void **)&rhs_old.wh[n], sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) );
		checkCudaErrors( hipMallocManaged((void **)&rhs_old.sh[n], sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) );

		checkCudaErrors( hipMalloc((void **)&temp_advective[n],   sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) );
		checkCudaErrors( hipMalloc((void **)&fft.temp[n], 			 	 sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) );
		checkCudaErrors( hipMalloc((void **)&fft.temp_reorder[n], sizeof(hipfftDoubleComplex)*gpu.nx[n]*NZ2) );
		
		// Statistics
		checkCudaErrors( hipMallocManaged( (void **)&stats[n], sizeof(statistics) ));
		
		// Averaged Profiles
		checkCudaErrors( hipMallocManaged( (void **)&Yprofile.u[n], sizeof(double)*NY) );
		checkCudaErrors( hipMallocManaged( (void **)&Yprofile.v[n], sizeof(double)*NY) );
		checkCudaErrors( hipMallocManaged( (void **)&Yprofile.w[n], sizeof(double)*NY) );
		checkCudaErrors( hipMallocManaged( (void **)&Yprofile.s[n], sizeof(double)*NY) );
		checkCudaErrors( hipMallocManaged( (void **)&Yprofile.uu[n], sizeof(double)*NY) );
		checkCudaErrors( hipMallocManaged( (void **)&Yprofile.vv[n], sizeof(double)*NY) );
		checkCudaErrors( hipMallocManaged( (void **)&Yprofile.ww[n], sizeof(double)*NY) );
		checkCudaErrors( hipMallocManaged( (void **)&Yprofile.ss[n], sizeof(double)*NY) );
		
		// Area statistics
		checkCudaErrors( hipMallocManaged( (void **)&stats[n].area_scalar, sizeof(double)*64) );
		checkCudaErrors( hipMallocManaged( (void **)&stats[n].area_omega, sizeof(double)*64) );

		printf("Data allocated on Device %d\n", n);
	}

		// Cast pointers to complex arrays to real array and store in the proper struct field
		vel.u = (hipfftDoubleReal **)vel.uh;
		vel.v = (hipfftDoubleReal **)vel.vh;
		vel.w = (hipfftDoubleReal **)vel.wh;
		vel.s = (hipfftDoubleReal **)vel.sh;
	
		rhs.u = (hipfftDoubleReal **)rhs.uh;
		rhs.v = (hipfftDoubleReal **)rhs.vh;
		rhs.w = (hipfftDoubleReal **)rhs.wh;
		rhs.s = (hipfftDoubleReal **)rhs.sh;

		rhs_old.u = (hipfftDoubleReal **)rhs_old.uh;
		rhs_old.v = (hipfftDoubleReal **)rhs_old.vh;
		rhs_old.w = (hipfftDoubleReal **)rhs_old.wh;
		rhs_old.s = (hipfftDoubleReal **)rhs_old.sh;			

	// Initialize everything to 0 before entering the rest of the routine
	for (n = 0; n<nGPUs; ++n){
		hipSetDevice(n);

		checkCudaErrors( hipMemset(k[n], 0, sizeof(double)*NX) );

		checkCudaErrors( hipMemset(vel.u[n], 0, sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) );
		checkCudaErrors( hipMemset(vel.v[n], 0, sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) );
		checkCudaErrors( hipMemset(vel.w[n], 0, sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) );
		checkCudaErrors( hipMemset(vel.s[n], 0, sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) );

		checkCudaErrors( hipMemset(rhs.u[n], 0, sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) );
		checkCudaErrors( hipMemset(rhs.v[n], 0, sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) );
		checkCudaErrors( hipMemset(rhs.w[n], 0, sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) );
		checkCudaErrors( hipMemset(rhs.s[n], 0, sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) );

		checkCudaErrors( hipMemset(rhs_old.u[n], 0, sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) );
		checkCudaErrors( hipMemset(rhs_old.v[n], 0, sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) );
		checkCudaErrors( hipMemset(rhs_old.w[n], 0, sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) );
		checkCudaErrors( hipMemset(rhs_old.s[n], 0, sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) );

		checkCudaErrors( hipMemset(temp_advective[n], 0, sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) );
		
		checkCudaErrors( hipMemset(Yprofile.u[n], 0, sizeof(double)*NY) );
		checkCudaErrors( hipMemset(Yprofile.v[n], 0, sizeof(double)*NY) );
		checkCudaErrors( hipMemset(Yprofile.w[n], 0, sizeof(double)*NY) );
		checkCudaErrors( hipMemset(Yprofile.s[n], 0, sizeof(double)*NY) );
	}

	return;
}


void deallocate_memory(){
	int n, nGPUs;
	// // Declare extern variables (to pull def's from declare.h)
	extern gpuinfo gpu;	
	extern fftinfo fft;
	extern statistics h_stats;
	extern statistics *stats;	
	extern profile Yprofile;

  extern double **k;

  extern fielddata h_vel;
  extern fielddata vel;
  extern fielddata rhs;
  extern fielddata rhs_old;

	extern hipfftDoubleComplex **temp_advective;

	// Make local copy of number of GPUs (for readability)
	nGPUs = gpu.nGPUs;

	// Deallocate GPU memory
	for(n = 0; n<nGPUs; ++n){
		hipSetDevice(n);

		hipFree(fft.temp[n]);
		hipFree(fft.temp_reorder[n]);
   	hipFree(fft.wspace[n]);

		hipFree(k[n]);

		free(h_vel.u[n]);
		free(h_vel.v[n]);
		free(h_vel.w[n]);
		free(h_vel.s[n]);

		hipFree(vel.u[n]);
		hipFree(vel.v[n]);
		hipFree(vel.w[n]);
		hipFree(vel.s[n]);

		hipFree(rhs.u[n]);
		hipFree(rhs.v[n]);
		hipFree(rhs.w[n]);
		hipFree(rhs.s[n]);

		hipFree(rhs_old.u[n]);
		hipFree(rhs_old.v[n]);
		hipFree(rhs_old.w[n]);
		hipFree(rhs_old.s[n]);

		hipFree(temp_advective[n]);

		hipFree(&stats[n]);
		// Averaged Profiles
		hipFree(Yprofile.u[n]);
		hipFree(Yprofile.v[n]);
		hipFree(Yprofile.w[n]);
		hipFree(Yprofile.s[n]);

		// Destroy cufft plans
		hipfftDestroy(fft.p1d[n]);
		hipfftDestroy(fft.p2d[n]);
		hipfftDestroy(fft.invp2d[n]);
	}
	
	// Deallocate pointer arrays on host memory
	hipHostFree(gpu.gpunum);
	hipHostFree(gpu.ny);
	hipHostFree(gpu.nx);
	hipHostFree(gpu.start_x);
	hipHostFree(gpu.start_y);

	hipHostFree(k);

	hipHostFree(temp_advective);

	hipHostFree(fft.wsize_f);
	hipHostFree(fft.wsize_i);
	hipHostFree(fft.wspace);
	hipHostFree(fft.temp);
	hipHostFree(fft.temp_reorder);
	hipHostFree(&fft);

	hipHostFree(vel.uh);
	hipHostFree(vel.vh);
	hipHostFree(vel.wh);
	hipHostFree(vel.sh);

	hipHostFree(rhs.uh);
	hipHostFree(rhs.vh);
	hipHostFree(rhs.wh);
	hipHostFree(rhs.sh);

	hipHostFree(rhs_old.uh);
	hipHostFree(rhs_old.vh);
	hipHostFree(rhs_old.wh);
	hipHostFree(rhs_old.sh);

	hipHostFree(stats);
	
	// Averaged Profiles
	hipHostFree(Yprofile.u);
	hipHostFree(Yprofile.v);
	hipHostFree(Yprofile.w);
	hipHostFree(Yprofile.s);
	hipHostFree(&Yprofile);

	// Deallocate memory on CPU
	free(h_vel.u);
	free(h_vel.v);
	free(h_vel.w);
	free(h_vel.s);

	return;
}
