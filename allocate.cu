#include <stdlib.h>
#include <complex.h>
#include <hipfft/hipfft.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#include "dnsparams.h"
#include "struct_def.h"

void allocate_memory(){
	// Allocate memory for statistics structs on both device and host
	int n, nGPUs;

	// Declare extern variables (to pull def's from declare.h)
	extern gpudata gpu;	
	extern fftdata fft;
	extern statistics *stats;	
	extern profile Yprof;
	
  extern griddata grid;

  extern fielddata h_vel;
  extern fielddata vel;
  extern fielddata rhs;
  extern fielddata rhs_old;
	extern fielddata temp;

	// Make local copy of number of GPUs (for readability)
	nGPUs = gpu.nGPUs;
	printf("Allocating data on %d GPUs!\n",nGPUs);
	
	// Allocate pinned memory on the host side that stores array of pointers for FFT operations
	hipHostAlloc((void**)&fft,         		 sizeof(fftdata),   					       hipHostMallocMapped);		
	hipHostAlloc((void**)&fft.p1d,    			 nGPUs*sizeof(hipfftHandle *), 			 hipHostMallocMapped);		// Allocate memory for array of cufftHandles to store nGPUs worth 1d plans
	hipHostAlloc((void**)&fft.p2d,    			 nGPUs*sizeof(hipfftHandle *), 			 hipHostMallocMapped);		// Allocate memory array of 2dplans
	hipHostAlloc((void**)&fft.invp2d, 			 nGPUs*sizeof(hipfftHandle *), 			 hipHostMallocMapped);		// Array of inverse 2d plans
	hipHostAlloc((void**)&fft.p3d,          1*sizeof(hipfftHandle *),                hipHostMallocMapped); // Only used when nGPUs=1
	hipHostAlloc((void**)&fft.invp3d,       1*sizeof(hipfftHandle *),                hipHostMallocMapped); // Only used when nGPUs=1
	hipHostAlloc((void**)&fft.wsize_f, 		 nGPUs*sizeof(size_t *), 						 hipHostMallocMapped);		// Size of workspace required for forward transform
	hipHostAlloc((void**)&fft.wsize_i, 		 nGPUs*sizeof(size_t *), 						 hipHostMallocMapped);		// Size of workspace required for inverse transform
	hipHostAlloc((void**)&fft.wspace, 			 nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);		// Array of pointers to FFT workspace on each device
	hipHostAlloc((void**)&fft.temp, 				 nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);		// Array of pointers to scratch (temporary) memory on each device
	hipHostAlloc((void**)&fft.temp_reorder_f, nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);		// Same as above, different temp variable
	hipHostAlloc((void**)&fft.temp_reorder_i, nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);		// Same as above, different temp variable
	
	// Allocate memory on host to store averaged profile data
	hipHostAlloc((void**)&Yprof,         sizeof(profile),   					       hipHostMallocMapped);
	hipHostAlloc((void**)&Yprof.u,       nGPUs*sizeof(double *),    	       hipHostMallocMapped);
	hipHostAlloc((void**)&Yprof.v,       nGPUs*sizeof(double *),    	       hipHostMallocMapped);
	hipHostAlloc((void**)&Yprof.w,       nGPUs*sizeof(double *),    	       hipHostMallocMapped);
	hipHostAlloc((void**)&Yprof.s,       nGPUs*sizeof(double *),    	       hipHostMallocMapped);
	hipHostAlloc((void**)&Yprof.c,       nGPUs*sizeof(double *),    	       hipHostMallocMapped);

	// Allocate pinned memory on the host side that stores array of pointers
	hipHostAlloc((void**)&grid, sizeof(fielddata), hipHostMallocMapped);
	hipHostAlloc((void**)&grid.kx, nGPUs*sizeof(double *), hipHostMallocMapped);
	hipHostAlloc((void**)&grid.ky, nGPUs*sizeof(double *), hipHostMallocMapped);
	hipHostAlloc((void**)&grid.kz, nGPUs*sizeof(double *), hipHostMallocMapped);

	// Allocate memory on host	
	hipHostAlloc((void**)&h_vel, sizeof(fielddata), hipHostMallocMapped);
	h_vel.uh = (hipfftDoubleComplex **)malloc(sizeof(hipfftDoubleComplex *)*nGPUs);
	h_vel.vh = (hipfftDoubleComplex **)malloc(sizeof(hipfftDoubleComplex *)*nGPUs);
	h_vel.wh = (hipfftDoubleComplex **)malloc(sizeof(hipfftDoubleComplex *)*nGPUs);
	h_vel.sh = (hipfftDoubleComplex **)malloc(sizeof(hipfftDoubleComplex *)*nGPUs);
	h_vel.ch = (hipfftDoubleComplex **)malloc(sizeof(hipfftDoubleComplex *)*nGPUs);

  hipHostAlloc((void**)&vel, sizeof(fielddata), hipHostMallocMapped);
	hipHostAlloc((void**)&vel.uh, 		 nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);
	hipHostAlloc((void**)&vel.vh, 		 nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);
	hipHostAlloc((void**)&vel.wh, 		 nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);
	hipHostAlloc((void**)&vel.sh, 		 nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);
	hipHostAlloc((void**)&vel.ch, 		 nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);
	hipHostAlloc((void**)&vel.left, 	 nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);
	hipHostAlloc((void**)&vel.right,  nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);

  hipHostAlloc((void**)&rhs, sizeof(fielddata), hipHostMallocMapped);
	hipHostAlloc((void**)&rhs.uh, 		 nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);
	hipHostAlloc((void**)&rhs.vh, 		 nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);
	hipHostAlloc((void**)&rhs.wh, 		 nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);
	hipHostAlloc((void**)&rhs.sh, 		 nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);
	hipHostAlloc((void**)&rhs.ch, 		 nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);
	hipHostAlloc((void**)&rhs.left, 	 nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);
	hipHostAlloc((void**)&rhs.right,  nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);

  hipHostAlloc((void**)&rhs_old, sizeof(fielddata), hipHostMallocMapped);
	hipHostAlloc((void**)&rhs_old.uh, nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);
	hipHostAlloc((void**)&rhs_old.vh, nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);
	hipHostAlloc((void**)&rhs_old.wh, nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);
	hipHostAlloc((void**)&rhs_old.sh, nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);
	hipHostAlloc((void**)&rhs_old.ch, nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);

  hipHostAlloc((void**)&temp, sizeof(fielddata), hipHostMallocMapped);
	hipHostAlloc((void**)&temp.uh, nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);
	hipHostAlloc((void**)&temp.vh, nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);
	hipHostAlloc((void**)&temp.wh, nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);
	
	// For statistics
	hipHostAlloc(&stats, nGPUs*sizeof(statistics *), hipHostMallocMapped);

	// Allocate memory for arrays on each GPU
	for (n = 0; n<nGPUs; ++n){
		hipSetDevice(n);
		h_vel.uh[n] = (hipfftDoubleComplex *)malloc(sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2);
		h_vel.vh[n] = (hipfftDoubleComplex *)malloc(sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2);
		h_vel.wh[n] = (hipfftDoubleComplex *)malloc(sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2);
		h_vel.sh[n] = (hipfftDoubleComplex *)malloc(sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2);
		h_vel.ch[n] = (hipfftDoubleComplex *)malloc(sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2);

		checkCudaErrors( hipMalloc((void **)&grid.kx[n], sizeof(double)*NX ) );
		checkCudaErrors( hipMalloc((void **)&grid.ky[n], sizeof(double)*NY ) );
		checkCudaErrors( hipMalloc((void **)&grid.kz[n], sizeof(double)*NZ ) );

		// Allocate memory for velocity fields
		checkCudaErrors( hipMalloc((void **)&vel.uh[n],    sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) ); 
		checkCudaErrors( hipMalloc((void **)&vel.vh[n],    sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) );
		checkCudaErrors( hipMalloc((void **)&vel.wh[n],    sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) );
		checkCudaErrors( hipMalloc((void **)&vel.sh[n],    sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) );
		checkCudaErrors( hipMalloc((void **)&vel.ch[n],    sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) );
		checkCudaErrors( hipMalloc((void **)&vel.left[n],  sizeof(hipfftDoubleComplex)*RAD*NY*NZ2) );
		checkCudaErrors( hipMalloc((void **)&vel.right[n], sizeof(hipfftDoubleComplex)*RAD*NY*NZ2) );

		checkCudaErrors( hipMalloc((void **)&rhs.uh[n], sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) ); 
		checkCudaErrors( hipMalloc((void **)&rhs.vh[n], sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) );
		checkCudaErrors( hipMalloc((void **)&rhs.wh[n], sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) );
		checkCudaErrors( hipMalloc((void **)&rhs.sh[n], sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) );
		checkCudaErrors( hipMalloc((void **)&rhs.ch[n], sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) );
		checkCudaErrors( hipMalloc((void **)&rhs.left[n],  sizeof(hipfftDoubleComplex)*RAD*NY*NZ2) );
		checkCudaErrors( hipMalloc((void **)&rhs.right[n],  sizeof(hipfftDoubleComplex)*RAD*NY*NZ2) );

		checkCudaErrors( hipMalloc((void **)&rhs_old.uh[n], sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) ); 
		checkCudaErrors( hipMalloc((void **)&rhs_old.vh[n], sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) );
		checkCudaErrors( hipMalloc((void **)&rhs_old.wh[n], sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) );
		checkCudaErrors( hipMalloc((void **)&rhs_old.sh[n], sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) );
    checkCudaErrors( hipMalloc((void **)&rhs_old.ch[n], sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) );
    
		checkCudaErrors( hipMalloc((void **)&temp.uh[n], sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) ); 
		checkCudaErrors( hipMalloc((void **)&temp.vh[n], sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) );
		checkCudaErrors( hipMalloc((void **)&temp.wh[n], sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) );
		
		checkCudaErrors( hipMalloc((void **)&fft.temp[n], 			 	 sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) );
		checkCudaErrors( hipMalloc((void **)&fft.temp_reorder_f[n], sizeof(hipfftDoubleComplex)*gpu.nx[n]*NZ2) );
		checkCudaErrors( hipMalloc((void **)&fft.temp_reorder_i[n], sizeof(hipfftDoubleComplex)*gpu.ny[n]*NZ2) );
		
		// Statistics
		checkCudaErrors( hipMallocManaged( (void **)&stats[n], sizeof(statistics) ));
		
		// Averaged Profiles
		checkCudaErrors( hipMallocManaged( (void **)&Yprof.u[n], sizeof(double)*NY) );
		checkCudaErrors( hipMallocManaged( (void **)&Yprof.v[n], sizeof(double)*NY) );
		checkCudaErrors( hipMallocManaged( (void **)&Yprof.w[n], sizeof(double)*NY) );
		checkCudaErrors( hipMallocManaged( (void **)&Yprof.s[n], sizeof(double)*NY) );
		checkCudaErrors( hipMallocManaged( (void **)&Yprof.c[n], sizeof(double)*NY) );

		printf("Data allocated on Device %d\n", n);
	}

		// Cast pointers to complex arrays to real array and store in the proper struct field
		h_vel.u = (hipfftDoubleReal **)h_vel.uh;
		h_vel.v = (hipfftDoubleReal **)h_vel.vh;
		h_vel.w = (hipfftDoubleReal **)h_vel.wh;
		h_vel.s = (hipfftDoubleReal **)h_vel.sh;
		h_vel.c = (hipfftDoubleReal **)h_vel.ch;
		
		vel.u = (hipfftDoubleReal **)vel.uh;
		vel.v = (hipfftDoubleReal **)vel.vh;
		vel.w = (hipfftDoubleReal **)vel.wh;
		vel.s = (hipfftDoubleReal **)vel.sh;
		vel.c = (hipfftDoubleReal **)vel.ch;
	
		rhs.u = (hipfftDoubleReal **)rhs.uh;
		rhs.v = (hipfftDoubleReal **)rhs.vh;
		rhs.w = (hipfftDoubleReal **)rhs.wh;
		rhs.s = (hipfftDoubleReal **)rhs.sh;
		rhs.c = (hipfftDoubleReal **)rhs.ch;

		rhs_old.u = (hipfftDoubleReal **)rhs_old.uh;
		rhs_old.v = (hipfftDoubleReal **)rhs_old.vh;
		rhs_old.w = (hipfftDoubleReal **)rhs_old.wh;
		rhs_old.s = (hipfftDoubleReal **)rhs_old.sh;
		rhs_old.c = (hipfftDoubleReal **)rhs_old.ch;
		
		temp.u = (hipfftDoubleReal **)temp.uh;
		temp.v = (hipfftDoubleReal **)temp.vh;
		temp.w = (hipfftDoubleReal **)temp.wh;

	// Initialize everything to 0 before entering the rest of the routine
	for (n = 0; n<nGPUs; ++n){
		hipSetDevice(n);

		checkCudaErrors( hipMemset(grid.kx[n], 0.0, sizeof(double)*NX) );
		checkCudaErrors( hipMemset(grid.ky[n], 0.0, sizeof(double)*NY) );
		checkCudaErrors( hipMemset(grid.kz[n], 0.0, sizeof(double)*NZ) );

		checkCudaErrors( hipMemset(vel.u[n], 0.0, sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) );
		checkCudaErrors( hipMemset(vel.v[n], 0.0, sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) );
		checkCudaErrors( hipMemset(vel.w[n], 0.0, sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) );
		checkCudaErrors( hipMemset(vel.s[n], 0.0, sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) );
		checkCudaErrors( hipMemset(vel.c[n], 0.0, sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) );

		checkCudaErrors( hipMemset(rhs.u[n], 0.0, sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) );
		checkCudaErrors( hipMemset(rhs.v[n], 0.0, sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) );
		checkCudaErrors( hipMemset(rhs.w[n], 0.0, sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) );
		checkCudaErrors( hipMemset(rhs.s[n], 0.0, sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) );
		checkCudaErrors( hipMemset(rhs.c[n], 0.0, sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) );

		checkCudaErrors( hipMemset(rhs_old.u[n], 0.0, sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) );
		checkCudaErrors( hipMemset(rhs_old.v[n], 0.0, sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) );
		checkCudaErrors( hipMemset(rhs_old.w[n], 0.0, sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) );
		checkCudaErrors( hipMemset(rhs_old.s[n], 0.0, sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) );
		checkCudaErrors( hipMemset(rhs_old.c[n], 0.0, sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) );
		
		checkCudaErrors( hipMemset(temp.u[n], 0.0, sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) );
		checkCudaErrors( hipMemset(temp.v[n], 0.0, sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) );
		checkCudaErrors( hipMemset(temp.w[n], 0.0, sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2) );
		
		checkCudaErrors( hipMemset(Yprof.u[n], 0.0, sizeof(double)*NY) );
		checkCudaErrors( hipMemset(Yprof.v[n], 0.0, sizeof(double)*NY) );
		checkCudaErrors( hipMemset(Yprof.w[n], 0.0, sizeof(double)*NY) );
		checkCudaErrors( hipMemset(Yprof.s[n], 0.0, sizeof(double)*NY) );
		checkCudaErrors( hipMemset(Yprof.c[n], 0.0, sizeof(double)*NY) );
	}

	return;
}


void deallocate_memory(){
	int n, nGPUs;
	// // Declare extern variables (to pull def's from declare.h)
	extern gpudata gpu;	
	extern fftdata fft;
	extern statistics h_stats;
	extern statistics *stats;	
	extern profile Yprof;

  extern griddata grid;

  extern fielddata h_vel;
  extern fielddata vel;
  extern fielddata rhs;
  extern fielddata rhs_old;
  extern fielddata temp;

	// Make local copy of number of GPUs (for readability)
	nGPUs = gpu.nGPUs;

	// Deallocate GPU memory
	for(n = 0; n<nGPUs; ++n){
		hipSetDevice(n);

		hipFree(fft.temp[n]);
		hipFree(fft.temp_reorder_f[n]);
		hipFree(fft.temp_reorder_i[n]);
   	hipFree(fft.wspace[n]);

		hipFree(grid.kx[n]);
		hipFree(grid.ky[n]);
		hipFree(grid.kz[n]);

		free(h_vel.u[n]);
		free(h_vel.v[n]);
		free(h_vel.w[n]);
		free(h_vel.s[n]);
		free(h_vel.c[n]);

		hipFree(vel.u[n]);
		hipFree(vel.v[n]);
		hipFree(vel.w[n]);
		hipFree(vel.s[n]);
		hipFree(vel.c[n]);

		hipFree(rhs.u[n]);
		hipFree(rhs.v[n]);
		hipFree(rhs.w[n]);
		hipFree(rhs.s[n]);
		hipFree(rhs.c[n]);

		hipFree(rhs_old.u[n]);
		hipFree(rhs_old.v[n]);
		hipFree(rhs_old.w[n]);
		hipFree(rhs_old.s[n]);
		hipFree(rhs_old.c[n]);

		hipFree(temp.u[n]);
		hipFree(temp.v[n]);
		hipFree(temp.w[n]);

		hipFree(&stats[n]);
		// Averaged Profiles
		hipFree(Yprof.u[n]);
		hipFree(Yprof.v[n]);
		hipFree(Yprof.w[n]);
		hipFree(Yprof.s[n]);
		hipFree(Yprof.c[n]);

		// Destroy cufft plans
		hipfftDestroy(fft.p1d[n]);
		hipfftDestroy(fft.p2d[n]);
		hipfftDestroy(fft.invp2d[n]);
		hipfftDestroy(fft.p3d[n]);
		hipfftDestroy(fft.invp3d[n]);
	}
	
	// Deallocate pointer arrays on host memory
	hipHostFree(gpu.gpunum);
	hipHostFree(gpu.ny);
	hipHostFree(gpu.nx);
	hipHostFree(gpu.start_x);
	hipHostFree(gpu.start_y);

	hipHostFree(grid.kx);
	hipHostFree(grid.ky);
	hipHostFree(grid.kz);
	hipHostFree(&grid);

	hipHostFree(temp.uh);
	hipHostFree(&temp);

	hipHostFree(fft.wsize_f);
	hipHostFree(fft.wsize_i);
	hipHostFree(fft.wspace);
	hipHostFree(fft.temp);
	hipHostFree(fft.temp_reorder_f);
	hipHostFree(fft.temp_reorder_i);
	hipHostFree(&fft);

	hipHostFree(vel.uh);
	hipHostFree(vel.vh);
	hipHostFree(vel.wh);
	hipHostFree(vel.sh);
	hipHostFree(vel.ch);
	hipHostFree(&vel);

	hipHostFree(rhs.uh);
	hipHostFree(rhs.vh);
	hipHostFree(rhs.wh);
	hipHostFree(rhs.sh);
	hipHostFree(rhs.ch);
	hipHostFree(&rhs);

	hipHostFree(rhs_old.uh);
	hipHostFree(rhs_old.vh);
	hipHostFree(rhs_old.wh);
	hipHostFree(rhs_old.sh);
	hipHostFree(rhs_old.ch);
	hipHostFree(&rhs_old);

	hipHostFree(temp.uh);
	hipHostFree(temp.vh);
	hipHostFree(temp.wh);
	
	hipHostFree(stats);
	
	// Averaged Profiles
	hipHostFree(Yprof.u);
	hipHostFree(Yprof.v);
	hipHostFree(Yprof.w);
	hipHostFree(Yprof.s);
	hipHostFree(Yprof.c);
	hipHostFree(&Yprof);

	// Deallocate memory on CPU
	free(h_vel.u);
	free(h_vel.v);
	free(h_vel.w);
	free(h_vel.s);
	free(h_vel.c);
	hipHostFree(&h_vel);

	return;
}
