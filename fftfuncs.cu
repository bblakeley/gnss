#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <hipfft/hipfft.h>
#include <hip/hip_complex.h>
#include <hip/hip_runtime_api.h>
#include "dnsparams.h"
#include "cudafuncs.h"
#include "fftfuncs.h"

//==============================================================================
// Transpose algorithm
//==============================================================================
__global__ 
void organizeData(hipfftDoubleComplex *in, hipfftDoubleComplex *out, int N, int j)
{// Function to grab non-contiguous chunks of data and make them contiguous

	const int k = blockIdx.x * blockDim.x + threadIdx.x;
	if(k >= NZ2) return;

	for(int i=0; i<N; ++i){

		// printf("For thread %d, indexing begins at local index of %d, which maps to temp at location %d\n", k, (k+ NZ*j), k);
		out[k + i*NZ2] = in[k + NZ2*j + i*NY*NZ2];

	}

	return;
}

__global__ 
void organizeData_coalesced(hipfftDoubleComplex *in, hipfftDoubleComplex *out, int N, int j)
{// Function to grab non-contiguous chunks of data and make them contiguous

	const int i = blockIdx.x * blockDim.x + threadIdx.x;
	if(i >= N) return;

	for(int k=0; k<NZ2; ++k){

		// printf("For thread %d, indexing begins at local index of %d, which maps to temp at location %d\n", k, (k+ NZ*j), k);
		out[k + i*NZ2] = in[k + NZ2*j + i*NY*NZ2];

	}

	return;
}

__global__ 
void organizeData_2d(hipfftDoubleComplex *in, hipfftDoubleComplex *out, int N, int j)
{// Function to grab non-contiguous chunks of data and make them contiguous

	const int i = blockIdx.x * blockDim.x + threadIdx.x;
	const int k = blockIdx.y * blockDim.y + threadIdx.y;
	if(i >= N || k >= NZ2) return;

	out[k + i*NZ2] = in[k + NZ2*j + i*NY*NZ2];

	return;
}

void transpose_xy_mgpu(gpudata gpu, hipfftDoubleComplex **src, hipfftDoubleComplex **dst, hipfftDoubleComplex **temp)
{   // Transpose x and y directions (for a z-contiguous 1d array distributed across multiple GPUs)
	// This function loops through GPUs to do the transpose. Requires extra conversion to calculate the local index at the source location.
	// printf("Taking Transpose...\n");

	int n, j, local_idx_dst, dstNum;

	for(j=0; j<NY; ++j){
		for(n=0; n<gpu.nGPUs; ++n){
			hipSetDevice(n); 

			// Determine which GPU to send data to based on y-index, j
			dstNum = (j*gpu.nGPUs)/NY;

      const dim3 blockSize(TX, TZ, 1);
		  const dim3 gridSize(divUp(NX, TX), divUp(NZ2, TZ), 1);
		  // Open kernel that grabs all data 
		  organizeData_2d<<<gridSize,blockSize>>>(src[n], temp[n], gpu.nx[n], j);
			
			local_idx_dst = gpu.start_x[n]*NZ2 + (j - gpu.start_y[dstNum])*NZ2*NX;

			checkCudaErrors( hipMemcpyAsync(&dst[dstNum][local_idx_dst], temp[n], sizeof( hipfftDoubleComplex )*NZ2*gpu.nx[n], hipMemcpyDefault) );
		}
	}

	return;
}

//==============================================================================
// FFT functions
//==============================================================================

void plan2dFFT(gpudata gpu, fftdata fft){
// This function plans a 2-dimensional FFT to operate on the Z and Y directions (assumes Z-direction is contiguous in memory)
	int result;

	int n;
	for(n = 0; n<gpu.nGPUs; ++n){
	  hipSetDevice(n);

	  //Create plan for 2-D cuFFT, set cuFFT parameters
	  int rank = 2;
	  int size[] = {NY,NZ};           
	  int inembed[] = {NY,2*NZ2};         // inembed measures distance between dimensions of data
	  int onembed[] = {NY,NZ2};     // Uses half the domain for a R2C transform
	  int istride = 1;                        // istride is distance between consecutive elements
	  int ostride = 1;
	  int idist = NY*2*NZ2;                      // idist is the total length of one signal
	  int odist = NY*NZ2;
	  int batch = gpu.nx[n];                        // # of 2D FFTs to perform

	  // Create empty plan handles
	  hipfftCreate(&fft.p2d[n]);
	  hipfftCreate(&fft.invp2d[n]);

	  // Disable auto allocation of workspace memory for cuFFT plans
	  result = hipfftSetAutoAllocation(fft.p2d[n], 0);
	  if ( result != HIPFFT_SUCCESS){
      printf("CUFFT error: hipfftSetAutoAllocation failed on line %d, Error code %d\n", __LINE__, result);
	  return; }
	  result = hipfftSetAutoAllocation(fft.invp2d[n], 0);
	  if ( result != HIPFFT_SUCCESS){
      printf("CUFFT error: hipfftSetAutoAllocation failed on line %d, Error code %d\n", __LINE__, result);
	  return; }

	  // Plan Forward 2DFFT
	  result = hipfftMakePlanMany(fft.p2d[n], rank, size, inembed, istride, idist, onembed, ostride, odist, HIPFFT_D2Z, batch, &fft.wsize_f[n]);
	  if ( result != HIPFFT_SUCCESS){
      fprintf(stderr, "CUFFT error: cufftPlanforward 2D failed");
      printf(", Error code %d\n", result);
	  return; 
	  }

	  // Plan inverse 2DFFT
	  result = hipfftMakePlanMany(fft.invp2d[n], rank, size, onembed, ostride, odist, inembed, istride, idist, HIPFFT_Z2D, batch, &fft.wsize_i[n]);
	  if ( result != HIPFFT_SUCCESS){
      fprintf(stderr, "CUFFT error: cufftPlanforward 2D failed");
      printf(", Error code %d\n", result);
	  return; 
	  }

	  printf("The workspace size required for the forward transform is %lu.\n", fft.wsize_f[n]);
	  // printf("The workspace size required for the inverse transform is %lu.\n", fft.wsize_i[n]);

	  // Assuming that both workspaces are the same size (seems to be generally true), then the two workspaces can share an allocation - need to use maximum value here
	  // Allocate workspace memory
	  checkCudaErrors( hipMalloc(&fft.wspace[n], fft.wsize_f[n]) );

	  // Set cuFFT to use allocated workspace memory
	  result = hipfftSetWorkArea(fft.p2d[n], fft.wspace[n]);
	  if ( result != HIPFFT_SUCCESS){
    	printf("CUFFT error: ExecD2Z failed on line %d, Error code %d\n", __LINE__, result);
	  return; }
	  result = hipfftSetWorkArea(fft.invp2d[n], fft.wspace[n]);
	  if ( result != HIPFFT_SUCCESS){
      printf("CUFFT error: ExecD2Z failed on line %d, Error code %d\n", __LINE__, result);
	  return; }    

	}

	return;
}

void plan1dFFT(int nGPUs, fftdata fft){
// This function plans a 1-dimensional FFT to operate on the X direction (for X-direction not contiguous in memory, offset by Z-dimension)
    int result;

    int n;
    for(n = 0; n<nGPUs; ++n){
        hipSetDevice(n);
        //Create plan for cuFFT, set cuFFT parameters
        int rank = 1;               // Dimensionality of the FFT - constant at rank 1
        int size[] = {NX};          // size of each rank
        int inembed[] = {0};            // inembed measures distance between dimensions of data
        int onembed[] = {0};       // For complex to complex transform, input and output data have same dimensions
        int istride = NZ2;                        // istride is distance between consecutive elements
        int ostride = NZ2;
        int idist = 1;                     // idist is the total length of one signal
        int odist = 1;
        int batch = NZ2;                      // # of 1D FFTs to perform (assuming data has been transformed previously in the Z-Y directions)

        // Plan Forward 1DFFT
        result = hipfftPlanMany(&fft.p1d[n], rank, size, inembed, istride, idist, onembed, ostride, odist, HIPFFT_Z2Z, batch);
        if ( result != HIPFFT_SUCCESS){
            fprintf(stderr, "CUFFT error: cufftPlanforward failed");
        return; 
        }
    }
    
    return;
}

void plan3dFFT(fftdata fft){
// This function plans a 3-dimensional FFT
  hipfftResult result;

	// Create forward and inverse plans
	result = hipfftPlan3d(&fft.p3d[0], NX, NY, NZ, HIPFFT_D2Z);
	if ( result != HIPFFT_SUCCESS){
		fprintf(stderr, "CUFFT error: ExecD2Z Planning failed");
		printf(", Error code %d\n", result);	
	}

	result = hipfftPlan3d(&fft.invp3d[0], NX, NY, NZ, HIPFFT_Z2D); 
	if (result != HIPFFT_SUCCESS){
		fprintf(stderr, "CUFFT error: ExecZ2D Planning failed");
		printf(", Error code %d\n",result);
	}
    
    return;
}

void Execute1DFFT_Forward(hipfftHandle plan, int NY_per_GPU, hipfftDoubleComplex *f, hipfftDoubleComplex *fhat)
{

	hipfftResult result;
	// Loop through each slab in the Y-direction
	// Perform forward FFT
	for(int i=0; i<NY_per_GPU; ++i){
		result = hipfftExecZ2Z(plan, &f[i*NZ2*NX], &fhat[i*NZ2*NX], HIPFFT_FORWARD);
		if (  result != HIPFFT_SUCCESS){
			fprintf(stderr, "CUFFT error: ExecZ2Z failed, error code %d\n",(int)result);
		return; 
		}       
	}

	return;
}

void Execute1DFFT_Inverse(hipfftHandle plan, int NY_per_GPU, hipfftDoubleComplex *fhat, hipfftDoubleComplex *f)
{
	hipfftResult result;

	// Loop through each slab in the Y-direction
	// Perform forward FFT
	for(int i=0; i<NY_per_GPU; ++i){
		result = hipfftExecZ2Z(plan, &fhat[i*NZ2*NX], &f[i*NZ2*NX], HIPFFT_BACKWARD);
		if (  result != HIPFFT_SUCCESS){
			fprintf(stderr, "CUFFT error: ExecZ2Z failed, error code %d\n",(int)result);
		return; 
		}       
	}

	return;
}


void forwardTransform(fftdata fft, gpudata gpu, hipfftDoubleReal **f )
{ // Transform from physical to wave domain

	int result, n;
  
	// Take FFT in Z and Y directions
	for(n = 0; n<gpu.nGPUs; ++n){
		hipSetDevice(n);

		result = hipfftExecD2Z(fft.p2d[n], f[n], (hipfftDoubleComplex *)f[n]);
		if ( result != HIPFFT_SUCCESS){
			printf("CUFFT error: ExecD2Z failed on line %d, Error code %d\n", __LINE__, result);
		return; }
		// printf("Taking 2D forward FFT on GPU #%2d\n",n);
	}

	// Transpose X and Y dimensions
	transpose_xy_mgpu(gpu, (hipfftDoubleComplex **)f, fft.temp, fft.temp_reorder);

	// Take FFT in X direction (which has been transposed to what used to be the Y dimension)
	for(n = 0; n<gpu.nGPUs; ++n){
		hipSetDevice(n);

		Execute1DFFT_Forward(fft.p1d[n], gpu.ny[n], fft.temp[n], (hipfftDoubleComplex *)f[n]);
		// printf("Taking 1D forward FFT on GPU #%2d\n",n);
	}
	
	// results remain in transposed coordinates

	// printf("Forward Transform Completed...\n");

	return;
}

void inverseTransform(fftdata fft, gpudata gpu, hipfftDoubleComplex **f)
{ // Transform variables from wavespace to the physical domain 
	int result, n;
	
	// Data starts in transposed coordinates, x,y flipped

	// Take FFT in X direction (which has been transposed to what used to be the Y dimension)
	for(n = 0; n<gpu.nGPUs; ++n){
		hipSetDevice(n);
		Execute1DFFT_Inverse(fft.p1d[n], gpu.ny[n], f[n], fft.temp[n]);
		// printf("Taking 1D inverse FFT on GPU #%2d\n",n);
	}

	// Transpose X and Y directions
	transpose_xy_mgpu(gpu, fft.temp, f, fft.temp_reorder);

	for(n = 0; n<gpu.nGPUs; ++n){
		hipSetDevice(n);
		// Take inverse FFT in Z and Y direction
		result = hipfftExecZ2D(fft.invp2d[n], f[n], (hipfftDoubleReal *)f[n]);
		if ( result != HIPFFT_SUCCESS){
			printf("CUFFT error: ExecD2Z failed on line %d, Error code %d\n", __LINE__, result);
		return; }
		// printf("Taking 2D inverse FFT on GPU #%2d\n",n);
	}
	
	for(n = 0; n<gpu.nGPUs; ++n){
		hipSetDevice(n);
		const dim3 blockSize(TX, TY, TZ);
		const dim3 gridSize(divUp(gpu.nx[n], TX), divUp(NY, TY), divUp(NZ, TZ));

		scaleKernel_mgpu<<<gridSize, blockSize>>>(gpu.start_x[n], (hipfftDoubleReal *)f[n]);
	}

	// printf("Scaled Inverse Transform Completed...\n");

	return;
}
