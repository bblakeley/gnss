
// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <complex.h>

// includes, project
#include <hip/hip_runtime.h>
#include <>
#include <hipfft/hipfft.h>
#include <hip/hip_complex.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#include <timer.h>

// include parameters for DNS
#include "dnsparams.h"
#include "solver.h"
#include "statistics.h"
#include "cudafuncs.h"
#include "iofuncs.h"
#include "initialize.h"
#include "fftfuncs.h"
#include "struct_def.h"
#include "declare.h"
#include "allocate.h"
#include "deallocate.h"

void splitData(int numGPUs, gpuinfo *gpu) {
	int i, n;
	gpu->nGPUs = numGPUs;

	// Allocate pinned memory on the host that stores GPU info
	hipHostAlloc((void**)&gpu->gpunum,  numGPUs*sizeof(gpu->gpunum),  hipHostMallocMapped);
	hipHostAlloc((void**)&gpu->ny,      numGPUs*sizeof(gpu->ny),      hipHostMallocMapped);
	hipHostAlloc((void**)&gpu->nx,      numGPUs*sizeof(gpu->nx),      hipHostMallocMapped);
	hipHostAlloc((void**)&gpu->start_y, numGPUs*sizeof(gpu->start_y), hipHostMallocMapped);
	hipHostAlloc((void**)&gpu->start_x, numGPUs*sizeof(gpu->start_x), hipHostMallocMapped);

	// Add numGPUs to each GPU struct:
	for(i=0;i<numGPUs;++i){
		gpu->gpunum[i] = i;
	}

	// Splitting data in x-direction
	if(NX % numGPUs == 0){
		for (i=0; i<numGPUs; ++i){
			gpu->nx[i] = NX/numGPUs;
			gpu->start_x[i] = i*gpu->nx[i];           
		}
	}
	else {
		printf("Warning: number of GPUs is not an even multiple of the data size\n");
		n = NX/numGPUs;
		for(i=0; i<(numGPUs-1); ++i){
			gpu->nx[i] = n;
			gpu->start_x[i] = i*gpu->nx[i];
		}
		gpu->nx[numGPUs-1] = n + NX%numGPUs;
		gpu->start_x[numGPUs-1] = (numGPUs-1)*n;
	}
	// Now splitting data across y-direction
	if(NY % numGPUs == 0){
		for (i=0; i<numGPUs; ++i){
			gpu->ny[i] = NY/numGPUs;
			gpu->start_y[i] = i*gpu->ny[i];           
		}
	}
	else {
		printf("Warning: number of GPUs is not an even multiple of the data size\n");
		n = NY/numGPUs;
		for(i=0; i<(numGPUs-1); ++i){
			gpu->ny[i] = n;
			gpu->start_y[i] = i*gpu->ny[i];
		}
		gpu->ny[numGPUs-1] = n + NY%numGPUs;
		gpu->start_y[numGPUs-1] = (numGPUs-1)*n;
	}
	return;
}

int main (void)
{

//=====================================================================================================
// Program Start-up 
//=====================================================================================================
	// Set GPU's to use and list device properties
	int n, nGPUs;
	// Query number of devices attached to host
	// hipGetDeviceCount(&nGPUs);
	nGPUs=2;

	printf("Welcome to the GPU-based Navier-Stokes Solver! Configuration: \n"
		"Number of GPUs = %d \n "
		"Grid size = %dx%dx%d \n ",nGPUs,NX,NY,NZ);
	// List properties of each device
	displayDeviceProps(nGPUs);
	
//=====================================================================================================
// Allocate Memory 
//=====================================================================================================
	splitData(nGPUs, &gpu);

	// Variables declared in "declare.h"
	// Allocate memory for variables
	allocate_memory();
	printf("Made it past allocation!\n");

  // Create plans for cuFFT on each GPU
  plan1dFFT(nGPUs, fft);
  plan2dFFT(gpu, fft);

	// Declare variables
	int c = 0;
	int euler = 0;
	double time=0.0;
	double steptime=0.0;
	
//=======================================================================================================
// Initialize simulation
//=======================================================================================================

	// printf("Starting Timer...\n");
	// StartTimer();

	// Setup wavespace domain
	initializeWaveNumbers(gpu, k);

	// Launch CUDA kernel to initialize velocity field
	importData(gpu, h_vel, vel);
	
	// initializeData(gpu, fft, vel);
	// initializeJet_Superposition(fft, gpu, k, h_vel, vel, rhs);	// Does not require importData
	// initializeJet_Convolution(fft, gpu, h_vel, vel, rhs);  // Does not require importData

	// Save Initial Data to file (t = 0)
	// Copy data to host   
	save3Dfields(c, fft, gpu, h_vel, vel);

	// Transform velocity to fourier space for timestepping
	forwardTransform(fft, gpu, vel.u);
	forwardTransform(fft, gpu, vel.v);
	forwardTransform(fft, gpu, vel.w);
	forwardTransform(fft, gpu, vel.s);

	// Calculate statistics at initial condition
	calcTurbStats_mgpu(0, gpu, fft, k, vel, stats);

	// Dealias the solution by truncating RHS
	deAlias(gpu, k, vel);

	// Synchronize GPUs before entering timestepping loop
	synchronizeGPUs(nGPUs);

	// Print statistics to screen
	int stats_count = 0;
	printTurbStats(stats_count,0.0,stats);
	stats_count += 1;

	// Start iteration timer
	// StartTimer();

//==================================================================================================
// Enter time-stepping loop
//==================================================================================================
	for ( c = 1; c <= nt; ++c ){
		// Start iteration timer
		StartTimer();

		// Create flags to specify Euler timesteps

		if (c == 1){
			euler = 1;
		}
		else{
			euler = 0;
		}

		// Call pseudospectral Navier-Stokes solver
		solver_ps(euler, fft, gpu, vel, rhs, rhs_old, k, temp_advective);

		//==============================================================================================
		// Calculate bulk turbulence statistics and print to screen
		//==============================================================================================
		if(c % n_stats == 0){
			calcTurbStats_mgpu(stats_count, gpu, fft, k, vel, stats);
			// Get elapsed time from Timer
			steptime = GetTimer();
		
			// Print statistics to screen
			printTurbStats(stats_count,steptime,stats);
			stats_count += 1;
		}

		if(c % n_save2D == 0){
			// save2Dfield(c, fft, gpus, zhat, &h_vel->s);
		}

		// Synchronize GPUs before moving to next timestep
		synchronizeGPUs(nGPUs);

		// Save data to file every n_checkpoint timesteps
		if ( c % n_checkpoint == 0 ){
			save3Dfields(c, fft, gpu, h_vel, vel);
		}

	//===============================================================================================
	// End of Timestep
	//===============================================================================================
		steptime = GetTimer();
		time += steptime;
		if(c%n_stats!=0)
			printIterTime(c,steptime);
	}

//================================================================================================
// End of time stepping loop - save final results and clean up workspace variables
//================================================================================================
	printf("Total time elapsed: %2.2fs\n", time/1000);

	// Synchronize devices
	hipSetDevice(0);
	hipDeviceSynchronize();

	// Copy turbulent results from GPU to CPU memory
	// Make sure that the stats counter is equal to the number of data points being saved
	if(stats_count != nt/n_stats+1)
		printf("Error: Length of stats not equal to counter!!\n");
	// Save data to HDD
	saveStatsData(stats, h_stats);

	// Post-Simulation cleanup
	// Deallocate resources
	deallocate_memory();

	// Reset all GPUs
	for(n = 0; n<nGPUs; ++n){
		hipSetDevice(n);
		hipDeviceReset();
	}

	return 0;
}
