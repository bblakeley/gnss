#include "hip/hip_runtime.h"
// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <complex.h>

// includes, project
#include <hip/hip_runtime.h>
#include <>
#include <hipfft/hipfft.h>
#include <hip/hip_complex.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

// include parameters for DNS
#include "dnsparams.h"
#include "statistics.h"
#include "cudafuncs.h"
#include "fftfuncs.h"

#define RAD 1

/*
__global__
void surfaceIntegral_kernel(double *F, int w, int h, int d, double ref, double *Q, double *surfInt_Q) {
	extern __shared__ double s_F[];

	double dFdx, dFdy, dFdz, dchidx, dchidy, dchidz;

	// global indices
	const int i = blockIdx.x * blockDim.x + threadIdx.x; // column
	const int j = blockIdx.y * blockDim.y + threadIdx.y; // row
	const int k = blockIdx.z * blockDim.z + threadIdx.z; // stack
	if ((i >= w) || (j >= h) || (k >= d)) return;
	const int idx = flatten(i, j, k, w, h, d);
	// local width and height
	const int s_w = blockDim.x + 2 * RAD;
	const int s_h = blockDim.y + 2 * RAD;
	const int s_d = blockDim.z + 2 * RAD;
	// local indices
	const int s_i = threadIdx.x + RAD;
	const int s_j = threadIdx.y + RAD;
	const int s_k = threadIdx.z + RAD;
	const int s_idx = flatten(s_i, s_j, s_k, s_w, s_h, s_d);

	// Creating arrays in shared memory
	// Regular cells
	s_F[s_idx] = F[idx];

	//Halo Cells
	if (threadIdx.x < RAD) {
		s_F[flatten(s_i - RAD, s_j, s_k, s_w, s_h, s_d)] =
			F[flatten(i - RAD, j, k, w, h, d)];
		s_F[flatten(s_i + blockDim.x, s_j, s_k, s_w, s_h, s_d)] =
			F[flatten(i + blockDim.x, j, k, w, h, d)];
	}
	if (threadIdx.y < RAD) {
		s_F[flatten(s_i, s_j - RAD, s_k, s_w, s_h, s_d)] =
			F[flatten(i, j - RAD, k, w, h, d)];
		s_F[flatten(s_i, s_j + blockDim.y, s_k, s_w, s_h, s_d)] =
			F[flatten(i, j + blockDim.y, k, w, h, d)];
	}
	if (threadIdx.z < RAD) {
		s_F[flatten(s_i, s_j, s_k - RAD, s_w, s_h, s_d)] =
			F[flatten(i, j, k - RAD, w, h, d)];
		s_F[flatten(s_i, s_j, s_k + blockDim.z, s_w, s_h, s_d)] =
			F[flatten(i, j, k + blockDim.z, w, h, d)];
	}

	__syncthreads();

	// Boundary Conditions
	// Making problem boundaries periodic
	if (i == 0){
		s_F[flatten(s_i - 1, s_j, s_k, s_w, s_h, s_d)] = 
			F[flatten(w, j, k, w, h, d)];
	}
	if (i == w - 1){
		s_F[flatten(s_i + 1, s_j, s_k, s_w, s_h, s_d)] =
			F[flatten(0, j, k, w, h, d)];
	}

	if (j == 0){
		s_F[flatten(s_i, s_j - 1, s_k, s_w, s_h, s_d)] = 
			F[flatten(i, h, k, w, h, d)];
	}
	if (j == h - 1){
		s_F[flatten(s_i, s_j + 1, s_k, s_w, s_h, s_d)] =
			F[flatten(i, 0, k, w, h, d)];
	}

	if (k == 0){
		s_F[flatten(s_i, s_j, s_k - 1, s_w, s_h, s_d)] = 
			F[flatten(i, j, d, w, h, d)];
	}
	if (k == d - 1){
		s_F[flatten(s_i, s_j, s_k + 1, s_w, s_h, s_d)] =
			F[flatten(i, j, 0, w, h, d)];
	}

	__syncthreads();

	// Calculating dFdx and dFdy
	// Take derivatives

	dFdx = ( s_F[flatten(s_i + 1, s_j, s_k, s_w, s_h, s_d)] - 
		s_F[flatten(s_i - 1, s_j, s_k, s_w, s_h, s_d)] ) / (2.0*dx);

	dFdy = ( s_F[flatten(s_i, s_j + 1, s_k, s_w, s_h, s_d)] - 
		s_F[flatten(s_i, s_j - 1, s_k, s_w, s_h, s_d)] ) / (2.0*dx);

	dFdz = ( s_F[flatten(s_i, s_j, s_k + 1, s_w, s_h, s_d)] - 
		s_F[flatten(s_i, s_j, s_k - 1, s_w, s_h, s_d)] ) / (2.0*dx);

	__syncthreads();

	// Test to see if z is <= Zst, which sets the value of chi
	s_F[s_idx] = (s_F[s_idx] <= ref); 

	// Test Halo Cells to form chi
	if (threadIdx.x < RAD) {
		s_F[flatten(s_i - RAD, s_j, s_k, s_w, s_h, s_d)] = (s_F[flatten(s_i - RAD, s_j, s_k, s_w, s_h, s_d)] <= ref);
		s_F[flatten(s_i + blockDim.x, s_j, s_k, s_w, s_h, s_d)] = (s_F[flatten(s_i + blockDim.x, s_j, s_k, s_w, s_h, s_d)] <= ref);
	}
	if (threadIdx.y < RAD) {
		s_F[flatten(s_i, s_j - RAD, s_k, s_w, s_h, s_d)] = (s_F[flatten(s_i, s_j - RAD, s_k, s_w, s_h, s_d)] <= ref);
		s_F[flatten(s_i, s_j + blockDim.y, s_k, s_w, s_h, s_d)] = (s_F[flatten(s_i, s_j + blockDim.y, s_k, s_w, s_h, s_d)] <= ref);
	}
	if (threadIdx.z < RAD) {
		s_F[flatten(s_i, s_j, s_k - RAD, s_w, s_h, s_d)] = (s_F[flatten(s_i, s_j, s_k - RAD, s_w, s_h, s_d)] <= ref);
		s_F[flatten(s_i, s_j, s_k + blockDim.z, s_w, s_h, s_d)] = (s_F[flatten(s_i, s_j, s_k + blockDim.z, s_w, s_h, s_d)] <= ref);
	}

	__syncthreads();

	// Take derivatives
	dchidx = ( s_F[flatten(s_i + 1, s_j, s_k, s_w, s_h, s_d)] - 
		s_F[flatten(s_i - 1, s_j, s_k, s_w, s_h, s_d)] ) / (2.0*dx);

	dchidy = ( s_F[flatten(s_i, s_j + 1, s_k, s_w, s_h, s_d)] - 
		s_F[flatten(s_i, s_j - 1, s_k, s_w, s_h, s_d)] ) / (2.0*dx);
	
	dchidz = ( s_F[flatten(s_i, s_j, s_k + 1, s_w, s_h, s_d)] - 
		s_F[flatten(s_i, s_j, s_k - 1, s_w, s_h, s_d)] ) / (2.0*dx);

	__syncthreads();

	// Compute Length contribution for each thread
	if (dFdx == 0 && dFdy == 0 && dFdz == 0){
		s_F[s_idx] = 0.0;
	}
	else if (dchidx == 0 && dchidy == 0 && dchidz == 0){
		s_F[s_idx] = 0.0;
	}
	else{
		s_F[s_idx] = -Q[idx]*(dFdx * dchidx + dFdy * dchidy + dFdz * dchidz) / sqrtf(dFdx * dFdx + dFdy * dFdy + dFdz * dFdz);
	}

	// __syncthreads();

	// Add length contribution from each thread into block memory
	if (threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0){
		double local_Q = 0.0;
		for (int q = 1; q <= blockDim.x; ++q) {
			for (int r = 1; r <= blockDim.y; ++r){
				for (int s = 1; s <= blockDim.z; ++s){
					int local_idx = flatten(q, r, s, s_w, s_h, s_d);
					local_Q += s_F[local_idx];
				}
			}
		}
		__syncthreads();
		atomicAdd(surfInt_Q, local_Q*dx*dx*dx);
	}

	return;
}
*/
/*
__global__
void multIk(hipfftDoubleComplex *f, hipfftDoubleComplex *fIk, double *waveNum, const int dir)
{	// Function to multiply the function fhat by i*k
	const int i = blockIdx.x * blockDim.x + threadIdx.x;
	const int j = blockIdx.y * blockDim.y + threadIdx.y;
	const int k = blockIdx.z * blockDim.z + threadIdx.z;
	if ((i >= NX) || (j >= NY) || (k >= NZ2)) return;
	const int idx = flatten(i, j, k, NX, NY, NZ2);

	// i*k*(a + bi) = -k*b + i*k*a
	
// Create temporary variables to store real and complex parts
	double a = f[idx].x;
	double b = f[idx].y;

	if(dir == 1){ // Takes derivative in 1 direction (usually x)
		fIk[idx].x = -waveNum[i]*b/((double)NN);
		fIk[idx].y = waveNum[i]*a/((double)NN);
	}
	if(dir == 2){	// Takes derivative in 2 direction (usually y)
		fIk[idx].x = -waveNum[j]*b/((double)NN);
		fIk[idx].y = waveNum[j]*a/((double)NN);
	}
	if(dir == 3){
		fIk[idx].x = -waveNum[k]*b/((double)NN);
		fIk[idx].y = waveNum[k]*a/((double)NN);
	}

	return;
}


// __global__
// void multIk_inplace(hipfftDoubleComplex *f, double *waveNum, const int dir)
// {	// Function to multiply the function fhat by i*k
// 	const int i = blockIdx.x * blockDim.x + threadIdx.x;
// 	const int j = blockIdx.y * blockDim.y + threadIdx.y;
// 	const int k = blockIdx.z * blockDim.z + threadIdx.z;
// 	if ((i >= NX) || (j >= NY) || (k >= NZ2)) return;
// 	const int idx = flatten(i, j, k, NX, NY, NZ2);

// 	// i*k*(a + bi) = -k*b + i*k*a
	
// // Create temporary variables to store real and complex parts
// 	double a = f[idx].x;
// 	double b = f[idx].y;

// 	if(dir == 1){ // Takes derivative in 1 direction (usually x)
// 		f[idx].x = -waveNum[i]*b/((double)NN);
// 		f[idx].y = waveNum[i]*a/((double)NN);
// 	}
// 	if(dir == 2){	// Takes derivative in 2 direction (usually y)
// 		f[idx].x = -waveNum[j]*b/((double)NN);
// 		f[idx].y = waveNum[j]*a/((double)NN);
// 	}
// 	if(dir == 3){
// 		f[idx].x = -waveNum[k]*b/((double)NN);
// 		f[idx].y = waveNum[k]*a/((double)NN);
// 	}

// 	return;
// }

__global__
void multIk2(hipfftDoubleComplex *f, hipfftDoubleComplex *fIk2, double *waveNum, const int dir)
{	// Function to multiply the function fhat by i*k
	const int i = blockIdx.x * blockDim.x + threadIdx.x;
	const int j = blockIdx.y * blockDim.y + threadIdx.y;
	const int k = blockIdx.z * blockDim.z + threadIdx.z;
	if ((i >= NX) || (j >= NY) || (k >= NZ2)) return;
	const int idx = flatten(i, j, k, NX, NY, NZ2);

	// i*k*(a + bi) = -k*b + i*k*a

	if(dir == 1){ // Takes derivative in 1 direction (usually x)
		fIk2[idx].x = -waveNum[i]*waveNum[i]*f[idx].x/((double)NN);
		fIk2[idx].y = -waveNum[i]*waveNum[i]*f[idx].y/((double)NN);
	}
	if(dir == 2){	// Takes derivative in 2 direction (usually y)
		fIk2[idx].x = -waveNum[j]*waveNum[j]*f[idx].x/((double)NN);
		fIk2[idx].y = -waveNum[j]*waveNum[j]*f[idx].y/((double)NN);
	}
	if(dir == 3){
		fIk2[idx].x = -waveNum[k]*waveNum[k]*f[idx].x/((double)NN);
		fIk2[idx].y = -waveNum[k]*waveNum[k]*f[idx].y/((double)NN);
	}

	return;
}


__global__
void magnitude(hipfftDoubleReal *f1, hipfftDoubleReal *f2, hipfftDoubleReal *f3, hipfftDoubleReal *mag){
	// Function to calculate the magnitude of a 3D vector field

	const int i = blockIdx.x * blockDim.x + threadIdx.x;
	const int j = blockIdx.y * blockDim.y + threadIdx.y;
	const int k = blockIdx.z * blockDim.z + threadIdx.z;
	if ((i >= NX) || (j >= NY) || (k >= NZ)) return;
	const int idx = flatten(i, j, k, NX, NY, NZ);

	// Magnitude of a 3d vector field = sqrt(f1^2 + f2^2 + f3^2)

	mag[idx] = sqrt(f1[idx]*f1[idx] + f2[idx]*f2[idx] + f3[idx]*f3[idx]);

	return;

}

__global__
void mult3AndAdd(hipfftDoubleReal *f1, hipfftDoubleReal *f2, hipfftDoubleReal *f3, hipfftDoubleReal *f4, const int flag)
{	// Function to multiply 3 functions and add (or subtract) the result to a 4th function

	const int i = blockIdx.x * blockDim.x + threadIdx.x;
	const int j = blockIdx.y * blockDim.y + threadIdx.y;
	const int k = blockIdx.z * blockDim.z + threadIdx.z;
	if ((i >= NX) || (j >= NY) || (k >= NZ)) return;
	const int idx = flatten(i, j, k, NX, NY, NZ);

	if ( flag == 1 ){
		f4[idx] = f4[idx] + f1[idx]*f2[idx]*f3[idx];
	}
	else if ( flag == 0 ){
		f4[idx] = f4[idx] - f1[idx]*f2[idx]*f3[idx];
	}
	else{
		printf("Multipy and Add function failed: please designate 1 (plus) or 0 (minus).\n");
	}
		
		return;
}

__global__
void mult2AndAdd(hipfftDoubleReal *f1, hipfftDoubleReal *f2, hipfftDoubleReal *f3, const int flag)
{	// Function to multiply 3 functions and add (or subtract) the result to a 4th function

	const int i = blockIdx.x * blockDim.x + threadIdx.x;
	const int j = blockIdx.y * blockDim.y + threadIdx.y;
	const int k = blockIdx.z * blockDim.z + threadIdx.z;
	if ((i >= NX) || (j >= NY) || (k >= NZ)) return;
	const int idx = flatten(i, j, k, NX, NY, NZ);

	if ( flag == 1 ){
		f3[idx] = f3[idx] + f1[idx]*f2[idx];
	}
	else if ( flag == 0 ){
		f3[idx] = f3[idx] - f1[idx]*f2[idx];
	}
	else{
		printf("Multipy and Add function failed: please designate 1 (plus) or 0 (minus).\n");
	}
		
		return;
}

__global__
void multiplyOrDivide(hipfftDoubleReal *f1, hipfftDoubleReal *f2, hipfftDoubleReal *f3, const int flag){
	// This function either multiplies two functions or divides two functions, depending on which flag is passed. The output is stored in the first array passed to the function.

	const int i = blockIdx.x * blockDim.x + threadIdx.x;
	const int j = blockIdx.y * blockDim.y + threadIdx.y;
	const int k = blockIdx.z * blockDim.z + threadIdx.z;
	if ((i >= NX) || (j >= NY) || (k >= NZ)) return;
	const int idx = flatten(i, j, k, NX, NY, NZ);

	if ( flag == 1 ){
		f3[idx] = f1[idx]*f2[idx];
	}
	else if ( flag == 0 ){
		f3[idx] = f1[idx]/f2[idx];
	}
	else{
		printf("Multipy or Divide function failed: please designate 1 (multiply) or 0 (divide).\n");
	}

	return;
}

__global__
void calcTermIV_kernel(hipfftDoubleReal *gradZ, hipfftDoubleReal *IV){

	const int i = blockIdx.x * blockDim.x + threadIdx.x;
	const int j = blockIdx.y * blockDim.y + threadIdx.y;
	const int k = blockIdx.z * blockDim.z + threadIdx.z;
	if ((i >= NX) || (j >= NY) || (k >= NZ)) return;
	const int idx = flatten(i, j, k, NX, NY, NZ);

	IV[idx] = 1.0/(gradZ[idx]*gradZ[idx])*IV[idx];
		
	return;

}

void calcTermIV(hipfftHandle p, hipfftHandle invp, double *k, hipfftDoubleReal *u, hipfftDoubleReal *v, hipfftDoubleReal *w, hipfftDoubleReal *s, double *T4){
// Function to calculate the 4th term at each grid point in the dSigmadt equation
	//  The equation for Term IV is:
	// IV = -( nx*nx*dudx + nx*ny*dudy + nx*nz*dudz + ny*nx*dvdx + ny*ny*dvdy ...
	//  		+ ny*nz*dvdz  + nz*nx*dwdx + nz*ny*dwdy + nz*nz*dwdz), 
	// where nx = -dsdx/grads, ny = -dsdy/grads, nz = -dsdz/grads,
	//  and grads = sqrt(dsdx^2 + dsdy^2 + dsdz^2).
	

	// Allocate temporary variables
	hipfftDoubleReal *dsdx, *dsdy, *dsdz, *grads;
	hipfftDoubleComplex *temp_c;

	// hipfftResult result;

	hipMallocManaged(&dsdx, sizeof(hipfftDoubleReal)*NN);
	hipMallocManaged(&dsdy, sizeof(hipfftDoubleReal)*NN);
	hipMallocManaged(&dsdz, sizeof(hipfftDoubleReal)*NN);
	hipMallocManaged(&grads, sizeof(hipfftDoubleReal)*NN);		// Variable to hold the magnitude of gradient of s as well as other temporary variables
	hipMallocManaged(&temp_c, sizeof(hipfftDoubleComplex)*NX*NY*NZ2);

	// Set kernel variables
	const dim3 blockSize(TX, TY, TZ);
	const dim3 gridSize(divUp(NX, TX), divUp(NY, TY), divUp(NZ, TZ));

// Initialize T4 to zero
	hipMemset(T4, 0.0, sizeof(double)*NX*NY*NZ);

// Calculate derivatives of scalar field
	// dsdx
	fftDer(p, invp, k, s, temp_c, dsdx, 1);
	// dsdy
	fftDer(p, invp, k, s, temp_c, dsdy, 2);
	// dsdz
	fftDer(p, invp, k, s, temp_c, dsdz, 3);

	// Approach: calculate each of the 9 required terms for Term IV separately and add them to the running total

// 1st term: nx*nx*dudx
	// Take derivative to get dudx
	fftDer(p, invp, k, u, temp_c, grads, 1);
	// Multiply by nx*nx and add to Term IV
	mult3AndAdd<<<gridSize, blockSize>>>(dsdx, dsdx, grads, T4, 0);

// 2nd term: nx*ny*dudy
	// Take derivative to get dudy
	fftDer(p, invp, k, u, temp_c, grads, 2);
	// Multiply by nx*ny and add to Term IV
	mult3AndAdd<<<gridSize, blockSize>>>(dsdx, dsdy, grads, T4, 0);

// 3rd term: nx*nz*dudz
	// Take derivative to get dudz
	fftDer(p, invp, k, u, temp_c, grads, 3);
	// Multiply by nx*nz and add to Term IV
	mult3AndAdd<<<gridSize, blockSize>>>(dsdx, dsdz, grads, T4, 0);

// 4th term: ny*nx*dvdx
	// Take derivative to get dvdx
	fftDer(p, invp, k, v, temp_c, grads, 1);
	// Multiply by ny*nx and add to Term IV
	mult3AndAdd<<<gridSize, blockSize>>>(dsdy, dsdx, grads, T4, 0);

// 5th term: ny*ny*dvdy
	// Take derivative to get dvdy
	fftDer(p, invp, k, v, temp_c, grads, 2);
	// Multiply by ny*ny and add to Term IV
	mult3AndAdd<<<gridSize, blockSize>>>(dsdy, dsdy, grads, T4, 0);

// 6th term: ny*nz*dvdz
	// Take derivative to get dvdz
	fftDer(p, invp, k, v, temp_c, grads, 3);
	// Multiply by ny*nz and add to Term IV
	mult3AndAdd<<<gridSize, blockSize>>>(dsdy, dsdz, grads, T4, 0);

// 7th term: nz*nx*dwdx
	// Take derivative to get dwdx
	fftDer(p, invp, k, w, temp_c, grads, 1);
	// Multiply by nz*nx and add to Term IV
	mult3AndAdd<<<gridSize, blockSize>>>(dsdz, dsdx, grads, T4, 0);

// 8th term: nz*ny*dwdy
	// Take derivative to get dwdy
	fftDer(p, invp, k, w, temp_c, grads, 2);
	// Multiply by nz*ny and add to Term IV
	mult3AndAdd<<<gridSize, blockSize>>>(dsdz, dsdy, grads, T4, 0);

// 9th term: nz*nz*dwdz
	// Take derivative to get dwdz
	fftDer(p, invp, k, w, temp_c, grads, 3);
	// Multiply by nz*nz and add to Term IV
	mult3AndAdd<<<gridSize, blockSize>>>(dsdz, dsdz, grads, T4, 0);

// Calculate grads
	magnitude<<<gridSize, blockSize>>>(dsdx, dsdy, dsdz, grads);

// Divide The sum of terms in T4 by grads^2
	calcTermIV_kernel<<<gridSize, blockSize>>>(grads, T4);

	hipFree(dsdx);
	hipFree(dsdy);
	hipFree(dsdz);
	hipFree(grads);
	hipFree(temp_c);

	return;
}

__global__
void sum_kernel(hipfftDoubleReal *f1, hipfftDoubleReal *f2, hipfftDoubleReal *f3, const int flag){
	// This kernel adds three functions, storing the result in the first array that was passed to it
	
	const int i = blockIdx.x * blockDim.x + threadIdx.x;
	const int j = blockIdx.y * blockDim.y + threadIdx.y;
	const int k = blockIdx.z * blockDim.z + threadIdx.z;
	if ((i >= NX) || (j >= NY) || (k >= NZ)) return;
	const int idx = flatten(i, j, k, NX, NY, NZ);

	if ( flag == 1 ){
		f3[idx] = f1[idx] + f2[idx];
	}
	else if ( flag == 0 ){
		f3[idx] = f1[idx] - f2[idx];
	}
	else{
		printf("Sum kernel function failed: please designate 1 (add) or 0 (subtract).\n");
	}

	return;
}

__global__
void calcDiffusionVelocity_kernel(const double D, hipfftDoubleReal *lapl_s, hipfftDoubleReal *grads, hipfftDoubleReal *diff_Vel){
// Function to calculate the diffusion velocity, given the diffusion coefficient, the laplacian of the scalar field, and the magnitude of the gradient of the scalar field
// The result of this is stored in the array holding |grads|
	const int i = blockIdx.x * blockDim.x + threadIdx.x;
	const int j = blockIdx.y * blockDim.y + threadIdx.y;
	const int k = blockIdx.z * blockDim.z + threadIdx.z;
	if ((i >= NX) || (j >= NY) || (k >= NZ)) return;
	const int idx = flatten(i, j, k, NX, NY, NZ);

	diff_Vel[idx] = D*lapl_s[idx]/grads[idx];

	return;
}

void calcTermV(hipfftHandle p, hipfftHandle invp, double *waveNum, hipfftDoubleReal *s, hipfftDoubleReal *T5){
// Function to calculate the 5th term at each grid point in the dSigmadt equation
	//  The equation for Term V is:
	// V = -D*(dsdx2 + dsdy2 + dsdz2)/|grads| * ...
	//  		(d/dx(-nx) + d/dy(-nx) + d/dz(-nx), 
	// where nx = -dsdx/|grads|, ny = -dsdy/grads, nz = -dsdz/grads,
	//  and grads = sqrt(dsdx^2 + dsdy^2 + dsdz^2).
	

	// Allocate temporary variables
	hipfftDoubleReal *dsdx, *dsdy, *dsdz;
	hipfftDoubleComplex *temp_c;

	// hipfftResult result;

	hipMallocManaged(&dsdx, sizeof(hipfftDoubleReal)*NN);
	hipMallocManaged(&dsdy, sizeof(hipfftDoubleReal)*NN);
	hipMallocManaged(&dsdz, sizeof(hipfftDoubleReal)*NN);
	hipMallocManaged(&temp_c, sizeof(hipfftDoubleComplex)*NX*NY*NZ2);

	// Set kernel variables
	const dim3 blockSize(TX, TY, TZ);
	const dim3 gridSize(divUp(NX, TX), divUp(NY, TY), divUp(NZ, TZ));

// Calculate derivatives of scalar field
	// dsdx
	fftDer(p, invp, waveNum, s, temp_c, dsdx, 1);
	// dsdy
	fftDer(p, invp, waveNum, s, temp_c, dsdy, 2);
	// dsdz
	fftDer(p, invp, waveNum, s, temp_c, dsdz, 3);

// Calculate grads
	magnitude<<<gridSize, blockSize>>>(dsdx, dsdy, dsdz, T5);

// Calculate normal vectors
	// Divide dsdx by |grads|
	multiplyOrDivide<<<gridSize, blockSize>>>(dsdx, T5, dsdx, 0);
	// Divide dsdy by |grads|
	multiplyOrDivide<<<gridSize, blockSize>>>(dsdy, T5, dsdy, 0);
	// Divide dsdz by |grads|
	multiplyOrDivide<<<gridSize, blockSize>>>(dsdz, T5, dsdz, 0);

// Take derivative of normal vectors 
	fftDer(p, invp, waveNum, dsdx, temp_c, dsdx, 1);
	fftDer(p, invp, waveNum, dsdy, temp_c, dsdy, 2);
	fftDer(p, invp, waveNum, dsdz, temp_c, dsdz, 3);

// Sum the derivatives of normal vectors together to form divergence(n)
	sum_kernel<<<gridSize, blockSize>>>(dsdx, dsdy, dsdx, 1);
	sum_kernel<<<gridSize, blockSize>>>(dsdx, dsdz, dsdx, 1);			// dsdx is holding the divergence of the normal vector

// Form Laplacian(s)
	// Take second derivative of scalar field in the x direction - the Laplacian will be stored in dsdy
	fft2ndDer(p, invp, waveNum, s, temp_c, dsdy, 1);		// dsdy is a placeholder variable only - don't pay attention to the name!
	
	// Take second derivative in y direction
	fft2ndDer(p, invp, waveNum, s, temp_c, dsdz, 2);		// dsdz is also a temporary placeholder
	// Add the 2nd y derivative of s to the Laplacian term (stored in dsdy)
	sum_kernel<<<gridSize, blockSize>>>(dsdy, dsdz, dsdy, 1);
	
	// Take the second derivative in the z direction
	fft2ndDer(p, invp, waveNum, s, temp_c, dsdz, 3);
	// Add the 2nd z derivative of s to the Laplacian term (stored in dsdy)
	sum_kernel<<<gridSize, blockSize>>>(dsdy, dsdz, dsdy, 1);

// Calculate the diffusion velocity
	calcDiffusionVelocity_kernel<<<gridSize, blockSize>>>(-nu/((double)Sc), dsdy, T5, T5);

// Calculate Term V
	multiplyOrDivide<<<gridSize, blockSize>>>(T5, dsdx, T5, 1);

	hipFree(dsdx);
	hipFree(dsdy);
	hipFree(dsdz);
	hipFree(temp_c);

	return;
}

__global__
void calcTermVa_kernel(const double D, hipfftDoubleReal *div_n, hipfftDoubleReal *Va){
// Function to calculate the diffusion velocity, given the diffusion coefficient, the laplacian of the scalar field, and the magnitude of the gradient of the scalar field
// The result of this is stored in the array holding |grads|
	const int i = blockIdx.x * blockDim.x + threadIdx.x;
	const int j = blockIdx.y * blockDim.y + threadIdx.y;
	const int k = blockIdx.z * blockDim.z + threadIdx.z;
	if ((i >= NX) || (j >= NY) || (k >= NZ)) return;
	const int idx = flatten(i, j, k, NX, NY, NZ);

	Va[idx] = -D*div_n[idx]*div_n[idx];

	return;
}

void calcTermVa(hipfftHandle p, hipfftHandle invp, double *waveNum, hipfftDoubleReal *s, hipfftDoubleReal *T5a){
// Function to calculate the decomposition of the 5th term at each grid point in the dSigmadt equation
	//  The equation for Term Va is:
	// Va = -D*(divergence(n))^2, 
	// where n = -dsdx/|grads|,
	

	// Allocate temporary variables
	hipfftDoubleReal *dsdx, *dsdy, *dsdz;
	hipfftDoubleComplex *temp_c;

	// hipfftResult result;

	hipMallocManaged(&dsdx, sizeof(hipfftDoubleReal)*NN);
	hipMallocManaged(&dsdy, sizeof(hipfftDoubleReal)*NN);
	hipMallocManaged(&dsdz, sizeof(hipfftDoubleReal)*NN);
	hipMallocManaged(&temp_c, sizeof(hipfftDoubleComplex)*NX*NY*NZ2);

	// Set kernel variables
	const dim3 blockSize(TX, TY, TZ);
	const dim3 gridSize(divUp(NX, TX), divUp(NY, TY), divUp(NZ, TZ));

// Calculate derivatives of scalar field
	// dsdx
	fftDer(p, invp, waveNum, s, temp_c, dsdx, 1);
	// dsdy
	fftDer(p, invp, waveNum, s, temp_c, dsdy, 2);
	// dsdz
	fftDer(p, invp, waveNum, s, temp_c, dsdz, 3);

// Calculate grads
	magnitude<<<gridSize, blockSize>>>(dsdx, dsdy, dsdz, T5a);

// Calculate normal vectors
	// Divide dsdx by |grads|
	multiplyOrDivide<<<gridSize, blockSize>>>(dsdx, T5a, dsdx, 0);
	// Divide dsdy by |grads|
	multiplyOrDivide<<<gridSize, blockSize>>>(dsdy, T5a, dsdy, 0);
	// Divide dsdz by |grads|
	multiplyOrDivide<<<gridSize, blockSize>>>(dsdz, T5a, dsdz, 0);

// Take derivative of normal vectors 
	fftDer(p, invp, waveNum, dsdx, temp_c, dsdx, 1);
	fftDer(p, invp, waveNum, dsdy, temp_c, dsdy, 2);
	fftDer(p, invp, waveNum, dsdz, temp_c, dsdz, 3);

// Zero out T5a
	hipMemset(T5a, 0.0, sizeof(double)*NN);

// Sum the derivatives of normal vectors together to form divergence(n)
	sum_kernel<<<gridSize, blockSize>>>(T5a, dsdx, T5a, 1);
	sum_kernel<<<gridSize, blockSize>>>(T5a, dsdy, T5a, 1);
	sum_kernel<<<gridSize, blockSize>>>(T5a, dsdz, T5a, 1);			// T5a is now holding the divergence of the normal vector

// Calculate Term Va
	calcTermVa_kernel<<<gridSize, blockSize>>>(nu/((double)Sc), T5a, T5a);

	hipFree(dsdx);
	hipFree(dsdy);
	hipFree(dsdz);
	hipFree(temp_c);

	return;
}

__global__
void calcTermVb_kernel(const double D, hipfftDoubleReal *Numerator, hipfftDoubleReal *gradZ, hipfftDoubleReal *div_n, hipfftDoubleReal *Vb){

	const int i = blockIdx.x * blockDim.x + threadIdx.x;
	const int j = blockIdx.y * blockDim.y + threadIdx.y;
	const int k = blockIdx.z * blockDim.z + threadIdx.z;
	if ((i >= NX) || (j >= NY) || (k >= NZ)) return;
	const int idx = flatten(i, j, k, NX, NY, NZ);

	Vb[idx] = -D*Numerator[idx]/(gradZ[idx]*gradZ[idx])*div_n[idx];
		
	return;

}

void calcTermVb(hipfftHandle p, hipfftHandle invp, double *waveNum, hipfftDoubleReal *s, hipfftDoubleReal *T5b){
// Function to calculate the decomposition of the 5th term at each grid point in the dSigmadt equation
	//  The equation for Term Va is:
	// Va = -D*(divergence(n))^2, 
	// where n = -dsdx/|grads|,
	

	// Allocate temporary variables
	hipfftDoubleReal *dsdx, *dsdy, *dsdz, *grads;
	hipfftDoubleComplex *temp_c;

	// hipfftResult result;

	hipMallocManaged(&dsdx, sizeof(hipfftDoubleReal)*NN);
	hipMallocManaged(&dsdy, sizeof(hipfftDoubleReal)*NN);
	hipMallocManaged(&dsdz, sizeof(hipfftDoubleReal)*NN);
	hipMallocManaged(&grads, sizeof(hipfftDoubleReal)*NN);
	hipMallocManaged(&temp_c, sizeof(hipfftDoubleComplex)*NX*NY*NZ2);		// Temporary variable that is passed to the fft derivative function for intermediate calculations

	// Set kernel variables
	const dim3 blockSize(TX, TY, TZ);
	const dim3 gridSize(divUp(NX, TX), divUp(NY, TY), divUp(NZ, TZ));

///////////////////////////////////////////
	//Step 1: Calculate divergence of the normal vector
// Calculate derivatives of scalar field
	// dsdx
	fftDer(p, invp, waveNum, s, temp_c, dsdx, 1);
	// dsdy
	fftDer(p, invp, waveNum, s, temp_c, dsdy, 2);
	// dsdz
	fftDer(p, invp, waveNum, s, temp_c, dsdz, 3);

// Calculate grads
	magnitude<<<gridSize, blockSize>>>(dsdx, dsdy, dsdz, T5b);		// T5b now holds |grads|

// Calculate normal vectors
	// Divide dsdx by |grads|
	multiplyOrDivide<<<gridSize, blockSize>>>(dsdx, T5b, dsdx, 0);
	// Divide dsdy by |grads|
	multiplyOrDivide<<<gridSize, blockSize>>>(dsdy, T5b, dsdy, 0);
	// Divide dsdz by |grads|
	multiplyOrDivide<<<gridSize, blockSize>>>(dsdz, T5b, dsdz, 0);

// Take derivative of normal vectors 
	fftDer(p, invp, waveNum, dsdx, temp_c, dsdx, 1);
	fftDer(p, invp, waveNum, dsdy, temp_c, dsdy, 2);
	fftDer(p, invp, waveNum, dsdz, temp_c, dsdz, 3);

// Zero out T5a
	hipMemset(T5b, 0.0, sizeof(double)*NN);

// Sum the derivatives of normal vectors together to form divergence(n)
	sum_kernel<<<gridSize, blockSize>>>(T5b, dsdx, T5b, 1);
	sum_kernel<<<gridSize, blockSize>>>(T5b, dsdy, T5b, 1);
	sum_kernel<<<gridSize, blockSize>>>(T5b, dsdz, T5b, 1);			// T5b is now holding the divergence of the normal vector

//////////////////////////////////////////////////////////////
	//Step 2: Calculate the numerator, grads*gradient(grads)
// Calculate |grads|
	// dsdx
	fftDer(p, invp, waveNum, s, temp_c, dsdx, 1);
	// dsdy
	fftDer(p, invp, waveNum, s, temp_c, dsdy, 2);
	// dsdz
	fftDer(p, invp, waveNum, s, temp_c, dsdz, 3);

// Calculate grads
	magnitude<<<gridSize, blockSize>>>(dsdx, dsdy, dsdz, grads);		// grads now holds |grads|

// Find the x derivative of |grads|
	fftDer(p, invp, waveNum, grads, temp_c, dsdz, 1);		// dsdz temporarily holds x derivative of |grads|
// Multiply dsdx and x derivative of |grads| and add to intermediate variable
	mult2AndAdd<<<gridSize, blockSize>>>(dsdx, dsdz, dsdx, 1);		// dsdx holds the current sum for this term

// Find the y derivative of |grads|
	fftDer(p, invp, waveNum, grads, temp_c, dsdz, 2);
// Multiply dsdy and y derivative of |grads| and add to intermediate variable
	mult2AndAdd<<<gridSize, blockSize>>>(dsdy, dsdz, dsdx, 1);

// Calculate dsdz
	fftDer(p, invp, waveNum, s, temp_c, dsdz, 3);			// Need to recalculate dsdz because the variable was used as a placeholder above
// Find the z derivative of |grads|
	fftDer(p, invp, waveNum, grads, temp_c, dsdy, 3);		// dsdy used as a placeholder for z derivative of |grads|
// Multiply dsdy and y derivative of |grads| and add to intermediate variable
	mult2AndAdd<<<gridSize, blockSize>>>(dsdy, dsdz, dsdx, 1);		// Multiplies dsdz and z derivative of |grads| and stores in dsdx variable

////////////////////////////////////////////////////////////////
	// Calculate Term Vb
	calcTermVb_kernel<<<gridSize, blockSize>>>(nu/((double)Sc), dsdx, grads, T5b, T5b);

	hipFree(dsdx);
	hipFree(dsdy);
	hipFree(dsdz);
	hipFree(grads);
	hipFree(temp_c);

	return;
}

void calcSurfaceProps(hipfftHandle p, hipfftHandle invp, double *waveNum, hipfftDoubleReal *u, hipfftDoubleReal *v, hipfftDoubleReal *w, hipfftDoubleReal *z, double Zst, double *SA, double *T4, double *T5, double *T5a, double *T5b){
// Function to calculate surface quantities

	// Declare and allocate temporary variables
	double *temp;
	hipMallocManaged(&temp, sizeof(double)*NN);

	const dim3 blockSize(TX, TY, TZ);
	const dim3 gridSize(divUp(NX, TX), divUp(NY, TY), divUp(NZ, TZ));
	const size_t smemSize = (TX + 2*RAD)*(TY + 2*RAD)*(TZ + 2*RAD)*sizeof(double);

// Calculate surface area based on Zst
	surfaceArea_kernel<<<gridSize, blockSize, smemSize>>>(z, NX, NY, NZ, Zst, SA);
	hipError_t err = hipGetLastError();
	if (err != hipSuccess) 
    printf("Error: %s\n", hipGetErrorString(err));

// Calculate Term IV
	calcTermIV(p, invp, waveNum, u, v, w, z, temp);

	// Integrate TermIV over the flame surface (Refer to Mete's thesis for more info on the surface integration technique)
	surfaceIntegral_kernel<<<gridSize, blockSize, smemSize>>>(z, NX, NY, NZ, Zst, temp, T4);
	err = hipGetLastError();
	if (err != hipSuccess) 
    printf("Error: %s\n", hipGetErrorString(err));

	hipDeviceSynchronize();

// Calculate Term V
	calcTermV(p, invp, waveNum, z, temp);

	// Integrate TermV over the flame surface (Refer to Mete's thesis for more info on the surface integration technique)
	surfaceIntegral_kernel<<<gridSize, blockSize, smemSize>>>(z, NX, NY, NZ, Zst, temp, T5);
	err = hipGetLastError();
	if (err != hipSuccess) 
    printf("Error: %s\n", hipGetErrorString(err));

	hipDeviceSynchronize();

// Calculate Term Va
	calcTermVa(p, invp, waveNum, z, temp);

	// Integrate TermV over the flame surface (Refer to Mete's thesis for more info on the surface integration technique)
	surfaceIntegral_kernel<<<gridSize, blockSize, smemSize>>>(z, NX, NY, NZ, Zst, temp, T5a);
	err = hipGetLastError();
	if (err != hipSuccess) 
    printf("Error: %s\n", hipGetErrorString(err));

	hipDeviceSynchronize();

// Calculate Term Vb
	calcTermVb(p, invp, waveNum, z, temp);

	// Integrate TermV over the flame surface (Refer to Mete's thesis for more info on the surface integration technique)
	surfaceIntegral_kernel<<<gridSize, blockSize, smemSize>>>(z, NX, NY, NZ, Zst, temp, T5b);
	err = hipGetLastError();
	if (err != hipSuccess) 
    printf("Error: %s\n", hipGetErrorString(err));

	hipDeviceSynchronize();

	//Post-processing
	T4[0] = T4[0]/SA[0];
	T5[0] = T5[0]/SA[0];
	T5a[0] = T5a[0]/SA[0];
	T5b[0] = T5b[0]/SA[0];

	hipFree(temp);

}
*/

// __global__
// void surfaceArea_kernel(double *F, int w, int h, int d, double ref, double *SA) {
// 	extern __shared__ double s_F[];

// 	double dFdx, dFdy, dFdz, dchidx, dchidy, dchidz;

// 	// global indices
// 	const int i = blockIdx.x * blockDim.x + threadIdx.x; // column
// 	const int j = blockIdx.y * blockDim.y + threadIdx.y; // row
// 	const int k = blockIdx.z * blockDim.z + threadIdx.z; // stack
// 	if ((i >= w) || (j >= h) || (k >= d)) return;
// 	const int idx = flatten(i, j, k, w, h, d);
// 	// local width and height
// 	const int s_w = blockDim.x + 2 * RAD;
// 	const int s_h = blockDim.y + 2 * RAD;
// 	const int s_d = blockDim.z + 2 * RAD;
// 	// local indices
// 	const int s_i = threadIdx.x + RAD;
// 	const int s_j = threadIdx.y + RAD;
// 	const int s_k = threadIdx.z + RAD;
// 	const int s_idx = flatten(s_i, s_j, s_k, s_w, s_h, s_d);

// 	// Creating arrays in shared memory
// 	// Regular cells
// 	s_F[s_idx] = F[idx];

// 	//Halo Cells
// 	if (threadIdx.x < RAD) {
// 		s_F[flatten(s_i - RAD, s_j, s_k, s_w, s_h, s_d)] =
// 			F[flatten(i - RAD, j, k, w, h, d)];
// 		s_F[flatten(s_i + blockDim.x, s_j, s_k, s_w, s_h, s_d)] =
// 			F[flatten(i + blockDim.x, j, k, w, h, d)];
// 	}
// 	if (threadIdx.y < RAD) {
// 		s_F[flatten(s_i, s_j - RAD, s_k, s_w, s_h, s_d)] =
// 			F[flatten(i, j - RAD, k, w, h, d)];
// 		s_F[flatten(s_i, s_j + blockDim.y, s_k, s_w, s_h, s_d)] =
// 			F[flatten(i, j + blockDim.y, k, w, h, d)];
// 	}
// 	if (threadIdx.z < RAD) {
// 		s_F[flatten(s_i, s_j, s_k - RAD, s_w, s_h, s_d)] =
// 			F[flatten(i, j, k - RAD, w, h, d)];
// 		s_F[flatten(s_i, s_j, s_k + blockDim.z, s_w, s_h, s_d)] =
// 			F[flatten(i, j, k + blockDim.z, w, h, d)];
// 	}

// 	__syncthreads();

// 	// Boundary Conditions
// 	// Making problem boundaries periodic
// 	if (i == 0){
// 		s_F[flatten(s_i - 1, s_j, s_k, s_w, s_h, s_d)] = 
// 			F[flatten(w, j, k, w, h, d)];
// 	}
// 	if (i == w - 1){
// 		s_F[flatten(s_i + 1, s_j, s_k, s_w, s_h, s_d)] =
// 			F[flatten(0, j, k, w, h, d)];
// 	}

// 	if (j == 0){
// 		s_F[flatten(s_i, s_j - 1, s_k, s_w, s_h, s_d)] = 
// 			F[flatten(i, h, k, w, h, d)];
// 	}
// 	if (j == h - 1){
// 		s_F[flatten(s_i, s_j + 1, s_k, s_w, s_h, s_d)] =
// 			F[flatten(i, 0, k, w, h, d)];
// 	}

// 	if (k == 0){
// 		s_F[flatten(s_i, s_j, s_k - 1, s_w, s_h, s_d)] = 
// 			F[flatten(i, j, d, w, h, d)];
// 	}
// 	if (k == d - 1){
// 		s_F[flatten(s_i, s_j, s_k + 1, s_w, s_h, s_d)] =
// 			F[flatten(i, j, 0, w, h, d)];
// 	}

// 	// __syncthreads();

// 	// Calculating dFdx and dFdy
// 	// Take derivatives

// 	dFdx = ( s_F[flatten(s_i + 1, s_j, s_k, s_w, s_h, s_d)] - 
// 		s_F[flatten(s_i - 1, s_j, s_k, s_w, s_h, s_d)] ) / (2.0*dx);

// 	dFdy = ( s_F[flatten(s_i, s_j + 1, s_k, s_w, s_h, s_d)] - 
// 		s_F[flatten(s_i, s_j - 1, s_k, s_w, s_h, s_d)] ) / (2.0*dx);

// 	dFdz = ( s_F[flatten(s_i, s_j, s_k + 1, s_w, s_h, s_d)] - 
// 		s_F[flatten(s_i, s_j, s_k - 1, s_w, s_h, s_d)] ) / (2.0*dx);

// 	__syncthreads();

// 	// Test to see if z is <= Zst, which sets the value of chi
// 	s_F[s_idx] = (s_F[s_idx] <= ref); 

// 	// Test Halo Cells to form chi
// 	if (threadIdx.x < RAD) {
// 		s_F[flatten(s_i - RAD, s_j, s_k, s_w, s_h, s_d)] = (s_F[flatten(s_i - RAD, s_j, s_k, s_w, s_h, s_d)] <= ref);
// 		s_F[flatten(s_i + blockDim.x, s_j, s_k, s_w, s_h, s_d)] = (s_F[flatten(s_i + blockDim.x, s_j, s_k, s_w, s_h, s_d)] <= ref);
// 	}
// 	if (threadIdx.y < RAD) {
// 		s_F[flatten(s_i, s_j - RAD, s_k, s_w, s_h, s_d)] = (s_F[flatten(s_i, s_j - RAD, s_k, s_w, s_h, s_d)] <= ref);
// 		s_F[flatten(s_i, s_j + blockDim.y, s_k, s_w, s_h, s_d)] = (s_F[flatten(s_i, s_j + blockDim.y, s_k, s_w, s_h, s_d)] <= ref);
// 	}
// 	if (threadIdx.z < RAD) {
// 		s_F[flatten(s_i, s_j, s_k - RAD, s_w, s_h, s_d)] = (s_F[flatten(s_i, s_j, s_k - RAD, s_w, s_h, s_d)] <= ref);
// 		s_F[flatten(s_i, s_j, s_k + blockDim.z, s_w, s_h, s_d)] = (s_F[flatten(s_i, s_j, s_k + blockDim.z, s_w, s_h, s_d)] <= ref);
// 	}

// 	__syncthreads();

// 	// Take derivatives
// 	dchidx = ( s_F[flatten(s_i + 1, s_j, s_k, s_w, s_h, s_d)] - 
// 		s_F[flatten(s_i - 1, s_j, s_k, s_w, s_h, s_d)] ) / (2.0*dx);

// 	dchidy = ( s_F[flatten(s_i, s_j + 1, s_k, s_w, s_h, s_d)] - 
// 		s_F[flatten(s_i, s_j - 1, s_k, s_w, s_h, s_d)] ) / (2.0*dx);
	
// 	dchidz = ( s_F[flatten(s_i, s_j, s_k + 1, s_w, s_h, s_d)] - 
// 		s_F[flatten(s_i, s_j, s_k - 1, s_w, s_h, s_d)] ) / (2.0*dx);

// 	__syncthreads();

// 	// Compute Length contribution for each thread
// 	if (dFdx == 0 && dFdy == 0 && dFdz == 0){
// 		s_F[s_idx] = 0;
// 	}
// 	else if (dchidx == 0 && dchidy == 0 && dchidz == 0){
// 		s_F[s_idx] = 0;
// 	}
// 	else{
// 		s_F[s_idx] = -(dFdx * dchidx + dFdy * dchidy + dFdz * dchidz) / sqrtf(dFdx * dFdx + dFdy * dFdy + dFdz * dFdz);
// 	}

// 	// __syncthreads();

// 	// Add length contribution from each thread into block memory
// 	if (threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0){
// 		double local_SA = 0.0;
// 		for (int q = 1; q <= blockDim.x; ++q) {
// 			for (int r = 1; r <= blockDim.y; ++r){
// 				for (int s = 1; s <= blockDim.z; ++s){
// 					int local_idx = flatten(q, r, s, s_w, s_h, s_d);
// 					local_SA += s_F[local_idx];
// 				}
// 			}
// 		}
// 		__syncthreads();
// 		atomicAdd(SA, local_SA*dx*dx*dx);
// 	}

// 	return;
// }


// __global__
// void surfaceArea_kernel_mgpu(const int start_x, const int w, const int h, const int d, double *F, double ref, double *SA) {
// 	extern __shared__ double s_F[];

// 	double dFdx, dFdy, dFdz, dchidx, dchidy, dchidz;

// 	// global indices
// 	const int i = blockIdx.x * blockDim.x + threadIdx.x; // column
// 	const int j = blockIdx.y * blockDim.y + threadIdx.y; // row
// 	const int k = blockIdx.z * blockDim.z + threadIdx.z; // stack
// 	if (((i+start_x) >= NX) || (j >= NY) || (k >= NZ)) return;
// 	const int idx = flatten(i, j, k, w, h, d);
// 	// local width and height
// 	const int s_w = blockDim.x + 2 * RAD;
// 	const int s_h = blockDim.y + 2 * RAD;
// 	const int s_d = blockDim.z + 2 * RAD;
// 	// local indices
// 	const int s_i = threadIdx.x + RAD;
// 	const int s_j = threadIdx.y + RAD;
// 	const int s_k = threadIdx.z + RAD;
// 	const int s_idx = flatten(s_i, s_j, s_k, s_w, s_h, s_d);

// 	// Creating arrays in shared memory
// 	// Regular cells
// 	s_F[s_idx] = F[idx];

// 	//Halo Cells
// 	if (threadIdx.x < RAD) {
// 		s_F[flatten(s_i - RAD, s_j, s_k, s_w, s_h, s_d)] =
// 			F[flatten(i - RAD, j, k, w, h, d)];
// 		s_F[flatten(s_i + blockDim.x, s_j, s_k, s_w, s_h, s_d)] =
// 			F[flatten(i + blockDim.x, j, k, w, h, d)];
// 	}
// 	if (threadIdx.y < RAD) {
// 		s_F[flatten(s_i, s_j - RAD, s_k, s_w, s_h, s_d)] =
// 			F[flatten(i, j - RAD, k, w, h, d)];
// 		s_F[flatten(s_i, s_j + blockDim.y, s_k, s_w, s_h, s_d)] =
// 			F[flatten(i, j + blockDim.y, k, w, h, d)];
// 	}
// 	if (threadIdx.z < RAD) {
// 		s_F[flatten(s_i, s_j, s_k - RAD, s_w, s_h, s_d)] =
// 			F[flatten(i, j, k - RAD, w, h, d)];
// 		s_F[flatten(s_i, s_j, s_k + blockDim.z, s_w, s_h, s_d)] =
// 			F[flatten(i, j, k + blockDim.z, w, h, d)];
// 	}

// 	__syncthreads();

// 	// Boundary Conditions
// 	// Making problem boundaries periodic
// 	if (i == 0){
// 		s_F[flatten(s_i - 1, s_j, s_k, s_w, s_h, s_d)] = 
// 			F[flatten(w, j, k, w, h, d)];
// 	}
// 	if (i == w - 1){
// 		s_F[flatten(s_i + 1, s_j, s_k, s_w, s_h, s_d)] =
// 			F[flatten(0, j, k, w, h, d)];
// 	}

// 	if (j == 0){
// 		s_F[flatten(s_i, s_j - 1, s_k, s_w, s_h, s_d)] = 
// 			F[flatten(i, h, k, w, h, d)];
// 	}
// 	if (j == h - 1){
// 		s_F[flatten(s_i, s_j + 1, s_k, s_w, s_h, s_d)] =
// 			F[flatten(i, 0, k, w, h, d)];
// 	}

// 	if (k == 0){
// 		s_F[flatten(s_i, s_j, s_k - 1, s_w, s_h, s_d)] = 
// 			F[flatten(i, j, d, w, h, d)];
// 	}
// 	if (k == d - 1){
// 		s_F[flatten(s_i, s_j, s_k + 1, s_w, s_h, s_d)] =
// 			F[flatten(i, j, 0, w, h, d)];
// 	}

// 	// __syncthreads();

// 	// Calculating dFdx and dFdy
// 	// Take derivatives

// 	dFdx = ( s_F[flatten(s_i + 1, s_j, s_k, s_w, s_h, s_d)] - 
// 		s_F[flatten(s_i - 1, s_j, s_k, s_w, s_h, s_d)] ) / (2.0*DX);

// 	dFdy = ( s_F[flatten(s_i, s_j + 1, s_k, s_w, s_h, s_d)] - 
// 		s_F[flatten(s_i, s_j - 1, s_k, s_w, s_h, s_d)] ) / (2.0*DX);

// 	dFdz = ( s_F[flatten(s_i, s_j, s_k + 1, s_w, s_h, s_d)] - 
// 		s_F[flatten(s_i, s_j, s_k - 1, s_w, s_h, s_d)] ) / (2.0*DX);

// 	__syncthreads();

// 	// Test to see if z is <= Zst, which sets the value of chi
// 	s_F[s_idx] = (s_F[s_idx] <= ref); 

// 	// Test Halo Cells to form chi
// 	if (threadIdx.x < RAD) {
// 		s_F[flatten(s_i - RAD, s_j, s_k, s_w, s_h, s_d)] = (s_F[flatten(s_i - RAD, s_j, s_k, s_w, s_h, s_d)] <= ref);
// 		s_F[flatten(s_i + blockDim.x, s_j, s_k, s_w, s_h, s_d)] = (s_F[flatten(s_i + blockDim.x, s_j, s_k, s_w, s_h, s_d)] <= ref);
// 	}
// 	if (threadIdx.y < RAD) {
// 		s_F[flatten(s_i, s_j - RAD, s_k, s_w, s_h, s_d)] = (s_F[flatten(s_i, s_j - RAD, s_k, s_w, s_h, s_d)] <= ref);
// 		s_F[flatten(s_i, s_j + blockDim.y, s_k, s_w, s_h, s_d)] = (s_F[flatten(s_i, s_j + blockDim.y, s_k, s_w, s_h, s_d)] <= ref);
// 	}
// 	if (threadIdx.z < RAD) {
// 		s_F[flatten(s_i, s_j, s_k - RAD, s_w, s_h, s_d)] = (s_F[flatten(s_i, s_j, s_k - RAD, s_w, s_h, s_d)] <= ref);
// 		s_F[flatten(s_i, s_j, s_k + blockDim.z, s_w, s_h, s_d)] = (s_F[flatten(s_i, s_j, s_k + blockDim.z, s_w, s_h, s_d)] <= ref);
// 	}

// 	__syncthreads();

// 	// Take derivatives
// 	dchidx = ( s_F[flatten(s_i + 1, s_j, s_k, s_w, s_h, s_d)] - 
// 		s_F[flatten(s_i - 1, s_j, s_k, s_w, s_h, s_d)] ) / (2.0*DX);

// 	dchidy = ( s_F[flatten(s_i, s_j + 1, s_k, s_w, s_h, s_d)] - 
// 		s_F[flatten(s_i, s_j - 1, s_k, s_w, s_h, s_d)] ) / (2.0*DX);
	
// 	dchidz = ( s_F[flatten(s_i, s_j, s_k + 1, s_w, s_h, s_d)] - 
// 		s_F[flatten(s_i, s_j, s_k - 1, s_w, s_h, s_d)] ) / (2.0*DX);

// 	__syncthreads();

// 	// Compute Length contribution for each thread
// 	if (dFdx == 0 && dFdy == 0 && dFdz == 0){
// 		s_F[s_idx] = 0;
// 	}
// 	else if (dchidx == 0 && dchidy == 0 && dchidz == 0){
// 		s_F[s_idx] = 0;
// 	}
// 	else{
// 		s_F[s_idx] = -(dFdx * dchidx + dFdy * dchidy + dFdz * dchidz) / sqrtf(dFdx * dFdx + dFdy * dFdy + dFdz * dFdz);
// 	}

// 	// __syncthreads();

// 	// Add length contribution from each thread into block memory
// 	if (threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0){
// 		double local_SA = 0.0;
// 		for (int p = RAD; p <= blockDim.x; ++p) {
// 			for (int q = RAD; q <= blockDim.y; ++q){
// 				for (int r = RAD; r <= blockDim.z; ++r){
// 					int local_idx = flatten(p, q, r, s_w, s_h, s_d);
// 					local_SA += s_F[local_idx];
// 				}
// 			}
// 		}
// 		__syncthreads();
// 		atomicAdd(SA, local_SA*DX*DX*DX);
// 	}

// 	return;
// }

__global__
void surfaceArea_kernel_mgpu(const int start_x, const int w, const int h, const int d, double *F, double ref, double *SA) {
	extern __shared__ double s_F[];

	double dFdx, dFdy, dFdz, dchidx, dchidy, dchidz;

	// global indices
	const int i = blockIdx.x * blockDim.x + threadIdx.x; // column
	const int j = blockIdx.y * blockDim.y + threadIdx.y; // row
	const int k = blockIdx.z * blockDim.z + threadIdx.z; // stack
	if (((i+start_x) >= NX) || (j >= NY) || (k >= NZ)) return;
	const int idx = flatten(i, j, k, w, h, d);
	// local width and height
	const int s_w = blockDim.x + 2 * RAD;
	const int s_h = blockDim.y + 2 * RAD;
	const int s_d = blockDim.z + 2 * RAD;
	// local indices
	const int s_i = threadIdx.x + RAD;
	const int s_j = threadIdx.y + RAD;
	const int s_k = threadIdx.z + RAD;
	const int s_idx = flatten(s_i, s_j, s_k, s_w, s_h, s_d);

	// Creating arrays in shared memory
	// Regular cells
	s_F[s_idx] = F[idx];

	//Halo Cells
	if (threadIdx.x < RAD) {
		s_F[flatten(s_i - RAD, s_j, s_k, s_w, s_h, s_d)] =
			F[flatten(i - RAD, j, k, w, h, d)];
		s_F[flatten(s_i + blockDim.x, s_j, s_k, s_w, s_h, s_d)] =
			F[flatten(i + blockDim.x, j, k, w, h, d)];
	}
	if (threadIdx.y < RAD) {
		s_F[flatten(s_i, s_j - RAD, s_k, s_w, s_h, s_d)] =
			F[flatten(i, j - RAD, k, w, h, d)];
		s_F[flatten(s_i, s_j + blockDim.y, s_k, s_w, s_h, s_d)] =
			F[flatten(i, j + blockDim.y, k, w, h, d)];
	}
	if (threadIdx.z < RAD) {
		s_F[flatten(s_i, s_j, s_k - RAD, s_w, s_h, s_d)] =
			F[flatten(i, j, k - RAD, w, h, d)];
		s_F[flatten(s_i, s_j, s_k + blockDim.z, s_w, s_h, s_d)] =
			F[flatten(i, j, k + blockDim.z, w, h, d)];
	}

	__syncthreads();

	// Boundary Conditions
	// Making problem boundaries periodic
	if (i == 0){
		s_F[flatten(s_i - 1, s_j, s_k, s_w, s_h, s_d)] = 
			F[flatten(w, j, k, w, h, d)];
	}
	if (i == w - 1){
		s_F[flatten(s_i + 1, s_j, s_k, s_w, s_h, s_d)] =
			F[flatten(0, j, k, w, h, d)];
	}

	if (j == 0){
		s_F[flatten(s_i, s_j - 1, s_k, s_w, s_h, s_d)] = 
			F[flatten(i, h, k, w, h, d)];
	}
	if (j == h - 1){
		s_F[flatten(s_i, s_j + 1, s_k, s_w, s_h, s_d)] =
			F[flatten(i, 0, k, w, h, d)];
	}

	if (k == 0){
		s_F[flatten(s_i, s_j, s_k - 1, s_w, s_h, s_d)] = 
			F[flatten(i, j, d, w, h, d)];
	}
	if (k == d - 1){
		s_F[flatten(s_i, s_j, s_k + 1, s_w, s_h, s_d)] =
			F[flatten(i, j, 0, w, h, d)];
	}

	// __syncthreads();

	// Calculating dFdx and dFdy
	// Take derivatives

	dFdx = ( s_F[flatten(s_i + 1, s_j, s_k, s_w, s_h, s_d)] - 
		s_F[flatten(s_i - 1, s_j, s_k, s_w, s_h, s_d)] ) / (2.0*DX);

	dFdy = ( s_F[flatten(s_i, s_j + 1, s_k, s_w, s_h, s_d)] - 
		s_F[flatten(s_i, s_j - 1, s_k, s_w, s_h, s_d)] ) / (2.0*DX);

	dFdz = ( s_F[flatten(s_i, s_j, s_k + 1, s_w, s_h, s_d)] - 
		s_F[flatten(s_i, s_j, s_k - 1, s_w, s_h, s_d)] ) / (2.0*DX);

	__syncthreads();

	// Test to see if z is <= Zst, which sets the value of chi
	s_F[s_idx] = (s_F[s_idx] <= ref); 

	// Test Halo Cells to form chi
	if (threadIdx.x < RAD) {
		s_F[flatten(s_i - RAD, s_j, s_k, s_w, s_h, s_d)] = (s_F[flatten(s_i - RAD, s_j, s_k, s_w, s_h, s_d)] <= ref);
		s_F[flatten(s_i + blockDim.x, s_j, s_k, s_w, s_h, s_d)] = (s_F[flatten(s_i + blockDim.x, s_j, s_k, s_w, s_h, s_d)] <= ref);
	}
	if (threadIdx.y < RAD) {
		s_F[flatten(s_i, s_j - RAD, s_k, s_w, s_h, s_d)] = (s_F[flatten(s_i, s_j - RAD, s_k, s_w, s_h, s_d)] <= ref);
		s_F[flatten(s_i, s_j + blockDim.y, s_k, s_w, s_h, s_d)] = (s_F[flatten(s_i, s_j + blockDim.y, s_k, s_w, s_h, s_d)] <= ref);
	}
	if (threadIdx.z < RAD) {
		s_F[flatten(s_i, s_j, s_k - RAD, s_w, s_h, s_d)] = (s_F[flatten(s_i, s_j, s_k - RAD, s_w, s_h, s_d)] <= ref);
		s_F[flatten(s_i, s_j, s_k + blockDim.z, s_w, s_h, s_d)] = (s_F[flatten(s_i, s_j, s_k + blockDim.z, s_w, s_h, s_d)] <= ref);
	}

	__syncthreads();

	// Take derivatives
	dchidx = ( s_F[flatten(s_i + 1, s_j, s_k, s_w, s_h, s_d)] - 
		s_F[flatten(s_i - 1, s_j, s_k, s_w, s_h, s_d)] ) / (2.0*DX);

	dchidy = ( s_F[flatten(s_i, s_j + 1, s_k, s_w, s_h, s_d)] - 
		s_F[flatten(s_i, s_j - 1, s_k, s_w, s_h, s_d)] ) / (2.0*DX);
	
	dchidz = ( s_F[flatten(s_i, s_j, s_k + 1, s_w, s_h, s_d)] - 
		s_F[flatten(s_i, s_j, s_k - 1, s_w, s_h, s_d)] ) / (2.0*DX);

	__syncthreads();

	// Compute Length contribution for each thread
	if (dFdx == 0 && dFdy == 0 && dFdz == 0){
		s_F[s_idx] = 0;
	}
	else if (dchidx == 0 && dchidy == 0 && dchidz == 0){
		s_F[s_idx] = 0;
	}
	else{
		s_F[s_idx] = -(dFdx * dchidx + dFdy * dchidy + dFdz * dchidz) / sqrtf(dFdx * dFdx + dFdy * dFdy + dFdz * dFdz);
	}

	// __syncthreads();

	// Add length contribution from each thread into block memory
	if (threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0){
		double local_SA = 0.0;
		for (int p = RAD; p <= blockDim.x; ++p) {
			for (int q = RAD; q <= blockDim.y; ++q){
				for (int r = RAD; r <= blockDim.z; ++r){
					int local_idx = flatten(p, q, r, s_w, s_h, s_d);
					local_SA += s_F[local_idx];
				}
			}
		}
		__syncthreads();
		atomicAdd(SA, local_SA*DX*DX*DX);
	}

	return;
}

void calcSurfaceArea(const int c, gpuinfo gpu, hipfftDoubleReal **f, double iso, double **Area){
// Function to calculate surface quantities
	int n;

	hipError_t err;
	for(n=0; n<gpu.nGPUs; ++n){
		hipSetDevice(n);

		// Declare and allocate temporary variables
		const dim3 blockSize(TX, TY, TZ);
		const dim3 gridSize(divUp(gpu.nx[n], TX), divUp(NY, TY), divUp(NZ, TZ));
		const size_t smemSize = (TX + 2*RAD)*(TY + 2*RAD)*(TZ + 2*RAD)*sizeof(double);

		// Calculate surface area based on the value of iso
		surfaceArea_kernel_mgpu<<<gridSize, blockSize, smemSize>>>(gpu.start_x[n], NX, NY, 2*NZ2, f[n], iso, &Area[n][c]);			// Using 2*NZ2 for z-index due to in-place FFT storage
		err = hipGetLastError();
		if (err != hipSuccess) 
	    printf("Error: %s\n", hipGetErrorString(err));
	}

		return;

}


__global__
void calcVrmsKernel_mgpu(int start_y, double *wave, hipfftDoubleComplex *u1hat, hipfftDoubleComplex *u2hat, hipfftDoubleComplex *u3hat, double *RMS, double *KE){
// Function to calculate the RMS velocity of a flow field

	// Declare variables
	extern __shared__ double vel_mag[];

	const int i = blockIdx.x * blockDim.x + threadIdx.x;
	const int j = blockIdx.y * blockDim.y + threadIdx.y;
	const int k = blockIdx.z * blockDim.z + threadIdx.z;
	if ((i >= NX) || ( (j+start_y) >= NY) || (k >= NZ)) return;
	int kp = NZ-k;
	const int idx = flatten(j, i, k, NY, NX, NZ2);
	const int idx2 = flatten(j, i, kp, NY, NX, NZ2);
	// Create shared memory indices
	// local width and height
	const int s_w = blockDim.x;
	const int s_h = blockDim.y;
	const int s_d = blockDim.z;
	// local indices
	const int s_col = threadIdx.x;
	const int s_row = threadIdx.y;
	const int s_sta = threadIdx.z;
	const int s_idx = flatten(s_row, s_col, s_sta, s_h, s_w, s_d);

// Step 1: Calculate velocity magnitude at each point in the domain
	// Requires calculation of uu*, or multiplication of u with its complex conjugate
	// Mathematically, multiplying a number u = a + ib by its complex conjugate means
	// uu* = (a + ib) * (a - ib) = a^2 + b^2.
	// Some funky indexing is required because only half of the domain is represented in the complex form
	// (or is it? Can potentially just compute on the standard grid and multiply by 2....)
	if (k < NZ2){
		vel_mag[s_idx] = (u1hat[idx].x*u1hat[idx].x + u1hat[idx].y*u1hat[idx].y)/((double)NN*NN) + (u2hat[idx].x*u2hat[idx].x + u2hat[idx].y*u2hat[idx].y)/((double)NN*NN) + (u3hat[idx].x*u3hat[idx].x + u3hat[idx].y*u3hat[idx].y)/((double)NN*NN);
	}
	else{
		vel_mag[s_idx] = (u1hat[idx2].x*u1hat[idx2].x + u1hat[idx2].y*u1hat[idx2].y)/((double)NN*NN) + (u2hat[idx2].x*u2hat[idx2].x + u2hat[idx2].y*u2hat[idx2].y)/((double)NN*NN) + (u3hat[idx2].x*u3hat[idx2].x + u3hat[idx2].y*u3hat[idx2].y)/((double)NN*NN);
	}

	__syncthreads();

// Step 2: Add all of the contributions together ( need to use Atomic Add to make sure that all points are added correctly)
// Need to perform data reduction
	// Calculate sum of the velocity magnitude for each block
	if (s_idx == 0){

		double blockSum = 0.0;
		int c;
		for (c = 0; c < blockDim.x*blockDim.y*blockDim.z; ++c) {
			blockSum += vel_mag[c];
		}

		__syncthreads();

		// Step 3: Add all blocks together into device memory using Atomic operations (requires -arch=sm_60 or higher)

		// Kinetic Energy
		atomicAdd(KE, blockSum/2.0);
		// RMS velocity
		atomicAdd(RMS, blockSum/3.0);

	}

	return;
}

__global__
void calcEpsilonKernel_mgpu(int start_y, double *wave, hipfftDoubleComplex *u1hat, hipfftDoubleComplex *u2hat, hipfftDoubleComplex *u3hat, double *eps){
// Function to calculate the rate of dissipation of kinetic energy in a flow field

	// Declare variables
	extern __shared__ double vel_mag[];

	const int i = blockIdx.x * blockDim.x + threadIdx.x;
	const int j = blockIdx.y * blockDim.y + threadIdx.y;
	const int k = blockIdx.z * blockDim.z + threadIdx.z;
	if ((i >= NX) || ((j+start_y) >= NY) || (k >= NZ)) return;
	int kp = NZ-k;
	const int idx = flatten(j, i, k, NY, NX, NZ2);
	const int idx2 = flatten(j, i, kp, NY, NX, NZ2);
	// Create shared memory indices
	// local width and height
	const int s_w = blockDim.x;
	const int s_h = blockDim.y;
	const int s_d = blockDim.z;
	// local indices
	const int s_col = threadIdx.x;
	const int s_row = threadIdx.y;
	const int s_sta = threadIdx.z;
	const int s_idx = flatten(s_row, s_col, s_sta, s_h, s_w, s_d);

// Step 1: Calculate k_sq*velocity magnitude at each point in the domain
	// Requires calculation of uu*, or multiplication of u with its complex conjugate
	// Mathematically, multiplying a number u = a + ib by its complex conjugate means
	// uu* = (a + ib) * (a - ib) = a^2 + b^2.
	// Some funky indexing is required because only half of the domain is represented in the complex form
	if (k < NZ2){
		vel_mag[s_idx] = (wave[i]*wave[i] + wave[(j+start_y)]*wave[(j+start_y)] + wave[k]*wave[k] )*( (u1hat[idx].x*u1hat[idx].x + u1hat[idx].y*u1hat[idx].y)/((double)NN*NN) + (u2hat[idx].x*u2hat[idx].x + u2hat[idx].y*u2hat[idx].y)/((double)NN*NN) + (u3hat[idx].x*u3hat[idx].x + u3hat[idx].y*u3hat[idx].y)/((double)NN*NN) );
	}
	else{
		vel_mag[s_idx] = (wave[i]*wave[i] + wave[(j+start_y)]*wave[(j+start_y)] + wave[k]*wave[k] )*( (u1hat[idx2].x*u1hat[idx2].x + u1hat[idx2].y*u1hat[idx2].y)/((double)NN*NN) + (u2hat[idx2].x*u2hat[idx2].x + u2hat[idx2].y*u2hat[idx2].y)/((double)NN*NN) + (u3hat[idx2].x*u3hat[idx2].x + u3hat[idx2].y*u3hat[idx2].y)/((double)NN*NN) );
	}

	__syncthreads();

// Step 2: Add all of the contributions together ( need to use Atomic Add to make sure that all points are added correctly)
// Need to perform data reduction
// Calculate sum of the nu*k_sq*velocity magnitude for each block
	if (threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0){

		double blockSum = 0.0;
		for (int i = 0; i < blockDim.x*blockDim.y*blockDim.z; ++i) {
			blockSum += nu*vel_mag[i];
		}
		__syncthreads();

		// Dissipation Rate
		atomicAdd(eps, blockSum);
	}

	return;
}

__global__
void calcIntegralLengthKernel_mgpu(int start_y, double *wave, hipfftDoubleComplex *u1hat, hipfftDoubleComplex *u2hat, hipfftDoubleComplex *u3hat, double *l){
// Function to calculate the integral length scale of a turbulent flow field

	// Declare variables
	extern __shared__ double vel_mag[];

	const int i = blockIdx.x * blockDim.x + threadIdx.x;
	const int j = blockIdx.y * blockDim.y + threadIdx.y;
	const int k = blockIdx.z * blockDim.z + threadIdx.z;
	if ((i >= NX) || ((j+start_y) >= NY) || (k >= NZ)) return;
	int kp = NZ-k;
	const int idx = flatten(j, i, k, NY, NX, NZ2);
	const int idx2 = flatten(j, i, kp, NY, NX, NZ2);
	// Create shared memory indices
	// local width and height
	const int s_w = blockDim.x;
	const int s_h = blockDim.y;
	const int s_d = blockDim.z;
	// local indices
	const int s_col = threadIdx.x;
	const int s_row = threadIdx.y;
	const int s_sta = threadIdx.z;
	const int s_idx = flatten(s_row, s_col, s_sta, s_h, s_w, s_d);

// Step 1: Calculate velocity magnitude at each point in the domain
	// Requires calculation of uu*, or multiplication of u with its complex conjugate
	// Mathematically, multiplying a number u = a + ib by its complex conjugate means
	// uu* = (a + ib) * (a - ib) = a^2 + b^2.
	// Some funky indexing is required because only half of the domain is represented in the complex form
	vel_mag[s_idx] = 0.0;
	if (wave[i]*wave[i] + wave[(j+start_y)]*wave[(j+start_y)] + wave[k]*wave[k] > 0){
		if (k < NZ2){
			vel_mag[s_idx] = ( (u1hat[idx].x*u1hat[idx].x + u1hat[idx].y*u1hat[idx].y)/((double)NN*NN) + (u2hat[idx].x*u2hat[idx].x + u2hat[idx].y*u2hat[idx].y)/((double)NN*NN) + (u3hat[idx].x*u3hat[idx].x + u3hat[idx].y*u3hat[idx].y)/((double)NN*NN) )/( 2.0*sqrt(wave[i]*wave[i] + wave[(j+start_y)]*wave[(j+start_y)] + wave[k]*wave[k]) );
		}
		else{
			vel_mag[s_idx] = ( (u1hat[idx2].x*u1hat[idx2].x + u1hat[idx2].y*u1hat[idx2].y)/((double)NN*NN) + (u2hat[idx2].x*u2hat[idx2].x + u2hat[idx2].y*u2hat[idx2].y)/((double)NN*NN) + (u3hat[idx2].x*u3hat[idx2].x + u3hat[idx2].y*u3hat[idx2].y)/((double)NN*NN) )/( 2.0*sqrt(wave[i]*wave[i] + wave[(j+start_y)]*wave[(j+start_y)] + wave[k]*wave[k]) );
		}
	}

	__syncthreads();

// Step 2: Add all of the contributions together ( need to use Atomic Add to make sure that all points are added correctly)
// Need to perform data reduction
// Calculate sum of the velocity magnitude for each block
	if (threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0){

		double blockSum = 0.0;
		for (int i = 0; i < blockDim.x*blockDim.y*blockDim.z; ++i) {
			blockSum += vel_mag[i];
		}

		__syncthreads();

		// Dissipation Rate
		atomicAdd(l, blockSum);
	}

	return;
}

__global__
void calcScalarDissipationKernel_mgpu(int start_y, double *wave, hipfftDoubleComplex *zhat, double *chi){
// Function to calculate the RMS velocity of a flow field

	// Declare variables
	extern __shared__ double sca_mag[];

	const int i = blockIdx.x * blockDim.x + threadIdx.x;
	const int j = blockIdx.y * blockDim.y + threadIdx.y;
	const int k = blockIdx.z * blockDim.z + threadIdx.z;
	if ((i >= NX) || ( (j+start_y) >= NY) || (k >= NZ)) return;
	int kp = NZ-k;
	const int idx = flatten(j, i, k, NY, NX, NZ2);
	const int idx2 = flatten(j, i, kp, NY, NX, NZ2);
	// Create shared memory indices
	// local width and height
	const int s_w = blockDim.x;
	const int s_h = blockDim.y;
	const int s_d = blockDim.z;
	// local indices
	const int s_col = threadIdx.x;
	const int s_row = threadIdx.y;
	const int s_sta = threadIdx.z;
	const int s_idx = flatten(s_row, s_col, s_sta, s_h, s_w, s_d);

// Step 1: Calculate velocity magnitude at each point in the domain
	// Requires calculation of uu*, or multiplication of u with its complex conjugate
	// Mathematically, multiplying a number u = a + ib by its complex conjugate means
	// uu* = (a + ib) * (a - ib) = a^2 + b^2.
	// Some funky indexing is required because only half of the domain is represented in the complex form
	// (or is it? Can potentially just compute on the standard grid and multiply by 2....)
	if (k < NZ2){
		sca_mag[s_idx] = (wave[i]*wave[i] + wave[(j+start_y)]*wave[(j+start_y)] + wave[k]*wave[k] )*(zhat[idx].x*zhat[idx].x + zhat[idx].y*zhat[idx].y)/((double)NN*NN);
	}
	else{
		sca_mag[s_idx] = (wave[i]*wave[i] + wave[(j+start_y)]*wave[(j+start_y)] + wave[k]*wave[k] )*(zhat[idx2].x*zhat[idx2].x + zhat[idx2].y*zhat[idx2].y)/((double)NN*NN);
	}

	__syncthreads();

// Step 2: Add all of the contributions together ( need to use Atomic Add to make sure that all points are added correctly)
// Need to perform data reduction
	// Calculate sum of the velocity magnitude for each block
	if (threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0){

		double blockSum = 0.0;

		for (int i = 0; i < blockDim.x*blockDim.y*blockDim.z; ++i) {
			blockSum += 2*(nu/Sc)*sca_mag[i];
		}

		__syncthreads();

		// Step 3: Add all blocks together into device memory using Atomic operations (requires -arch=sm_60 or higher)

		// Scalar Dissipation
		atomicAdd(chi, blockSum);

	}

	return;
}

__global__
void calcEnergySpectraKernel_mgpu(int start_y, double *wave, hipfftDoubleComplex *u1hat, hipfftDoubleComplex *u2hat, hipfftDoubleComplex *u3hat, double *e){
// Function to calculate the integral length scale of a turbulent flow field

	// Declare variables
	extern __shared__ double vel_mag[];

	const int i = blockIdx.x * blockDim.x + threadIdx.x;
	const int j = blockIdx.y * blockDim.y + threadIdx.y;
	const int k = blockIdx.z * blockDim.z + threadIdx.z;
	if ((i >= NX) || ((j+start_y) >= NY) || (k >= NZ)) return;
	int kp = NZ-k;
	const int idx = flatten(j, i, k, NY, NX, NZ2);
	const int idx2 = flatten(j, i, kp, NY, NX, NZ2);
	// Create shared memory indices
	// local width and height
	const int s_w = blockDim.x;
	const int s_h = blockDim.y;
	const int s_d = blockDim.z;
	// local indices
	const int s_col = threadIdx.x;
	const int s_row = threadIdx.y;
	const int s_sta = threadIdx.z;
	const int s_idx = flatten(s_row, s_col, s_sta, s_h, s_w, s_d);

// Step 1: Calculate velocity magnitude at each point in the domain
	// Requires calculation of uu*, or multiplication of u with its complex conjugate
	// Mathematically, multiplying a number u = a + ib by its complex conjugate means
	// uu* = (a + ib) * (a - ib) = a^2 + b^2.
	// Some funky indexing is required because only half of the domain is represented in the complex form
	vel_mag[s_idx] = 0.0;
	// if (wave[i]*wave[i] + wave[(j+start_y)]*wave[(j+start_y)] + wave[k]*wave[k] > 0){
		if (k < NZ2){
			vel_mag[s_idx] = ( (u1hat[idx].x*u1hat[idx].x + u1hat[idx].y*u1hat[idx].y)/((double)NN*NN) + (u2hat[idx].x*u2hat[idx].x + u2hat[idx].y*u2hat[idx].y)/((double)NN*NN) + (u3hat[idx].x*u3hat[idx].x + u3hat[idx].y*u3hat[idx].y)/((double)NN*NN) )/( 2.0*sqrt(wave[i]*wave[i] + wave[(j+start_y)]*wave[(j+start_y)] + wave[k]*wave[k]) );
		}
		else{
			vel_mag[s_idx] = ( (u1hat[idx2].x*u1hat[idx2].x + u1hat[idx2].y*u1hat[idx2].y)/((double)NN*NN) + (u2hat[idx2].x*u2hat[idx2].x + u2hat[idx2].y*u2hat[idx2].y)/((double)NN*NN) + (u3hat[idx2].x*u3hat[idx2].x + u3hat[idx2].y*u3hat[idx2].y)/((double)NN*NN) )/( 2.0*sqrt(wave[i]*wave[i] + wave[(j+start_y)]*wave[(j+start_y)] + wave[k]*wave[k]) );
		}
	// }

	__syncthreads();

// Step 2: Add all of the contributions together ( need to use Atomic Add to make sure that all points are added correctly)
// Need to perform data reduction
// Calculate sum of the velocity magnitude for each block
	if (threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0){

		double blockSum = 0.0;
		for (int i = 0; i < blockDim.x*blockDim.y*blockDim.z; ++i) {
			blockSum += vel_mag[i];
		}

		__syncthreads();

		// Dissipation Rate
		atomicAdd(e, blockSum);
	}

	return;
}


void calcSpectra_mgpu(const int c, gpuinfo gpu, fftinfo fft, double **wave, fielddata vel, statistics stats)
{ // Calculate sperical energy and scalar spectra
	int n;

	// Loop over GPUs to call kernels
	for(n=0; n<gpu.nGPUs; ++n){
		hipSetDevice(n);

		// Set thread and block dimensions for kernal calls
		const dim3 blockSize(TX, TY, TZ);
		const dim3 gridSize(divUp(NX, TX), divUp(gpu.ny[n], TY), divUp(NZ, TZ));
		// const size_t smemSize = TX*TY*TZ*sizeof(double);
		hipError_t err;

		// Call kernels to calculate spherical energy spectra
		calcEnergySpectraKernel_mgpu<<<gridSize, blockSize>>>(gpu.start_y[n], wave[n], vel.uh[n], vel.vh[n], vel.wh[n], &stats.energy_spect[n][c]);
		err = hipGetLastError();
		if (err != hipSuccess) 
	    printf("Error: %s\n", hipGetErrorString(err));
	}

	return;
}

void calcTurbStats_mgpu(const int c, gpuinfo gpu, fftinfo fft, double **wave, fielddata vel, statistics stats)
{// Function to call a cuda kernel that calculates the relevant turbulent statistics

	// Synchronize GPUs before calculating statistics
	int n, nGPUs;

	// Make local copy of number of GPUs (for readability)
	nGPUs = gpu.nGPUs;	

	synchronizeGPUs(nGPUs);

	// Statistics for turbulent velocity field
	// Launch kernels to calculate stats
	for(n=0; n<nGPUs; ++n){
		hipSetDevice(n);

		// Set thread and block dimensions for kernal calls
		const dim3 blockSize(TX, TY, TZ);
		const dim3 gridSize(divUp(NX, TX), divUp(gpu.ny[n], TY), divUp(NZ, TZ));
		const size_t smemSize = TX*TY*TZ*sizeof(double);
		hipError_t err;

		// Call kernels to calculate turbulence statistics
		calcVrmsKernel_mgpu<<<gridSize, blockSize, smemSize>>>(gpu.start_y[n], wave[n], vel.uh[n], vel.vh[n], vel.wh[n], &stats.Vrms[n][c], &stats.KE[n][c]);
		err = hipGetLastError();
		if (err != hipSuccess) 
	    printf("Error: %s\n", hipGetErrorString(err));

		calcEpsilonKernel_mgpu<<<gridSize, blockSize, smemSize>>>(gpu.start_y[n], wave[n], vel.uh[n], vel.vh[n], vel.wh[n], &stats.epsilon[n][c]);
		err = hipGetLastError();
		if (err != hipSuccess) 
	    printf("Error: %s\n", hipGetErrorString(err));

		calcIntegralLengthKernel_mgpu<<<gridSize, blockSize, smemSize>>>(gpu.start_y[n], wave[n], vel.uh[n], vel.vh[n], vel.wh[n], &stats.l[n][c]);
		err = hipGetLastError();
		if (err != hipSuccess) 
	    printf("Error: %s\n", hipGetErrorString(err));

		calcScalarDissipationKernel_mgpu<<<gridSize, blockSize, smemSize>>>(gpu.start_y[n], wave[n], vel.sh[n], &stats.chi[n][c]);
		err = hipGetLastError();
		if (err != hipSuccess) 
	    printf("Error: %s\n", hipGetErrorString(err));

	}

	// Calculate energy and scalar spectra
	// calcSpectra_mgpu(c, gpu, fft, wave, vel, stats);

	synchronizeGPUs(nGPUs);

	// Post-processing for surface area, etc.

	// Transform scalar field to physical domain
	inverseTransform(fft, gpu, vel.sh);

	double iso = 0.5;
	calcSurfaceArea(c, gpu, vel.s, iso, stats.area_scalar);

	forwardTransform(fft, gpu, vel.s);
	
	synchronizeGPUs(nGPUs);			// Synchronize GPUs

	// Adding together results from all GPUs
	for(n=1; n<nGPUs; ++n){
		hipSetDevice(n);	
		hipDeviceSynchronize();

		stats.KE[0][c] += stats.KE[n][c];
		stats.Vrms[0][c] += stats.Vrms[n][c];
		stats.epsilon[0][c] += stats.epsilon[n][c];
		stats.l[0][c] += stats.l[n][c];
		stats.chi[0][c] += stats.chi[n][c];
		stats.area_scalar[0][c] += stats.area_scalar[n][c];
	}

	// "Post-processing" results from kernel calls - Calculating the remaining statistics
	//calcVrms kernel doesn't actually calculate the RMS velocity - Take square root to get Vrms
	stats.Vrms[0][c] = sqrt(stats.Vrms[0][c]);
	stats.lambda[0][c] = sqrt( 15.0*nu*stats.Vrms[0][c]*stats.Vrms[0][c]/stats.epsilon[0][c] );
	stats.eta[0][c] = sqrt(sqrt(nu*nu*nu/stats.epsilon[0][c]));
	stats.l[0][c] = 3*PI/4*stats.l[0][c]/stats.KE[0][c];

	return;
}
/*
int main()
{
// Function to calculate the relevant turbulent statistics of the flow at each time step.

// Set GPU's to use and list device properties
	int n, nGPUs;
	// Query number of devices attached to host
	nGPUs = 1;//hipGetDeviceCount(&nGPUs);
	// List properties of each device
	displayDeviceProps(nGPUs);

	printf("Running calcStats_mgpu using %d GPUs on a %dx%dx%d grid.\n",nGPUs,NX,NY,NZ);

	int i, c;

	// Split data according to number of GPUs
	int NX_per_GPU[nGPUs], NY_per_GPU[nGPUs], start_x[nGPUs], start_y[nGPUs];
	splitData(nGPUs, NX, NX_per_GPU, start_x);
	splitData(nGPUs, NY, NY_per_GPU, start_y);

	// Declare array of pointers to hold cuFFT plans
	hipfftHandle *plan2d;
	hipfftHandle *invplan2d;
	hipfftHandle *plan1d;
    size_t *worksize_f, *worksize_i;
    hipfftDoubleComplex **workspace;

	// Allocate memory for cuFFT plans
	// Allocate pinned memory on the host side that stores array of pointers
	hipHostAlloc((void**)&plan2d, nGPUs*sizeof(hipfftHandle), hipHostMallocMapped);
	hipHostAlloc((void**)&invplan2d, nGPUs*sizeof(hipfftHandle), hipHostMallocMapped);
	hipHostAlloc((void**)&plan1d, nGPUs*sizeof(hipfftHandle), hipHostMallocMapped);
    hipHostAlloc((void**)&worksize_f, nGPUs*sizeof(size_t *), hipHostMallocMapped);
    hipHostAlloc((void**)&worksize_i, nGPUs*sizeof(size_t *), hipHostMallocMapped);
    hipHostAlloc((void**)&workspace, nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);

    // Create plans for cuFFT on each GPU
    plan1dFFT(nGPUs, plan1d);
    plan2dFFT(nGPUs, NX_per_GPU, worksize_f, worksize_i, workspace, plan2d, invplan2d);

	// Allocate memory on host
	double **h_u;
	double **h_v;
	double **h_w;
	double **h_z;

	h_u = (double **)malloc(sizeof(double *)*nGPUs);
	h_v = (double **)malloc(sizeof(double *)*nGPUs);
	h_w = (double **)malloc(sizeof(double *)*nGPUs);
	h_z = (double **)malloc(sizeof(double *)*nGPUs);

	for(n=0; n<nGPUs; ++n){
		h_u[n] = (double *)malloc(sizeof(complex double)*NX_per_GPU[n]*NY*NZ2);
		h_v[n] = (double *)malloc(sizeof(complex double)*NX_per_GPU[n]*NY*NZ2);
		h_w[n] = (double *)malloc(sizeof(complex double)*NX_per_GPU[n]*NY*NZ2);
		h_z[n] = (double *)malloc(sizeof(complex double)*NX_per_GPU[n]*NY*NZ2);
	}

	// Allocate host memory for the statistics
	double *h_Vrms;
	double *h_KE;
	double *h_epsilon;
	double *h_eta;
	double *h_l;
	double *h_lambda;
	double *h_chi;

	h_Vrms = (double *)malloc(sizeof(double)*size_Stats);
	h_KE = (double *)malloc(sizeof(double)*size_Stats);
	h_epsilon = (double *)malloc(sizeof(double)*size_Stats);
	h_eta = (double *)malloc(sizeof(double)*size_Stats);
	h_l = (double *)malloc(sizeof(double)*size_Stats);
	h_lambda = (double *)malloc(sizeof(double)*size_Stats);
	h_chi = (double *)malloc(sizeof(double)*size_Stats);
	
	// Declare variables
	double **k;
	double **Vrms;
	double **epsilon;
	double **KE;
	double **eta;
	double **l;
	double **lambda;
	double **chi;
	// double **T4;
	// double **T5;
	// double **T5a;
	// double **T5b;

	hipfftDoubleReal **u;
	hipfftDoubleReal **v;
	hipfftDoubleReal **w;
	hipfftDoubleReal **z;

	hipfftDoubleComplex **uhat;
	hipfftDoubleComplex **vhat;
	hipfftDoubleComplex **what;
	hipfftDoubleComplex **zhat;

	hipfftDoubleComplex **temp;
	hipfftDoubleComplex **temp_reorder;
	hipfftDoubleComplex **temp_advective;

	// Allocate pinned memory on the host side that stores array of pointers
	hipHostAlloc((void**)&k, nGPUs*sizeof(double *), hipHostMallocMapped);

	hipHostAlloc((void**)&uhat, nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);
	hipHostAlloc((void**)&vhat, nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);
	hipHostAlloc((void**)&what, nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);
	hipHostAlloc((void**)&zhat, nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);

	hipHostAlloc((void**)&temp, nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);
	hipHostAlloc((void**)&temp_reorder, nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);
	hipHostAlloc((void**)&temp_advective, nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);
		
	hipHostAlloc((void**)&Vrms, nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);
	hipHostAlloc((void**)&KE, nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);
	hipHostAlloc((void**)&epsilon, nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);
	hipHostAlloc((void**)&eta, nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);
	hipHostAlloc((void**)&l, nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);
	hipHostAlloc((void**)&lambda, nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);
	hipHostAlloc((void**)&chi, nGPUs*sizeof(hipfftDoubleComplex *), hipHostMallocMapped);
	
	// Allocate memory for arrays
	for (n = 0; n<nGPUs; ++n){
		hipSetDevice(n);

		checkCudaErrors( hipMalloc((void **)&k[n], sizeof(double)*NX ) );

		checkCudaErrors( hipMalloc((void **)&uhat[n], sizeof(hipfftDoubleComplex)*NX_per_GPU[n]*NY*NZ2) ); 
		checkCudaErrors( hipMalloc((void **)&vhat[n], sizeof(hipfftDoubleComplex)*NX_per_GPU[n]*NY*NZ2) );
		checkCudaErrors( hipMalloc((void **)&what[n], sizeof(hipfftDoubleComplex)*NX_per_GPU[n]*NY*NZ2) );
		checkCudaErrors( hipMalloc((void **)&zhat[n], sizeof(hipfftDoubleComplex)*NX_per_GPU[n]*NY*NZ2) );

		checkCudaErrors( hipMalloc((void **)&temp[n], sizeof(hipfftDoubleComplex)*NX_per_GPU[n]*NY*NZ2) );
		checkCudaErrors( hipMalloc((void **)&temp_reorder[n], sizeof(hipfftDoubleComplex)*NX_per_GPU[n]*NZ2) );
		checkCudaErrors( hipMalloc((void **)&temp_advective[n], sizeof(hipfftDoubleComplex)*NX_per_GPU[n]*NY*NZ2) );
		
		checkCudaErrors( hipMallocManaged((void **)&Vrms[n], sizeof(double)*size_Stats) );
		checkCudaErrors( hipMallocManaged((void **)&KE[n], sizeof(double)*size_Stats) );
		checkCudaErrors( hipMallocManaged((void **)&epsilon[n], sizeof(double)*size_Stats) );
		checkCudaErrors( hipMallocManaged((void **)&eta[n], sizeof(double)*size_Stats) );
		checkCudaErrors( hipMallocManaged((void **)&l[n], sizeof(double)*size_Stats) );
		checkCudaErrors( hipMallocManaged((void **)&lambda[n], sizeof(double)*size_Stats) );
		checkCudaErrors( hipMallocManaged((void **)&chi[n], sizeof(double)*size_Stats) );
	
	// hipMallocManaged(&T4, sizeof(double)*size_Stats);
	// hipMallocManaged(&T5, sizeof(double)*size_Stats);
	// hipMallocManaged(&T5a, sizeof(double)*size_Stats);
	// hipMallocManaged(&T5b, sizeof(double)*size_Stats);
		printf("Memory allocated on Device #%d\n", n);
	}

	// Set pointers for real arrays
	u = (hipfftDoubleReal **)uhat;
	v = (hipfftDoubleReal **)vhat;
	w = (hipfftDoubleReal **)what;
	z = (hipfftDoubleReal **)zhat;

	// printf("Starting Timer...\n");
	// StartTimer();

	// Setup wavespace domain
	initializeWaveNumbers(nGPUs, k);

/////////////////////////////////////////////////////////////////////////////////////
// Calculate Turbulence statistics
/////////////////////////////////////////////////////////////////////////////////////

// Enter timestepping loop
	for (i = 0; i < size_Stats; ++i){

		// Calculate cation number based on how often data is saved
		c = i*n_save;

		// Import data to GPU memory and distribute across GPUs for calculations
		importFields_mgpu(nGPUs, start_x, NX_per_GPU, c, h_u, h_v, h_w, h_z, u, v, w, z);
		printf("Data imported successfully!\n");

		for(n=0; n<nGPUs; ++n){
			hipSetDevice(n);
			hipDeviceSynchronize();
		}

		// Transform real data to Fourier space
		forwardTransform(plan1d, plan2d, nGPUs, NX_per_GPU, start_x, NY_per_GPU, start_y, temp, temp_reorder, u);
		forwardTransform(plan1d, plan2d, nGPUs, NX_per_GPU, start_x, NY_per_GPU, start_y, temp, temp_reorder, v);
		forwardTransform(plan1d, plan2d, nGPUs, NX_per_GPU, start_x, NY_per_GPU, start_y, temp, temp_reorder, w);
		forwardTransform(plan1d, plan2d, nGPUs, NX_per_GPU, start_x, NY_per_GPU, start_y, temp, temp_reorder, z);

		for(n=0; n<nGPUs; ++n){
			hipSetDevice(n);
			hipDeviceSynchronize();
		}

		// Calculate RMS velocity
		calcTurbStats_mgpu(i, nGPUs, NY_per_GPU, start_y, k, uhat, vhat, what, zhat, Vrms, KE, epsilon, l, eta, lambda, chi);

		printf("The RMS velocity is %g \n", Vrms[0][i]);
		printf("The Kinetic Energy is %g \n", KE[0][i]);
		printf("The Dissipation Rate is %g \n", epsilon[0][i]);
		printf("The Integral Length Scale is %g \n", l[0][i]);
		printf("The Kolmogorov Length Scale is %g \n", eta[0][i]);
		printf("The Taylor Micro Scale is %g \n", lambda[0][i]);
		printf("The Scalar Dissipation is %g \n", chi[0][i]);
	}
	// Exit timestepping loop

	// Copy turbulent results from GPU to CPU memory
	printf("Copy results to CPU memory...\n");

	hipSetDevice(0);
	hipDeviceSynchronize();

	checkCudaErrors( hipMemcpyAsync(h_Vrms, Vrms[0], sizeof(double)*size_Stats, hipMemcpyDefault) );
	checkCudaErrors( hipMemcpyAsync(h_KE, KE[0], sizeof(double)*size_Stats, hipMemcpyDefault) );
	checkCudaErrors( hipMemcpyAsync(h_epsilon, epsilon[0], sizeof(double)*size_Stats, hipMemcpyDefault) );
	checkCudaErrors( hipMemcpyAsync(h_eta, eta[0], sizeof(double)*size_Stats, hipMemcpyDefault) );
	checkCudaErrors( hipMemcpyAsync(h_l, l[0], sizeof(double)*size_Stats, hipMemcpyDefault) );
	checkCudaErrors( hipMemcpyAsync(h_lambda, lambda[0], sizeof(double)*size_Stats, hipMemcpyDefault) );
	checkCudaErrors( hipMemcpyAsync(h_chi, chi[0], sizeof(double)*size_Stats, hipMemcpyDefault) );

	// Save turbulence data
	writeStats("Vrms", h_Vrms, .0);
	writeStats("epsilon", h_epsilon, .0);
	writeStats("eta", h_eta, .0);
	writeStats("KE", h_KE, .0);
	writeStats("lambda", h_lambda, .0);
	writeStats("l", h_l, .0);
	writeStats("chi", h_chi, .0);

	// Deallocate resources
	for(n = 0; n<nGPUs; ++n){
		hipfftDestroy(plan1d[n]);
		hipfftDestroy(plan2d[n]);
		hipfftDestroy(invplan2d[n]);
	}

	free(h_Vrms);
	free(h_KE);
	free(h_epsilon);
	free(h_eta);
	free(h_l);
	free(h_lambda);
	free(h_chi);


	// Deallocate GPU memory
	for(n = 0; n<nGPUs; ++n){
		hipSetDevice(n);

    hipFree(plan1d);
    hipFree(plan2d);
    hipFree(invplan2d);
    hipFree(worksize_f);
    hipFree(worksize_i);
    hipFree(workspace);

		hipFree(k[n]);

		hipFree(uhat[n]);
		hipFree(vhat[n]);
		hipFree(what[n]);
		hipFree(zhat[n]);

		hipFree(temp[n]);
		hipFree(temp_reorder[n]);
		hipFree(temp_advective[n]);
		
	}
	
	// Deallocate pointer arrays on host memory
	hipHostFree(k);

	hipHostFree(uhat);
	hipHostFree(vhat);
	hipHostFree(what);
	hipHostFree(zhat);

	hipHostFree(temp);
	hipHostFree(temp_reorder);
	hipHostFree(temp_advective);

	hipHostFree(Vrms);
	hipHostFree(KE);
	hipHostFree(epsilon);
	hipHostFree(eta);
	hipHostFree(l);
	hipHostFree(lambda);
	hipHostFree(chi);

	hipHostFree(plan1d);
	hipHostFree(plan2d);
	hipHostFree(invplan2d);

/////////////////////////////////////////////////////////////////////////////////////
// Finished calculating turbulence statistics
/////////////////////////////////////////////////////////////////////////////////////


/////////////////////////////////////////////////////////////////////////////////////
// Calculate Flame Surface properties
/////////////////////////////////////////////////////////////////////////////////////
	n = 1;
	hipSetDevice(n-1);		// Device is set to 0 as the flame surface properties is currently designed to run on a single GPU

	// Define the stoichiometric value of the mixture fraction:
 	int n_Z = 6;
	double Zst[n_Z] = {0.05, 0.1, 0.2, 0.3, 0.4, 0.5};
	// int n_Z = 1;
	// double Zst[n_Z] = {0.5};
	
	// Declare Variables
	int j;
	double *SurfArea;
	double *f;		// Mixture fraction data (Z data, but renamed it for the surface area calcs)
	
	// Allocate memory
	hipMallocManaged(&SurfArea, sizeof(double)*size_Stats);
	hipMallocManaged(&f, sizeof(double)*NN);

// Loop through values of Zst
/////////////////////////////////////////////////////////////////////////////////////
	for (j = 0; j < n_Z; ++j){

		// Initialize surface properties to 0
		hipMemset(SurfArea, 0.0, sizeof(double)*size_Stats);
		// hipMemset(T4, 0.0, sizeof(double)*size_Stats);
		// hipMemset(T5, 0.0, sizeof(double)*size_Stats);
		// hipMemset(T5a, 0.0, sizeof(double)*size_Stats);
		// hipMemset(T5b, 0.0, sizeof(double)*size_Stats);

// Enter timestepping loop
/////////////////////////////////////////////////////////////////////////////////////
		for (i = 0; i < size_Stats; ++i){

			// Calculate cation number based on how often data is saved
			c = i*n_save;

			// Import data to CPU memory for calculations
			importF(c, "z", f);

			// Calculate Integral Properties (uses only physical space variables)
			calcSurfaceArea(f, Zst[j], &SurfArea[i]);
			// calcSurfaceProps(plan, invplan, kx, u, v, w, z, Zst[j], &SurfArea[i], &T4[i], &T5[i], &T5a[i], &T5b[i]);

			hipDeviceSynchronize();

			printf("The Surface Area of the flame is %g \n", SurfArea[i]);
			// printf("The value of Term IV is %g \n", T4[i]);
			// printf("The value of Term V is %g \n", T5[i]);
			// printf("The value of Term Va is %g \n", T5a[i]);
			// printf("The value of Term Vb is %g \n", T5b[i]);

		}
		// Exit timestepping loop

		// Save Zst-dependent data
		writeStats("Area", SurfArea, Zst[j]);
		// writeStats("IV", T4, Zst[j]);
		// writeStats("V", T5, Zst[j]);
		// writeStats("Va", T5a, Zst[j]);
		// writeStats("Vb", T5b, Zst[j]);

	}
	// Exit Zst loop

	// Deallocate Variables
	hipFree(SurfArea);
	hipFree(f);

//////////////////////////////////////////////////////////////////////////////////////
// Finished calculating surface properties
//////////////////////////////////////////////////////////////////////////////////////
	
	printf("Analysis complete, Data saved!\n");

	hipDeviceReset();

	return 0;
}
*/