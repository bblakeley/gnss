#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#include <math.h>
#include <hipfft/hipfft.h>
#include <complex.h>
#include <hip/hip_complex.h>

#include "dnsparams.h"
#include "cudafuncs.h"
#include "initialize.h"
#include "fftfuncs.h"
#include "solver.h"
#include "iofuncs.h"

__global__ 
void initializeTGkernel(int start_x, hipfftDoubleReal *f1, hipfftDoubleReal *f2, hipfftDoubleReal *f3, hipfftDoubleReal *f4)
{
	const int i = blockIdx.x * blockDim.x + threadIdx.x;
	const int j = blockIdx.y * blockDim.y + threadIdx.y;
	const int k = blockIdx.z * blockDim.z + threadIdx.z;
	if (( (i+start_x) >= NX) || (j >= NY) || (k >= NZ)) return;
	const int idx = flatten( i, j, k, NX, NY, 2*NZ2);	// Index local to the GPU

/*	// For domain centered at (pi,pi,pi)
	double x = (i + start_x) * (double)LX / NX;
	double y = j * (double)LY / NY;
	double z = k * (double)LZ / NZ; */

	// For domain centered at (0,0,0):
	double x = -(double)LX/2 + (i + start_x)*(double)LX/NX;
	double y = -(double)LY/2 + j*(double)LY/NY;
	double z = -(double)LZ/2 + k*(double)LZ/NZ;

	// Initialize starting array - Taylor Green Vortex
	f1[idx] = sin(x)*cos(y)*cos(z);		// u
	f2[idx] = -cos(x)*sin(y)*cos(z);	// v
	f3[idx] = 0.0;						// w
	f4[idx] = 0.5 - 0.5*tanh( H/(4.0*theta)*( 2.0*fabs(y)/H - 1.0 ));	// z

	return;
}

void initializeTaylorGreen(gpudata gpu, fielddata vel)
{
	int n;
	for (n = 0; n<gpu.nGPUs; ++n){
		hipSetDevice(n);

		const dim3 blockSize(TX, TY, TZ);
		const dim3 gridSize(divUp(gpu.nx[n], TX), divUp(NY, TY), divUp(NZ, TZ));

		initializeTGkernel<<<gridSize, blockSize>>>(gpu.start_x[n], vel.u[n], vel.v[n], vel.w[n], vel.s[n]);
		printf("Velocity initialized on GPU #%d...\n",n);
	}

	return;
}

__global__
void hpFilterKernel_mgpu(int start_y, double *k1, double *k2, double *k3, hipfftDoubleComplex *fhat){

	const int i = blockIdx.x * blockDim.x + threadIdx.x;
	const int j = blockIdx.y * blockDim.y + threadIdx.y;
	const int k = blockIdx.z * blockDim.z + threadIdx.z;
	if (( i >= NX) || ((j+start_y) >= NY) || (k >= NZ2)) return;
	const int idx = flatten( j, i, k, NY, NX, NZ2);

	double k_sq = k1[i]*k1[i] + k2[(j+start_y)]*k2[(j+start_y)] + k3[k]*k3[k];

	if( k_sq <= (k_fil*k_fil) )
	{
		fhat[idx].x = 0.0;
		fhat[idx].y = 0.0;
	}

	return;
}

void hpFilter(gpudata gpu, fftdata fft, griddata grid, fielddata vel)
{	// Filter out low wavenumbers

	// Transform isotropic noise (stored in rhs_u) to Fourier Space
	forwardTransform(fft, gpu, vel.u);
	forwardTransform(fft, gpu, vel.v);
	forwardTransform(fft, gpu, vel.w);

	int n;
	for(n = 0; n<gpu.nGPUs; ++n){
		hipSetDevice(n);

		// Set thread and block dimensions for kernal calls
		const dim3 blockSize(TX, TY, TZ);
		const dim3 gridSize(divUp(NX, TX), divUp(gpu.ny[n], TY), divUp(NZ2, TZ));

		// Call the kernel
		hpFilterKernel_mgpu<<<gridSize, blockSize>>>(gpu.start_y[n], grid.kx[n], grid.ky[n], grid.kz[n], vel.uh[n]);
		hpFilterKernel_mgpu<<<gridSize, blockSize>>>(gpu.start_y[n], grid.kx[n], grid.ky[n], grid.kz[n], vel.vh[n]);
		hpFilterKernel_mgpu<<<gridSize, blockSize>>>(gpu.start_y[n], grid.kx[n], grid.ky[n], grid.kz[n], vel.wh[n]);
	}
	
	// Transform filtered noise back to physical space
	inverseTransform(fft, gpu, vel.uh);
	inverseTransform(fft, gpu, vel.vh);
	inverseTransform(fft, gpu, vel.wh);

	return;
}

__global__ 
void initializeVelocityKernel_mgpu(int start_x, hipfftDoubleReal *f1, hipfftDoubleReal *f2, hipfftDoubleReal *f3)
{
	const int i = blockIdx.x * blockDim.x + threadIdx.x;
	const int j = blockIdx.y * blockDim.y + threadIdx.y;
	const int k = blockIdx.z * blockDim.z + threadIdx.z;
	if (( (i+start_x) >= NX) || (j >= NY) || (k >= NZ)) return;
	const int idx = flatten( i, j, k, NX, NY, 2*NZ2);	// Index local to the GPU

	// Create physical vectors in temporary memory
	// For domain centered at (0,0,0):
	// double x = -(double)LX/2 + (i + start_x)*(double)LX/NX ;
	double y = -(double)LY/2 + j*(double)LY/NY;
	// double z = -(double)LZ/2 + k*(double)LZ/NZ;

/*
	// Initialize velocity array - adding shear profile onto isotropic velocity field
	f1[idx] = 0.5 - 0.5*tanh( H/(4.0*theta)*( 2.0*fabs(y)/H - 1.0 )) + 0.02*f1[idx];
	f2[idx] = 0.02*f2[idx];
	f3[idx] = 0.02*f3[idx];
*/
	// Initialize smooth jet velocity profile
	f1[idx] = 0.5 - 0.5*tanh( H/(4.0*theta)*( 2.0*fabs(y)/H - 1.0 ));
	f2[idx] = 0.0;
	f3[idx] = 0.0;


	return;
}

void initializeVelocity(gpudata gpu, fielddata vel)
{
	int n;
	for (n = 0; n<gpu.nGPUs; ++n){
		hipSetDevice(n);

		const dim3 blockSize(TX, TY, TZ);
		const dim3 gridSize(divUp(gpu.nx[n], TX), divUp(NY, TY), divUp(NZ, TZ));

		initializeVelocityKernel_mgpu<<<gridSize, blockSize>>>(gpu.start_x[n], vel.u[n], vel.v[n], vel.w[n]);
		printf("Velocity initialized on GPU #%d...\n",n);
	}

	return;
}

__global__ 
void initializeScalarKernel_mgpu(int start_x, hipfftDoubleReal *Z)
{	// Creates initial conditions in the physical domain
	const int i = blockIdx.x * blockDim.x + threadIdx.x;
	const int j = blockIdx.y * blockDim.y + threadIdx.y;
	const int k = blockIdx.z * blockDim.z + threadIdx.z;
	if (((i+start_x) >= NX) || (j >= NY) || (k >= NZ)) return;
	const int idx = flatten(i, j, k, NX, NY, 2*NZ2);	// Index local to each GPU

	// Create physical vectors in temporary memory
	double y = -(double)LY/2 + j*(double)LY/NY;

	// Initialize scalar field
	Z[idx] = 0.5 - 0.5*tanh( H/(4.0*theta)*( 2.0*fabs(y)/H - 1.0 ));

	return;
}

__global__ 
void initializeColloidKernel_mgpu(int start_x, hipfftDoubleReal *C)
{	// Creates initial conditions in the physical domain
	const int i = blockIdx.x * blockDim.x + threadIdx.x;
	const int j = blockIdx.y * blockDim.y + threadIdx.y;
	const int k = blockIdx.z * blockDim.z + threadIdx.z;
	if (((i+start_x) >= NX) || (j >= NY) || (k >= NZ)) return;
	const int idx = flatten(i, j, k, NX, NY, 2*NZ2);	// Index local to each GPU

	// Create physical vectors in temporary memory
	double y = -(double)LY/2 + j*(double)LY/NY;

	// Initialize scalar field
	C[idx] = 0.5 - 0.5*tanh( H/(4.0*theta_c)*( 2.0*fabs(y)/H - 1.0 ));

	return;
}

void initializeScalar(gpudata gpu, fielddata vel)
{
	int n;
	for (n = 0; n<gpu.nGPUs; ++n){
		hipSetDevice(n);

		const dim3 blockSize(TX, TY, TZ);
		const dim3 gridSize(divUp(gpu.nx[n], TX), divUp(NY, TY), divUp(NZ, TZ));

		initializeScalarKernel_mgpu<<<gridSize, blockSize>>>(gpu.start_x[n], vel.s[n]);
		initializeColloidKernel_mgpu<<<gridSize, blockSize>>>(gpu.start_x[n], vel.c[n]);
		printf("Scalar field initialized on GPU #%d...\n",n);
	}

	return;

}

__global__ 
void unit_test_kernel_mgpu(int start_x, hipfftDoubleReal *Z)
{	// Creates initial conditions in the physical domain
	const int i = blockIdx.x * blockDim.x + threadIdx.x;
	const int j = blockIdx.y * blockDim.y + threadIdx.y;
	const int k = blockIdx.z * blockDim.z + threadIdx.z;
	if (((i+start_x) >= NX) || (j >= NY) || (k >= NZ)) return;
	const int idx = flatten(i, j, k, NX, NY, 2*NZ2);	// Index local to each GPU

  double x = -(double)LX/2 + (i + start_x)*(double)LX/NX ;
	double y = -(double)LY/2 + j*(double)LY/NY;

	// Initialize scalar field
	Z[idx] = sin(x)*cos(y);

	return;
}

void init_unit_test(gpudata gpu, fftdata fft, fielddata vel)
{ // Initialize DNS data

	int n;
	for (n = 0; n<gpu.nGPUs; ++n){
		hipSetDevice(n);

		const dim3 blockSize(TX, TY, TZ);
		const dim3 gridSize(divUp(gpu.nx[n], TX), divUp(NY, TY), divUp(NZ, TZ));

		unit_test_kernel_mgpu<<<gridSize, blockSize>>>(gpu.start_x[n], vel.s[n]);
		printf("Scalar field initialized on GPU #%d...\n",n);
	}

	return;
}

__global__
void velocitySuperpositionKernel_mgpu(int start_x, hipfftDoubleReal *u, hipfftDoubleReal *v, hipfftDoubleReal *w, hipfftDoubleReal *noise_u, hipfftDoubleReal *noise_v, hipfftDoubleReal *noise_w, double scale )
{ // This function is designed to add a 3D isotropic turbulent velocity background perturbation
	// onto the shear layer region of a temporal jet.  
	const int i = blockIdx.x * blockDim.x + threadIdx.x;
	const int j = blockIdx.y * blockDim.y + threadIdx.y;
	const int k = blockIdx.z * blockDim.z + threadIdx.z;
	if (((i+start_x) >= NX) || (j >= NY) || (k >= NZ)) return;
	const int idx = flatten(i, j, k, NX, NY, 2*NZ2);	// Index local to each GPU

	u[idx] = u[idx] + scale*noise_u[idx];
	v[idx] = v[idx] + scale*noise_v[idx];
	w[idx] = w[idx] + scale*noise_w[idx];

	return;
}

// Adding isotropic velocity field only in shear layer of temporal jet
void initializeJet_Superposition(fftdata fft, gpudata gpu, griddata grid, fielddata h_vel, fielddata vel, fielddata rhs)
{
	int n;

	// Import isotropic velocity field
	importData(gpu, h_vel, rhs);

	// High-pass filter to remove lowest wavenumbers
	hpFilter(gpu, fft, grid, rhs);

	// Initialize smooth jet velocity field (hyperbolic tangent profile from da Silva and Pereira)
	initializeVelocity(gpu, vel);

	// Superimpose isotropic noise on top of jet velocity initialization
	for (n = 0; n<gpu.nGPUs; ++n){
		hipSetDevice(n);

		const dim3 blockSize(TX, TY, TZ);
		const dim3 gridSize(divUp(gpu.nx[n], TX), divUp(NY, TY), divUp(NZ, TZ));

		velocitySuperpositionKernel_mgpu<<<gridSize, blockSize>>>(gpu.start_x[n], vel.u[n], vel.v[n], vel.w[n], rhs.u[n], rhs.v[n], rhs.w[n], 0.02);
		printf("Superimposing Jet velocity profile with isotropic noise...\n");
	}	

	initializeScalar(gpu, vel);

	synchronizeGPUs(gpu.nGPUs);

	return;
}

double max_value(double **array, gpudata gpu)
{
    int i,j,k,idx,n;
    double max_val = -1.0;
    
    for (n=0; n<gpu.nGPUs; ++n){
     for (i=0; i<gpu.nx[n]; ++i){
	    for (j=0; j<NY; ++j){
	     for (k=0; k<NZ; ++k){
	      idx = k + j*2*NZ2 + i*2*NZ2*NY;
         if (fabs(array[n][idx]) > max_val)
          max_val = fabs(array[n][idx]);
       }
      }
     }
    }
    
    return max_val;
}

// Generate random, solenoidal velocity fields
void generateNoise(fftdata fft, gpudata gpu, griddata grid, fielddata h_vel, fielddata vel, fielddata rhs)
{
	int n,i,j,k,idx;
	double max_val;
	
	// Generate 3 random fields
	for (n = 0; n<gpu.nGPUs; ++n){
	  srand(n);
	  for (i=0; i<gpu.nx[n]; ++i){
	    for (j=0; j<NY; ++j){
	      for (k=0; k<NZ; ++k){
	        idx = k + j*2*NZ2 + i*2*NZ2*NY;
	        h_vel.u[n][idx] = (double)rand()/RAND_MAX*2.0-1.0;
	        h_vel.v[n][idx] = (double)rand()/RAND_MAX*2.0-1.0;
	        h_vel.w[n][idx] = (double)rand()/RAND_MAX*2.0-1.0;
	      }
	    }
	  }
	}
	
	// Copy random field to GPU
  for(n=0; n<gpu.nGPUs; ++n){
		hipSetDevice(n);
		hipDeviceSynchronize();
		checkCudaErrors( hipMemcpyAsync(vel.u[n], h_vel.u[n], sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2, hipMemcpyDefault) );
		checkCudaErrors( hipMemcpyAsync(vel.v[n], h_vel.v[n], sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2, hipMemcpyDefault) );
		checkCudaErrors( hipMemcpyAsync(vel.w[n], h_vel.w[n], sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2, hipMemcpyDefault) );
	}
	
	// Transform random field to Fourier space
	forwardTransform(fft, gpu, vel.u);
	forwardTransform(fft, gpu, vel.v);
	forwardTransform(fft, gpu, vel.w);
	
	// Take curl of random scalar field, stored in rhs
	vorticity(gpu, grid, vel, rhs);
	
	// Remove highest wavenumber modes from random noise
	deAlias(gpu, grid, rhs);
		
	// Transform from Fourier Space to physical space for normalization
	inverseTransform(fft, gpu, rhs.uh);
	inverseTransform(fft, gpu, rhs.vh);
	inverseTransform(fft, gpu, rhs.wh);
  
  // Copy to host for normalization
  for(n=0; n<gpu.nGPUs; ++n){
		hipSetDevice(n);
		hipDeviceSynchronize();
		checkCudaErrors( hipMemcpyAsync(h_vel.u[n], rhs.u[n], sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2, hipMemcpyDefault) );
		checkCudaErrors( hipMemcpyAsync(h_vel.v[n], rhs.v[n], sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2, hipMemcpyDefault) );
		checkCudaErrors( hipMemcpyAsync(h_vel.w[n], rhs.w[n], sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2, hipMemcpyDefault) );
	}
	
  // Find maximum value in array to normalize against
  max_val = max_value(h_vel.u, gpu);
	
	// Normalize curl of random fields to 1,-1
	for (n = 0; n<gpu.nGPUs; ++n){
	  for (i=0; i<gpu.nx[n]; ++i){
	    for (j=0; j<NY; ++j){
	      for (k=0; k<NZ; ++k){
	        idx = k + j*2*NZ2 + i*2*NZ2*NY;
	        h_vel.u[n][idx] = h_vel.u[n][idx]/max_val;
	        h_vel.v[n][idx] = h_vel.v[n][idx]/max_val;
	        h_vel.w[n][idx] = h_vel.w[n][idx]/max_val;
	      }
	    }
	  }
	}	

	// Copy random, solenoidal random field normalized to [1,-1] to GPU
  for(n=0; n<gpu.nGPUs; ++n){
		hipSetDevice(n);
		hipDeviceSynchronize();
		checkCudaErrors( hipMemcpyAsync(rhs.u[n], h_vel.u[n], sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2, hipMemcpyDefault) );
		checkCudaErrors( hipMemcpyAsync(rhs.v[n], h_vel.v[n], sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2, hipMemcpyDefault) );
		checkCudaErrors( hipMemcpyAsync(rhs.w[n], h_vel.w[n], sizeof(hipfftDoubleComplex)*gpu.nx[n]*NY*NZ2, hipMemcpyDefault) );
	}
	
	return;
}

// Initializing temporal jet and adding random noise to velocity field
void initializeJet(fftdata fft, gpudata gpu, griddata grid, fielddata h_vel, fielddata vel, fielddata rhs)
{
	int n;
	
	// Generate psuedo-random velocity field
	generateNoise(fft, gpu, grid, h_vel, vel, rhs);

	// Initialize smooth jet velocity field (hyperbolic tangent profile from da Silva and Pereira)
	initializeVelocity(gpu, vel);

	// Superimpose isotropic noise on top of jet velocity initialization
	for (n = 0; n<gpu.nGPUs; ++n){
		hipSetDevice(n);

		const dim3 blockSize(TX, TY, TZ);
		const dim3 gridSize(divUp(gpu.nx[n], TX), divUp(NY, TY), divUp(NZ, TZ));

		velocitySuperpositionKernel_mgpu<<<gridSize, blockSize>>>(gpu.start_x[n], vel.u[n], vel.v[n], vel.w[n], rhs.u[n], rhs.v[n], rhs.w[n], pert_amp);
		printf("Superimposing Jet velocity profile with isotropic noise...\n");
	}	

	initializeScalar(gpu, vel);

	synchronizeGPUs(gpu.nGPUs);

	return;
}

__global__
void scaleDataKernel_mgpu(int start_x, hipfftDoubleReal *u, hipfftDoubleReal *v, hipfftDoubleReal *w, double val)
{ // This function is designed to add a 3D isotropic turbulent velocity background perturbation
	// onto the shear layer region of a temporal jet.  
	const int i = blockIdx.x * blockDim.x + threadIdx.x;
	const int j = blockIdx.y * blockDim.y + threadIdx.y;
	const int k = blockIdx.z * blockDim.z + threadIdx.z;
	if (((i+start_x) >= NX) || (j >= NY) || (k >= NZ)) return;
	const int idx = flatten(i, j, k, NX, NY, 2*NZ2);	// Index local to each GPU

	u[idx] = val*u[idx];
	v[idx] = val*v[idx];
	w[idx] = val*w[idx];

	return;
}

void scaleData(gpudata gpu, fielddata vel, double val)
{	// Subroutine to scale the velocity field prior to convolution

	int n;

	for (n = 0; n<gpu.nGPUs; ++n){
		hipSetDevice(n);

		const dim3 blockSize(TX, TY, TZ);
		const dim3 gridSize(divUp(gpu.nx[n], TX), divUp(NY, TY), divUp(NZ, TZ));

		scaleDataKernel_mgpu<<<gridSize, blockSize>>>(gpu.start_x[n], vel.u[n], vel.v[n], vel.w[n], val);
		printf("Scaling isotropic velocity...\n");
	}	

	return;
}

__global__
void waveNumber_kernel(int n, double l, double *waveNum)
{   // Creates the wavenumber vectors used in Fourier space
	const int i = blockIdx.x*blockDim.x + threadIdx.x;

	if (i >= n) return;

	if (i < n/2)
		waveNum[i] = (2*PI/l)*(double)i;
	else
		waveNum[i] = (2*PI/l)*( (double)i - n );

	return;
}

void initializeWaveNumbers(gpudata gpu, griddata grid)
{    // Initialize wavenumbers in Fourier space

	int n;
	for (n = 0; n<gpu.nGPUs; ++n){
		hipSetDevice(n);
		waveNumber_kernel<<<divUp(NX,TX), TX>>>(NX,LX,grid.kx[n]);
		waveNumber_kernel<<<divUp(NY,TX), TX>>>(NY,LY,grid.ky[n]);
		waveNumber_kernel<<<divUp(NZ,TX), TX>>>(NZ,LZ,grid.kz[n]);
	}

	printf("Wave domain setup complete..\n");

	return;
}
