#include <stdlib.h>
#include <complex.h>
#include <hipfft/hipfft.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#include "dnsparams.h"
#include "struct_def.h"
#include "deallocate.h"

void deallocate_memory(){
	int n, nGPUs;
	// // Declare extern variables (to pull def's from declare.h)
	extern gpuinfo gpu;	
	extern fftinfo fft;
	extern statistics h_stats;
	extern statistics stats;	

  extern double **k;

  extern fielddata h_vel;
  extern fielddata vel;
  extern fielddata rhs;
  extern fielddata rhs_old;

	extern hipfftDoubleComplex **temp_advective;

	// Make local copy of number of GPUs (for readability)
	nGPUs = gpu.nGPUs;

	// Deallocate GPU memory
	for(n = 0; n<nGPUs; ++n){
		hipSetDevice(n);

		hipFree(fft.temp[n]);
		hipFree(fft.temp_reorder[n]);
   	hipFree(fft.wspace[n]);

		hipFree(k[n]);

		free(h_vel.u[n]);
		free(h_vel.v[n]);
		free(h_vel.w[n]);
		free(h_vel.s[n]);

		hipFree(vel.u[n]);
		hipFree(vel.v[n]);
		hipFree(vel.w[n]);
		hipFree(vel.s[n]);

		hipFree(rhs.u[n]);
		hipFree(rhs.v[n]);
		hipFree(rhs.w[n]);
		hipFree(rhs.s[n]);

		hipFree(rhs_old.u[n]);
		hipFree(rhs_old.v[n]);
		hipFree(rhs_old.w[n]);
		hipFree(rhs_old.s[n]);

		hipFree(temp_advective[n]);

		hipFree(stats.Vrms[n]);
		hipFree(stats.KE[n]);
		hipFree(stats.epsilon[n]);
		hipFree(stats.eta[n]);
		hipFree(stats.l[n]);
		hipFree(stats.lambda[n]);
		hipFree(stats.chi[n]);
		hipFree(stats.area_scalar[n]);
		hipFree(stats.area_tnti[n]);
		hipFree(stats.energy_spect[n]);

		// Destroy cufft plans
		hipfftDestroy(fft.p1d[n]);
		hipfftDestroy(fft.p2d[n]);
		hipfftDestroy(fft.invp2d[n]);
	}
	
	// Deallocate pointer arrays on host memory
	hipHostFree(gpu.gpunum);
	hipHostFree(gpu.ny);
	hipHostFree(gpu.nx);
	hipHostFree(gpu.start_x);
	hipHostFree(gpu.start_y);

	hipHostFree(k);

	hipHostFree(temp_advective);

	hipHostFree(fft.wsize_f);
	hipHostFree(fft.wsize_i);
	hipHostFree(fft.wspace);
	hipHostFree(fft.temp);
	hipHostFree(fft.temp_reorder);
	hipHostFree(&fft);

	// hipHostFree(vel.u);
	// hipHostFree(vel.v);
	// hipHostFree(vel.w);
	// hipHostFree(vel.s);

	hipHostFree(vel.uh);
	hipHostFree(vel.vh);
	hipHostFree(vel.wh);
	hipHostFree(vel.sh);

	// hipHostFree(rhs.u);
	// hipHostFree(rhs.v);
	// hipHostFree(rhs.w);
	// hipHostFree(rhs.s);

	hipHostFree(rhs.uh);
	hipHostFree(rhs.vh);
	hipHostFree(rhs.wh);
	hipHostFree(rhs.sh);

	// hipHostFree(rhs_old.u);
	// hipHostFree(rhs_old.v);
	// hipHostFree(rhs_old.w);
	// hipHostFree(rhs_old.s);

	hipHostFree(rhs_old.uh);
	hipHostFree(rhs_old.vh);
	hipHostFree(rhs_old.wh);
	hipHostFree(rhs_old.sh);

	hipHostFree(stats.Vrms);
	hipHostFree(stats.KE);
	hipHostFree(stats.epsilon);
	hipHostFree(stats.eta);
	hipHostFree(stats.l);
	hipHostFree(stats.lambda);
	hipHostFree(stats.chi);
	hipHostFree(stats.area_scalar);
	hipHostFree(stats.area_tnti);
	hipHostFree(stats.energy_spect);

	// Deallocate memory on CPU
	free(h_vel.u);
	free(h_vel.v);
	free(h_vel.w);
	free(h_vel.s);

	free(h_stats.Vrms[0]   );
	free(h_stats.KE[0]     );
	free(h_stats.epsilon[0]);
	free(h_stats.eta[0]    );
	free(h_stats.l[0]      );
	free(h_stats.lambda[0] );
	free(h_stats.chi[0]    );
	free(h_stats.area_scalar[0]);
	free(h_stats.area_tnti[0]  );
	free(h_stats.energy_spect[0]  );

	free(h_stats.Vrms   );
	free(h_stats.KE     );
	free(h_stats.epsilon);
	free(h_stats.eta    );
	free(h_stats.l      );
	free(h_stats.lambda );
	free(h_stats.chi    );
	free(h_stats.area_scalar);
	free(h_stats.area_tnti  );
	free(h_stats.energy_spect  );

	return;
}